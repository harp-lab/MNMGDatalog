#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/kernels.cu"

using namespace std;


#define BLOCK_START(process_id, total_process, n) ((process_id)*(n)/(total_process))
#define BLOCK_SIZE(process_id, total_process, n) \
    (BLOCK_START(process_id + 1, total_process, n) - BLOCK_START(process_id, total_process, n))


Entity *get_split_relation(int rank, Entity *local_data_device,
                           int row_size, int total_columns, int nprocs,
                           int grid_size, int block_size, int cuda_aware_mpi, int *size, MPI_Comm comm) {
    int *send_count;
    checkCuda(hipMalloc((void **) &send_count, nprocs * sizeof(int)));
    checkCuda(hipMemset(send_count, 0, nprocs * sizeof(int)));
    int *send_displacements;
    checkCuda(hipMalloc((void **) &send_displacements, nprocs * sizeof(int)));
    checkCuda(hipMemset(send_displacements, 0, nprocs * sizeof(int)));
    int *send_displacements_temp;
    checkCuda(hipMalloc((void **) &send_displacements_temp, nprocs * sizeof(int)));
    checkCuda(hipMemset(send_displacements_temp, 0, nprocs * sizeof(int)));

    get_send_count<<<grid_size, block_size>>>(local_data_device, row_size, send_count, nprocs);
    thrust::exclusive_scan(thrust::device, send_count, send_count + nprocs, send_displacements);
    hipMemcpy(send_displacements_temp, send_displacements, nprocs * sizeof(int), hipMemcpyDeviceToDevice);
    Entity *send_data;
    checkCuda(hipMalloc((void **) &send_data, row_size * sizeof(Entity)));
    get_rank_data<<<grid_size, block_size>>>(local_data_device, row_size, send_displacements_temp, nprocs, send_data);
    checkCuda(hipDeviceSynchronize());
    int *receive_count;
    checkCuda(hipMalloc((void **) &receive_count, nprocs * sizeof(int)));
    checkCuda(hipMemset(receive_count, 0, nprocs * sizeof(int)));

    int *receive_displacements;
    checkCuda(hipMalloc((void **) &receive_displacements, nprocs * sizeof(int)));
    checkCuda(hipMemset(receive_displacements, 0, nprocs * sizeof(int)));

    if (cuda_aware_mpi) {
        int mpi_error = MPI_Alltoall(send_count, 1, MPI_INT, receive_count, 1, MPI_INT, MPI_COMM_WORLD);
        if (mpi_error != MPI_SUCCESS) {
            char error_string[BUFSIZ];
            int length_of_error_string;
            MPI_Error_string(mpi_error, error_string, &length_of_error_string);
            fprintf(stderr, "MPI error on device MPI_Alltoall call: %s\n", error_string);
            MPI_Abort(MPI_COMM_WORLD, mpi_error);
        }
    } else {
        int *send_count_host = (int *) malloc(nprocs * sizeof(int));;
        int *receive_count_host = (int *) malloc(nprocs * sizeof(int));;
        hipMemcpy(send_count_host, send_count, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        int mpi_error = MPI_Alltoall(send_count_host, 1, MPI_INT, receive_count_host, 1, MPI_INT, MPI_COMM_WORLD);
        if (mpi_error != MPI_SUCCESS) {
            char error_string[BUFSIZ];
            int length_of_error_string;
            MPI_Error_string(mpi_error, error_string, &length_of_error_string);
            fprintf(stderr, "MPI error on host MPI_Alltoall call: %s\n", error_string);
            MPI_Abort(MPI_COMM_WORLD, mpi_error);
        }
        hipMemcpy(receive_count, receive_count_host, nprocs * sizeof(int), hipMemcpyHostToDevice);
        free(send_count_host);
        free(receive_count_host);
    }
    int total_receive = thrust::reduce(thrust::device, receive_count, receive_count + nprocs, 0, thrust::plus<int>());
    thrust::exclusive_scan(thrust::device, receive_count, receive_count + nprocs, receive_displacements);
    checkCuda(hipDeviceSynchronize());
    Entity *receive_data;
    checkCuda(hipMalloc((void **) &receive_data, total_receive * sizeof(Entity)));
    if (cuda_aware_mpi) {

        int *send_count_host = (int *) malloc(nprocs * sizeof(int));
        int *receive_count_host = (int *) malloc(nprocs * sizeof(int));
        int *send_displacements_host = (int *) malloc(nprocs * sizeof(int));
        int *receive_displacements_host = (int *) malloc(nprocs * sizeof(int));
        hipMemcpy(send_count_host, send_count, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(receive_count_host, receive_count, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(send_displacements_host, send_displacements, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(receive_displacements_host, receive_displacements, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        int mpi_error = MPI_Alltoallv(send_data, send_count_host, send_displacements_host, MPI_UINT64_T,
                                      receive_data, receive_count_host, receive_displacements_host, MPI_UINT64_T,
                                      MPI_COMM_WORLD);
        if (mpi_error != MPI_SUCCESS) {
            char error_string[BUFSIZ];
            int length_of_error_string;
            MPI_Error_string(mpi_error, error_string, &length_of_error_string);
            fprintf(stderr, "MPI error on device MPI_Alltoallv call: %s\n", error_string);
            MPI_Abort(MPI_COMM_WORLD, mpi_error);
        }
    } else {
        int *send_count_host = (int *) malloc(nprocs * sizeof(int));;
        int *receive_count_host = (int *) malloc(nprocs * sizeof(int));;
        int *send_displacements_host = (int *) malloc(nprocs * sizeof(int));;
        int *receive_displacements_host = (int *) malloc(nprocs * sizeof(int));;
        Entity *send_data_host = (Entity *) malloc(row_size * sizeof(Entity));;
        Entity *receive_data_host = (Entity *) malloc(total_receive * sizeof(Entity));;
        hipMemcpy(send_count_host, send_count, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(receive_count_host, receive_count, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(send_displacements_host, send_displacements, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(receive_displacements_host, receive_displacements, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(send_data_host, send_data, row_size * sizeof(Entity), hipMemcpyDeviceToHost);
        int mpi_error = MPI_Alltoallv(send_data_host, send_count_host, send_displacements_host, MPI_UINT64_T,
                                      receive_data_host, receive_count_host, receive_displacements_host, MPI_UINT64_T,
                                      MPI_COMM_WORLD);
        if (mpi_error != MPI_SUCCESS) {
            char error_string[BUFSIZ];
            int length_of_error_string;
            MPI_Error_string(mpi_error, error_string, &length_of_error_string);
            fprintf(stderr, "MPI error on host MPI_Alltoallv call: %s\n", error_string);
            MPI_Abort(MPI_COMM_WORLD, mpi_error);
        }
        hipMemcpy(receive_data, receive_data_host, total_receive * sizeof(Entity), hipMemcpyHostToDevice);
        free(send_count_host);
        free(receive_count_host);
        free(send_displacements_host);
        free(receive_displacements_host);
        free(send_data_host);
        free(receive_data_host);
    }
    *size = total_receive;
    Entity *result_data;
    checkCuda(hipMalloc((void **) &result_data, total_receive * sizeof(Entity)));
    hipMemcpy(result_data, receive_data, total_receive * sizeof(Entity), hipMemcpyDeviceToDevice);
    hipFree(send_count);
    hipFree(receive_count);
    hipFree(send_displacements);
    hipFree(send_displacements_temp);
    hipFree(receive_displacements);
    hipFree(send_data);
    hipFree(receive_data);
    return result_data;
}

int main(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    double elapsed_time = -MPI_Wtime();
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    setlocale(LC_ALL, "");
    double max_time = 0.0;
    int nprocs, rank;
    int i;
    MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    // Should pass the input filename in command line argument
    const char *input_file;
    int cuda_aware_mpi = 0;
    if (argc == 3) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
    } else if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "hipc_2019.bin";
    }
    // READ THE FILE IN PARALLEL
    // Reading filesize in bytes
    struct stat filestats;
    stat(input_file, &filestats);
    off_t filesize = filestats.st_size;

    // Calculating the current rank's starting row and number of rows
    // Scatter larger blocks among processes (non-uniform)
    int total_columns = 2;
    int total_rows = filesize / (sizeof(int) * total_columns);
    int row_start = BLOCK_START(rank, nprocs, total_rows);
    int row_size = BLOCK_SIZE(rank, nprocs, total_rows);
    int local_count = row_size * total_columns;

    // Reading specific portion from the file as char in parallel
    int offset = row_start * total_columns * sizeof(int);
    int *local_data_host = (int *) malloc(local_count * sizeof(int));
    MPI_File mpi_file_buffer;
    if (MPI_File_open(MPI_COMM_WORLD, input_file, MPI_MODE_RDONLY,
                      MPI_INFO_NULL, &mpi_file_buffer) != MPI_SUCCESS) {
        printf("Error opening file %s", input_file);
        MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    }
    MPI_File_read_at(mpi_file_buffer, offset, local_data_host, local_count, MPI_INT, MPI_STATUS_IGNORE);
    MPI_File_close(&mpi_file_buffer);


    int *local_data_device;
    checkCuda(hipMalloc((void **) &local_data_device, local_count * sizeof(int)));
    hipMemcpy(local_data_device, local_data_host, local_count * sizeof(int), hipMemcpyHostToDevice);

    Entity *local_data;
    checkCuda(hipMalloc((void **) &local_data, row_size * sizeof(Entity)));
    Entity *local_data_reverse;
    checkCuda(hipMalloc((void **) &local_data_reverse, row_size * sizeof(Entity)));
    create_entity_ar<<<grid_size, block_size>>>(local_data, row_size, local_data_device);
    create_entity_ar_reverse<<<grid_size, block_size>>>(local_data_reverse, row_size, local_data_device);
    int input_relation_size = 0;
    Entity *input_relation = get_split_relation(rank, local_data,
                                                row_size, total_columns, nprocs,
                                                grid_size, block_size, cuda_aware_mpi, &input_relation_size,
                                                MPI_COMM_WORLD);

    int t_delta_size;
    Entity *t_delta = get_split_relation(rank, local_data_reverse,
                                         row_size, total_columns, nprocs,
                                         grid_size, block_size, cuda_aware_mpi, &t_delta_size,
                                         MPI_COMM_WORLD);
    thrust::stable_sort(thrust::device, t_delta, t_delta + t_delta_size, set_cmp());
    t_delta_size = (thrust::unique(thrust::device,
                                   t_delta, t_delta + t_delta_size,
                                   is_equal())) - t_delta;

    // T_FULL is t delta with first column as key
    Entity *t_full;
    checkCuda(hipMalloc((void **) &t_full, t_delta_size * sizeof(Entity)));
    hipMemcpy(t_full, t_delta, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);

    int global_t_full_size;
    int t_full_size = t_delta_size;
    MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);

    Entity *hash_table;
    double load_factor = 0.4;
    int hash_table_rows = (int) input_relation_size / load_factor;
    hash_table_rows = pow(2, ceil(log(hash_table_rows) / log(2)));
    checkCuda(hipMalloc((void **) &hash_table, hash_table_rows * sizeof(Entity)));
    Entity negative_entity;
    negative_entity.key = -1;
    negative_entity.value = -1;
//    cout << "Size of entity: " << sizeof(negative_entity) << endl;
    thrust::fill(thrust::device, hash_table, hash_table + hash_table_rows, negative_entity);
    build_hash_table_entity<<<grid_size, block_size>>>(hash_table, hash_table_rows, input_relation,
                                                       input_relation_size);
    int iterations = 0;


    while (true) {
        int join_result_size;
        int *join_offset;
        Entity *join_result;
        Entity *new_t_full;
        checkCuda(hipMalloc((void **) &join_offset, t_delta_size * sizeof(int)));
        checkCuda(hipMemset(join_offset, 0, t_delta_size * sizeof(int)));

        get_join_result_size_entity<<<grid_size, block_size>>>(hash_table, hash_table_rows,
                                                               t_delta, t_delta_size, join_offset);
        join_result_size = thrust::reduce(thrust::device, join_offset, join_offset + t_delta_size, 0);
        thrust::exclusive_scan(thrust::device, join_offset, join_offset + t_delta_size, join_offset);
        checkCuda(hipMalloc((void **) &join_result, join_result_size * sizeof(Entity)));

        get_join_result_entity<<<grid_size, block_size>>>(hash_table, hash_table_rows,
                                                          t_delta, t_delta_size, join_offset, join_result);

        // Scatter the new facts among relevant processes
        Entity *t_delta_temp = get_split_relation(rank, join_result,
                                                  join_result_size, total_columns, nprocs,
                                                  grid_size, block_size, cuda_aware_mpi, &t_delta_size,
                                                  MPI_COMM_WORLD);
        // Deduplicate scattered facts
        thrust::stable_sort(thrust::device, t_delta_temp, t_delta_temp + t_delta_size, set_cmp());
        t_delta_size = (thrust::unique(thrust::device,
                                       t_delta_temp, t_delta_temp + t_delta_size,
                                       is_equal())) - t_delta_temp;
        hipFree(t_delta);
        checkCuda(hipMalloc((void **) &t_delta, t_delta_size * sizeof(Entity)));
        hipMemcpy(t_delta, t_delta_temp, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);

        // set union of two sets (sorted t full and t delta)
        int new_t_full_size = t_delta_size + t_full_size;
        checkCuda(hipMalloc((void **) &new_t_full, new_t_full_size * sizeof(Entity)));
        new_t_full_size = thrust::set_union(thrust::device,
                                            t_full, t_full + t_full_size,
                                            t_delta, t_delta + t_delta_size,
                                            new_t_full, set_cmp()) - new_t_full;

        // Update t delta which is the only new facts which are not in t full and will be used in next iteration
        t_delta_size = thrust::set_difference(thrust::device,
                                              new_t_full, new_t_full + new_t_full_size,
                                              t_full, t_full + t_full_size,
                                              t_delta, set_cmp()) - t_delta;
        hipFree(t_full);
        checkCuda(hipMalloc((void **) &t_full, new_t_full_size * sizeof(Entity)));
        hipMemcpy(t_full, new_t_full, new_t_full_size * sizeof(Entity), hipMemcpyDeviceToDevice);
        t_full_size = new_t_full_size;
        hipFree(join_offset);
        hipFree(join_result);
        hipFree(new_t_full);
        hipFree(t_delta_temp);
        // Check if the global t full size has changed in this iteration
        int old_global_t_full_size = global_t_full_size;
        MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
        iterations++;
        if (old_global_t_full_size == global_t_full_size) {
            break;
        }
    }

    // Reverse the t_full as we stored it in reverse order initially
    int *t_full_ar;
    checkCuda(hipMalloc((void **) &t_full_ar, t_full_size * total_columns * sizeof(int)));
    reverse_t_full<<<grid_size, block_size>>>(t_full_ar, t_full_size, t_full);
    int *t_full_ar_host = (int *) malloc(t_full_size * total_columns * sizeof(int));
    hipMemcpy(t_full_ar_host, t_full_ar, t_full_size * total_columns * sizeof(int), hipMemcpyDeviceToHost);


    // List the t full counts for each process and calculate the displacements in the final result
    int *t_full_counts = (int *) calloc(nprocs, sizeof(int));
    MPI_Allgather(&t_full_size, 1, MPI_INT,
                  t_full_counts, 1, MPI_INT, MPI_COMM_WORLD);

    int *t_full_displacements = (int *) calloc(nprocs, sizeof(int));
    for (i = 1; i < nprocs; i++) {
        t_full_displacements[i] = t_full_displacements[i - 1] + (t_full_counts[i - 1] * total_columns);
    }

    // Write the t full to an offset of the output file
    MPI_File fh;
    string output_file = string(input_file) + "_tc.bin";
    const char *output_file_name = output_file.c_str();
    MPI_File_open(MPI_COMM_WORLD, output_file_name, MPI_MODE_WRONLY | MPI_MODE_CREATE, MPI_INFO_NULL, &fh);
    int file_offset = t_full_displacements[rank] * sizeof(int);
    MPI_File_write_at(fh, file_offset, t_full_ar_host, t_full_size * total_columns, MPI_INT, MPI_STATUS_IGNORE);
    // Close the file and clean up
    MPI_File_close(&fh);

    hipFree(local_data_device);
    hipFree(input_relation);
    hipFree(local_data);
    hipFree(local_data_reverse);
    hipFree(t_full);
    hipFree(t_delta);
    hipFree(t_full_ar);
    hipFree(hash_table);

    free(t_full_ar_host);
    free(t_full_counts);
    free(t_full_displacements);
    free(local_data_host);
    elapsed_time += MPI_Wtime();
    MPI_Allreduce(&elapsed_time, &max_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    if (rank == 0) {
        printf("\nGenerated file %s\n", output_file_name);
        printf("| # Input | # Process | # Iterations | # TC | Time (s) |\n");
        printf("| --- | --- | --- | --- | --- |\n");
        printf("| %'d | %'d | %'d | %'d | %'8.4lf |\n", total_rows, nprocs, iterations, global_t_full_size, max_time);
    }

    MPI_Finalize();
    return 0;
}
// make runsemi DATA_FILE=data/data_10.bin NPROCS=8 CUDA_AWARE_MPI=0
// make runsemi DATA_FILE=data/data_23874.bin NPROCS=8 CUDA_AWARE_MPI=1
// make runsemi DATA_FILE=data/data_147892.bin NPROCS=8 CUDA_AWARE_MPI=0
