#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/kernels.cu"

using namespace std;


#define BLOCK_START(process_id, total_process, n) ((process_id)*(n)/(total_process))
#define BLOCK_SIZE(process_id, total_process, n) \
    (BLOCK_START(process_id + 1, total_process, n) - BLOCK_START(process_id, total_process, n))


Entity *get_split_relation_pass_method(int rank, Entity *local_data_device,
                                       int row_size, int total_columns, int total_rank,
                                       int grid_size, int block_size, int cuda_aware_mpi, int *size) {
    int *send_count;
    checkCuda(hipMalloc((void **) &send_count, total_rank * sizeof(int)));
    checkCuda(hipMemset(send_count, 0, total_rank * sizeof(int)));
    int *send_displacements;
    checkCuda(hipMalloc((void **) &send_displacements, total_rank * sizeof(int)));
    checkCuda(hipMemset(send_displacements, 0, total_rank * sizeof(int)));
    int *send_displacements_temp;
    checkCuda(hipMalloc((void **) &send_displacements_temp, total_rank * sizeof(int)));
    checkCuda(hipMemset(send_displacements_temp, 0, total_rank * sizeof(int)));
    get_send_count<<<grid_size, block_size>>>(local_data_device, row_size, send_count, total_rank);
    thrust::exclusive_scan(thrust::device, send_count, send_count + total_rank, send_displacements);
    hipMemcpy(send_displacements_temp, send_displacements, total_rank * sizeof(int), hipMemcpyDeviceToDevice);
    Entity *send_data;
    checkCuda(hipMalloc((void **) &send_data, row_size * sizeof(Entity)));
    get_rank_data<<<grid_size, block_size>>>(local_data_device, row_size, send_displacements_temp, total_rank,
                                             send_data);
    int mpi_error;

    int *send_count_host = (int *) malloc(total_rank * sizeof(int));
    int *receive_count_host = (int *) malloc(total_rank * sizeof(int));
    int *send_displacements_host = (int *) malloc(total_rank * sizeof(int));
    int *receive_displacements_host = (int *) malloc(total_rank * sizeof(int));
    hipMemcpy(send_count_host, send_count, total_rank * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(send_displacements_host, send_displacements, total_rank * sizeof(int), hipMemcpyDeviceToHost);

    mpi_error = MPI_Alltoall(send_count_host, 1, MPI_INT, receive_count_host, 1, MPI_INT, MPI_COMM_WORLD);
    if (mpi_error != MPI_SUCCESS) {
        char error_string[BUFSIZ];
        int length_of_error_string;
        MPI_Error_string(mpi_error, error_string, &length_of_error_string);
        fprintf(stderr, "MPI error on MPI_Alltoall call: %s\n", error_string);
        MPI_Abort(MPI_COMM_WORLD, mpi_error);
    }

    int total_receive = thrust::reduce(receive_count_host, receive_count_host + total_rank, 0, thrust::plus<int>());
    thrust::exclusive_scan(receive_count_host, receive_count_host + total_rank, receive_displacements_host);
    Entity *receive_data;
    checkCuda(hipMalloc((void **) &receive_data, total_receive * sizeof(Entity)));

    if (cuda_aware_mpi) {
        mpi_error = MPI_Alltoallv(send_data, send_count_host, send_displacements_host, MPI_UINT64_T,
                                  receive_data, receive_count_host, receive_displacements_host, MPI_UINT64_T,
                                  MPI_COMM_WORLD);
        if (mpi_error != MPI_SUCCESS) {
            char error_string[BUFSIZ];
            int length_of_error_string;
            MPI_Error_string(mpi_error, error_string, &length_of_error_string);
            fprintf(stderr, "MPI error on CUDA AWARE MPI MPI_Alltoallv call: %s\n", error_string);
            MPI_Abort(MPI_COMM_WORLD, mpi_error);
        }
    } else {
        Entity *send_data_host = (Entity *) malloc(row_size * sizeof(Entity));
        Entity *receive_data_host = (Entity *) malloc(total_receive * sizeof(Entity));
        hipMemcpy(send_data_host, send_data, row_size * sizeof(Entity), hipMemcpyDeviceToHost);
        mpi_error = MPI_Alltoallv(send_data_host, send_count_host, send_displacements_host, MPI_UINT64_T,
                                  receive_data_host, receive_count_host, receive_displacements_host, MPI_UINT64_T,
                                  MPI_COMM_WORLD);
        if (mpi_error != MPI_SUCCESS) {
            char error_string[BUFSIZ];
            int length_of_error_string;
            MPI_Error_string(mpi_error, error_string, &length_of_error_string);
            fprintf(stderr, "MPI error on host MPI_Alltoallv call: %s\n", error_string);
            MPI_Abort(MPI_COMM_WORLD, mpi_error);
        }
        hipMemcpy(receive_data, receive_data_host, total_receive * sizeof(Entity), hipMemcpyHostToDevice);
        free(send_data_host);
        free(receive_data_host);
    }
    *size = total_receive;
    free(send_count_host);
    free(receive_count_host);
    free(send_displacements_host);
    free(receive_displacements_host);
    hipFree(send_count);
    hipFree(send_displacements);
    hipFree(send_displacements_temp);
    hipFree(send_data);
    return receive_data;
}

Entity *get_split_relation_sort_method(int rank, Entity *local_data_device,
                                       int row_size, int total_columns, int total_rank,
                                       int grid_size, int block_size, int cuda_aware_mpi, int *size) {

    thrust::device_vector <uint8_t> row_mapping(row_size);

    thrust::transform(
            thrust::device, local_data_device,
            local_data_device + row_size, row_mapping.begin(),
    [total_rank = total_rank] __device__(
    const Entity &entity) -> uint8_t{
            return (uint8_t)(entity.key % total_rank);
    });

    thrust::stable_sort_by_key(thrust::device, row_mapping.begin(), row_mapping.end(), local_data_device);

    thrust::device_vector<int> unique_rank_row_count(total_rank);
    thrust::device_vector <uint8_t> unique_rank(total_rank);

    auto unique_rank_range = thrust::reduce_by_key(
            thrust::device, row_mapping.begin(), row_mapping.end(),
            thrust::constant_iterator<int>(1), unique_rank.begin(),
            unique_rank_row_count.begin());
    auto total_unique_rank = unique_rank_range.first - unique_rank.begin();
    unique_rank_row_count.resize(total_unique_rank);
    unique_rank.resize(total_unique_rank);
    thrust::host_vector<int> unique_rank_row_count_host(unique_rank_row_count);
    thrust::host_vector <uint8_t> unique_rank_host(unique_rank);
    thrust::host_vector<int> send_count_host(total_rank);
    for (int i = 0; i < total_unique_rank; i++) {
        send_count_host[unique_rank_host[i]] = unique_rank_row_count_host[i];
    }
    thrust::host_vector<int> receive_count_host(total_rank);

    MPI_Alltoall(send_count_host.data(), 1, MPI_INT,
                 receive_count_host.data(), 1, MPI_INT, MPI_COMM_WORLD);
    int total_receive = thrust::reduce(receive_count_host.begin(), receive_count_host.end());

    thrust::host_vector<int> send_displacements_host(total_rank);
    thrust::host_vector<int> receive_displacements_host(total_rank);

    thrust::exclusive_scan(send_count_host.begin(), send_count_host.end(), send_displacements_host.begin());
    thrust::exclusive_scan(receive_count_host.begin(), receive_count_host.end(), receive_displacements_host.begin());

    Entity *receive_data;
    checkCuda(hipMalloc((void **) &receive_data, total_receive * sizeof(Entity)));
    int mpi_error;
    if (cuda_aware_mpi) {
        mpi_error = MPI_Alltoallv(local_data_device, send_count_host.data(), send_displacements_host.data(),
                                  MPI_UINT64_T,
                                  receive_data, receive_count_host.data(), receive_displacements_host.data(),
                                  MPI_UINT64_T,
                                  MPI_COMM_WORLD);
        if (mpi_error != MPI_SUCCESS) {
            char error_string[BUFSIZ];
            int length_of_error_string;
            MPI_Error_string(mpi_error, error_string, &length_of_error_string);
            fprintf(stderr, "MPI error on CUDA AWARE MPI MPI_Alltoallv call: %s\n", error_string);
            MPI_Abort(MPI_COMM_WORLD, mpi_error);
        }
    } else {
        Entity *send_data_host = (Entity *) malloc(row_size * sizeof(Entity));
        Entity *receive_data_host = (Entity *) malloc(total_receive * sizeof(Entity));
        hipMemcpy(send_data_host, local_data_device, row_size * sizeof(Entity), hipMemcpyDeviceToHost);
        mpi_error = MPI_Alltoallv(send_data_host, send_count_host.data(), send_displacements_host.data(),
                                  MPI_UINT64_T,
                                  receive_data_host, receive_count_host.data(), receive_displacements_host.data(),
                                  MPI_UINT64_T,
                                  MPI_COMM_WORLD);
        if (mpi_error != MPI_SUCCESS) {
            char error_string[BUFSIZ];
            int length_of_error_string;
            MPI_Error_string(mpi_error, error_string, &length_of_error_string);
            fprintf(stderr, "MPI error on host MPI_Alltoallv call: %s\n", error_string);
            MPI_Abort(MPI_COMM_WORLD, mpi_error);
        }
        hipMemcpy(receive_data, receive_data_host, total_receive * sizeof(Entity), hipMemcpyHostToDevice);
        free(send_data_host);
        free(receive_data_host);
    }
    *size = total_receive;
    return receive_data;
}

Entity *get_split_relation(int rank, Entity *local_data_device,
                           int row_size, int total_columns, int total_rank,
                           int grid_size, int block_size, int cuda_aware_mpi, int *size, int method) {
    if (method == 0) {
        return get_split_relation_pass_method(rank, local_data_device, row_size,
                                              total_columns, total_rank, grid_size, block_size, cuda_aware_mpi, size);
    } else {
        return get_split_relation_sort_method(rank, local_data_device, row_size,
                                              total_columns, total_rank, grid_size, block_size, cuda_aware_mpi, size);
    }
}

int main(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    double elapsed_time = -MPI_Wtime();
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    setlocale(LC_ALL, "");
    double max_time = 0.0;
    int total_rank, rank;
    int i;
    MPI_Comm_size(MPI_COMM_WORLD, &total_rank);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    // Should pass the input filename in command line argument
    const char *input_file;
    int comm_method = 0;
    int cuda_aware_mpi = 0;

    if (argc == 4) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
    } else if (argc == 3) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
    } else if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "hipc_2019.bin";
    }

    // READ THE FILE IN PARALLEL
    // Reading filesize in bytes
    struct stat filestats;
    stat(input_file, &filestats);
    off_t filesize = filestats.st_size;

    // Calculating the current rank's starting row and number of rows
    // Scatter larger blocks among processes (non-uniform)
    int total_columns = 2;
    int total_rows = filesize / (sizeof(int) * total_columns);
    int row_start = BLOCK_START(rank, total_rank, total_rows);
    int row_size = BLOCK_SIZE(rank, total_rank, total_rows);
    int local_count = row_size * total_columns;

    // Reading specific portion from the file as char in parallel
    int offset = row_start * total_columns * sizeof(int);
    int *local_data_host = (int *) malloc(local_count * sizeof(int));
    MPI_File mpi_file_buffer;
    if (MPI_File_open(MPI_COMM_WORLD, input_file, MPI_MODE_RDONLY,
                      MPI_INFO_NULL, &mpi_file_buffer) != MPI_SUCCESS) {
        printf("Error opening file %s", input_file);
        MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    }
    MPI_File_read_at(mpi_file_buffer, offset, local_data_host, local_count, MPI_INT, MPI_STATUS_IGNORE);
    MPI_File_close(&mpi_file_buffer);


    int *local_data_device;
    checkCuda(hipMalloc((void **) &local_data_device, local_count * sizeof(int)));
    hipMemcpy(local_data_device, local_data_host, local_count * sizeof(int), hipMemcpyHostToDevice);

    Entity *local_data;
    checkCuda(hipMalloc((void **) &local_data, row_size * sizeof(Entity)));
    Entity *local_data_reverse;
    checkCuda(hipMalloc((void **) &local_data_reverse, row_size * sizeof(Entity)));
    create_entity_ar<<<grid_size, block_size>>>(local_data, row_size, local_data_device);
    create_entity_ar_reverse<<<grid_size, block_size>>>(local_data_reverse, row_size, local_data_device);
    int input_relation_size = 0;
    Entity *input_relation = get_split_relation(rank, local_data,
                                                row_size, total_columns, total_rank,
                                                grid_size, block_size, cuda_aware_mpi,
                                                &input_relation_size, comm_method);

    int t_delta_size;
    Entity *t_delta = get_split_relation(rank, local_data_reverse,
                                         row_size, total_columns, total_rank,
                                         grid_size, block_size, cuda_aware_mpi, &t_delta_size, comm_method);
    thrust::stable_sort(thrust::device, t_delta, t_delta + t_delta_size, set_cmp());
    t_delta_size = (thrust::unique(thrust::device,
                                   t_delta, t_delta + t_delta_size,
                                   is_equal())) - t_delta;

    // T_FULL is t delta with first column as key
    Entity *t_full;
    checkCuda(hipMalloc((void **) &t_full, t_delta_size * sizeof(Entity)));
    hipMemcpy(t_full, t_delta, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);

    int global_t_full_size;
    int t_full_size = t_delta_size;
    MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);

    Entity *hash_table;
    double load_factor = 0.4;
    int hash_table_rows = (int) input_relation_size / load_factor;
    hash_table_rows = pow(2, ceil(log(hash_table_rows) / log(2)));
    checkCuda(hipMalloc((void **) &hash_table, hash_table_rows * sizeof(Entity)));
    Entity negative_entity;
    negative_entity.key = -1;
    negative_entity.value = -1;
    thrust::fill(thrust::device, hash_table, hash_table + hash_table_rows, negative_entity);
    build_hash_table_entity<<<grid_size, block_size>>>(hash_table, hash_table_rows, input_relation,
                                                       input_relation_size);
    int iterations = 0;


    while (true) {
        int join_result_size;
        int *join_offset;
        Entity *join_result;
        Entity *new_t_full;
        checkCuda(hipMalloc((void **) &join_offset, t_delta_size * sizeof(int)));
        checkCuda(hipMemset(join_offset, 0, t_delta_size * sizeof(int)));

        get_join_result_size_entity<<<grid_size, block_size>>>(hash_table, hash_table_rows,
                                                               t_delta, t_delta_size, join_offset);
        join_result_size = thrust::reduce(thrust::device, join_offset, join_offset + t_delta_size, 0);
        thrust::exclusive_scan(thrust::device, join_offset, join_offset + t_delta_size, join_offset);
        checkCuda(hipMalloc((void **) &join_result, join_result_size * sizeof(Entity)));

        get_join_result_entity<<<grid_size, block_size>>>(hash_table, hash_table_rows,
                                                          t_delta, t_delta_size, join_offset, join_result);

        // Scatter the new facts among relevant processes
        Entity *t_delta_temp = get_split_relation(rank, join_result,
                                                  join_result_size, total_columns, total_rank,
                                                  grid_size, block_size, cuda_aware_mpi, &t_delta_size,
                                                  comm_method);
        // Deduplicate scattered facts
        thrust::stable_sort(thrust::device, t_delta_temp, t_delta_temp + t_delta_size, set_cmp());
        t_delta_size = (thrust::unique(thrust::device,
                                       t_delta_temp, t_delta_temp + t_delta_size,
                                       is_equal())) - t_delta_temp;
        hipFree(t_delta);
        checkCuda(hipMalloc((void **) &t_delta, t_delta_size * sizeof(Entity)));
        hipMemcpy(t_delta, t_delta_temp, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);

        // set union of two sets (sorted t full and t delta)
        int new_t_full_size = t_delta_size + t_full_size;
        checkCuda(hipMalloc((void **) &new_t_full, new_t_full_size * sizeof(Entity)));
        new_t_full_size = thrust::set_union(thrust::device,
                                            t_full, t_full + t_full_size,
                                            t_delta, t_delta + t_delta_size,
                                            new_t_full, set_cmp()) - new_t_full;

        // Update t delta which is the only new facts which are not in t full and will be used in next iteration
        t_delta_size = thrust::set_difference(thrust::device,
                                              new_t_full, new_t_full + new_t_full_size,
                                              t_full, t_full + t_full_size,
                                              t_delta, set_cmp()) - t_delta;
        hipFree(t_full);
        checkCuda(hipMalloc((void **) &t_full, new_t_full_size * sizeof(Entity)));
        hipMemcpy(t_full, new_t_full, new_t_full_size * sizeof(Entity), hipMemcpyDeviceToDevice);
        t_full_size = new_t_full_size;
        hipFree(join_offset);
        hipFree(join_result);
        hipFree(new_t_full);
        hipFree(t_delta_temp);
        // Check if the global t full size has changed in this iteration
        int old_global_t_full_size = global_t_full_size;
        MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
        iterations++;
        if (old_global_t_full_size == global_t_full_size) {
            break;
        }
    }

    // Reverse the t_full as we stored it in reverse order initially
    int *t_full_ar;
    checkCuda(hipMalloc((void **) &t_full_ar, t_full_size * total_columns * sizeof(int)));
    reverse_t_full<<<grid_size, block_size>>>(t_full_ar, t_full_size, t_full);
    int *t_full_ar_host = (int *) malloc(t_full_size * total_columns * sizeof(int));
    hipMemcpy(t_full_ar_host, t_full_ar, t_full_size * total_columns * sizeof(int), hipMemcpyDeviceToHost);

    // List the t full counts for each process and calculate the displacements in the final result
    int *t_full_counts = (int *) calloc(total_rank, sizeof(int));
    MPI_Allgather(&t_full_size, 1, MPI_INT,
                  t_full_counts, 1, MPI_INT, MPI_COMM_WORLD);

    int *t_full_displacements = (int *) calloc(total_rank, sizeof(int));
    for (i = 1; i < total_rank; i++) {
        t_full_displacements[i] = t_full_displacements[i - 1] + (t_full_counts[i - 1] * total_columns);
    }

    // Write the t full to an offset of the output file
    MPI_File fh;
    string output_file = string(input_file) + "_tc.bin";
    const char *output_file_name = output_file.c_str();
    MPI_File_open(MPI_COMM_WORLD, output_file_name, MPI_MODE_WRONLY | MPI_MODE_CREATE, MPI_INFO_NULL, &fh);
    int file_offset = t_full_displacements[rank] * sizeof(int);
    MPI_File_write_at(fh, file_offset, t_full_ar_host, t_full_size * total_columns, MPI_INT, MPI_STATUS_IGNORE);
    // Close the file and clean up
    MPI_File_close(&fh);

    hipFree(local_data_device);
    hipFree(input_relation);
    hipFree(local_data);
    hipFree(local_data_reverse);
    hipFree(t_full);
    hipFree(t_delta);
    hipFree(t_full_ar);
    hipFree(hash_table);

    free(t_full_ar_host);
    free(t_full_counts);
    free(t_full_displacements);
    free(local_data_host);
    elapsed_time += MPI_Wtime();
    MPI_Allreduce(&elapsed_time, &max_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    if (rank == 0) {
        printf("\nGenerated file %s\n", output_file_name);
        printf("| # Input | # Process | # Iterations | # TC | Time (s) |\n");
        printf("| --- | --- | --- | --- | --- |\n");
        printf("| %'d | %'d | %'d | %'d | %'8.4lf |\n", total_rows, total_rank, iterations, global_t_full_size,
               max_time);
    }

    MPI_Finalize();
    return 0;
}
// METHOD 0 = two pass method, 1 = sorting method
// make runsemi DATA_FILE=data/data_23874.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=1
// make runsemi DATA_FILE=data/data_10.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runsemi DATA_FILE=data/data_23874.bin NPROCS=8 CUDA_AWARE_MPI=1 METHOD=1
// make runsemi DATA_FILE=data/data_147892.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
