
#include <hip/hip_runtime.h>
__host__ __device__ int get_rank(int key, int total_rank) {
    key ^= key >> 16;
    key *= 0x85ebca6b;
    key ^= key >> 13;
    key *= 0xc2b2ae35;
    key ^= key >> 16;
//    return key % total_rank;

    int total_buckets = 1025;
    int bucket_id = key % total_buckets;
    return bucket_id % total_rank;
}
