#include <stdio.h>
#include <string.h>
#include <mpi.h>

using namespace std;
int main(int argc, char *argv[])
{
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);

    int rank, tag=99;
    MPI_Status status;

    /* Initialize the MPI library */
    MPI_Init(&argc, &argv);
    /* Determine unique id of the calling process of all processes participating
       in this MPI program. This id is usually called MPI rank. */
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    int n = 20;
    int values[n];
    for(int i = 0; i < n; i++) {
        values[i] = i;
    }

    int *values_device;
    hipMalloc((void **) &values_device, n * sizeof(int));
    hipMemcpy(values_device, values, n * sizeof(int), hipMemcpyHostToDevice);

    if (rank == 0) {
//        MPI_Send(values, 20, MPI_INT, 1, tag, MPI_COMM_WORLD);
        MPI_Send(values_device, 20, MPI_INT, 1, tag, MPI_COMM_WORLD);
    } else {
//        int receive_values[20];
//        MPI_Recv(receive_values, 20, MPI_INT, 0, tag, MPI_COMM_WORLD, &status);
        int *receive_values = (int *) malloc(n * sizeof(int));;
        int *receive_values_device;
        hipMalloc((void **) &receive_values_device, n * sizeof(int));
        MPI_Recv(receive_values_device, 20, MPI_INT, 0, tag, MPI_COMM_WORLD, &status);
        hipMemcpy(receive_values, receive_values_device, n * sizeof(int), hipMemcpyDeviceToHost);
        printf("received: ");
        for(int i = 0; i < n; i++) {
            printf("%d ", receive_values[i]);
        }
        printf("\n");
    }

    /* Finalize the MPI library to free resources acquired by it. */
    MPI_Finalize();
    return 0;
}