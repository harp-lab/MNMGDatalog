#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <fstream>
#include <iostream>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/pair.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/unique.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/kernels.cu"

using namespace std;

int get_edge_count(const char *data_path) {
    std::ifstream f;
    f.open(data_path);
    char c;
    int i = 0;
    while (f.get(c)) {
        if (c == '\n') {
            i++;
        }
    }
    f.close();
    return i;
}

void *get_data_from_file(const char *file_path, int *data, int total_rows,
                         int total_columns, char separator) {
    FILE *data_file = fopen(file_path, "r");
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            if (j != (total_columns - 1)) {
                fscanf(data_file, "%d%c", &data[(i * total_columns) + j], &separator);
            } else {
                fscanf(data_file, "%d", &data[(i * total_columns) + j]);
            }
        }
    }
}

/**
 * @brief fnv1-a hash used in original slog backend
 *
 * @param start_ptr
 * @param prefix_len
 * @return __host__ __device__
 */
__host__ __device__ inline unsigned int prefix_hash(int *data, int start_index,
                                                    int prefix_len) {
    const unsigned int base = 2166136261U;
    const unsigned int prime = 16777619U;

    unsigned int hash = base;
    for (int i = 0; i < prefix_len; i++) {
        int chunk = data[start_index + i];
        hash ^= chunk & 255U;
        hash *= prime;
        for (int j = 0; j < 3; j++) {
            chunk = chunk >> 8;
            hash ^= chunk & 255U;
            hash *= prime;
        }
    }
aa    return hash;
}

__global__ void get_send_count(Entity *local_data, int local_data_row_count,
                               int *send_count, int nprocs) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= local_data_row_count) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < local_data_row_count; i += stride) {
        int key = local_data[i].key;
        int destination_rank = key % nprocs;
        atomicAdd(&send_count[destination_rank], 1);
    }
}

__global__ void get_rank_data(Entity *local_data, int local_data_row_count,
                              int *send_count_offset, int nprocs, Entity *rank_data) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= local_data_row_count) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < local_data_row_count; i += stride) {
        int key = local_data[i].key;
        int value = local_data[i].value;
        int destination_rank = key % nprocs;
        int current_position = atomicAdd(&send_count_offset[destination_rank], 1);
        rank_data[current_position].key = key;
        rank_data[current_position].value = value;
    }
}



//__global__
//void get_rank_data_size(int *local_data, int local_data_row_count,
//                        int *rank_data_size, int nprocs) {
//    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
//    if (index >= local_data_row_count) return;
//
//    int stride = blockDim.x * gridDim.x;
//
//    for (int i = index; i < local_data_row_count; i += stride) {
//        int key = local_data[i * 2];
//        int destination_rank = key % nprocs;
//        atomicAdd(&rank_data_size[destination_rank], 2);
//    }
//}
//
//__global__
//void get_rank_data(int *local_data, int local_data_row_count,
//                   int *rank_data_size, int nprocs, int *rank_data, int rank) {
//    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
//    if (index >= local_data_row_count) return;
//
//    int stride = blockDim.x * gridDim.x;
//
//    for (int i = index; i < local_data_row_count; i += stride) {
//        int key = local_data[i * 2];
//        int value = local_data[(i * 2) + 1];
//        int destination_rank = key % nprocs;
////        int current_position = rank_data_size[destination_rank];
//        int current_position = atomicAdd(&rank_data_size[destination_rank], 2);
////        if (rank == 0) {
////            printf("Hello from %d, key %d, value %d, destination rank %d, pos %d\n",
////                   i, key, value, destination_rank, current_position);
////        }
//        rank_data[current_position] = key;
//        rank_data[current_position + 1] = value;
////        atomicExch(&rank_data[current_position - 2], key);
////        atomicExch(&rank_data[current_position - 1], value);
////        atomicAdd(&rank_data_size[destination_rank], 2);
////        rank_data[current_position] = key;
////        rank_data[current_position + 1] = value;
////        atomicCAS(&rank_data[current_position], 0, key);
////        atomicCAS(&rank_data[current_position+1], 0, value);
//
////        atomicExch(&rank_data[current_position], key);
////        atomicExch(&rank_data[current_position + 1], value);
////        rank_data[rank_data_size[destination_rank] - 2] = key;
////        rank_data[rank_data_size[destination_rank] - 1] = value;
//    }
//}





//5 rows, 2 cols
//10 data local_data
//hash_values 5
//
//
//__global__ void get_split_relation_gpu(int *local_data, int *hash_values, int edge_count,
//                                       int total_columns, int rank) {
//    int id = threadIdx.x * threadIdx.y;
//    for (int i = 0; i < graph_edge_counts; i++) {
//        unsigned int hash_value = prefix_hash(local_data, id * 2, 1);
//        int hash_value_mod = hash_value % nprocs;
//        hash_values
//    }
//}


int main(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    setlocale(LC_ALL, "");
    double elapsed_time = -MPI_Wtime();
    double max_time = 0.0;
    int nprocs, rank;
    MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    int i, j, k;
    int total_columns = 2;
    // Should pass the input filename in command line argument
    const char *input_file;
    if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "data/hipc_2019.txt";
    }

    // load the raw graph
    int graph_edge_counts = get_edge_count(input_file);
    if (rank == 0) {
        std::cout << "Input graph edge counts: " << graph_edge_counts << std::endl;
    }
    int *raw_graph_data = (int *) malloc(graph_edge_counts * total_columns * sizeof(int));
    get_data_from_file(input_file, raw_graph_data, graph_edge_counts, total_columns, '\t');
    int *raw_reverse_graph_data = (int *) malloc(graph_edge_counts * total_columns * sizeof(int));

    for (i = 0; i < graph_edge_counts; i++) {
        raw_reverse_graph_data[i * 2 + 1] = raw_graph_data[i * total_columns];
        raw_reverse_graph_data[i * 2] = raw_graph_data[i * total_columns + 1];
    }

    Entity *entity_graph = (Entity *) malloc(graph_edge_counts * total_columns * sizeof(Entity));
    for (i = 0; i < graph_edge_counts; i++) {
        entity_graph[i].key = raw_graph_data[i * total_columns];
        entity_graph[i].value = raw_graph_data[i * total_columns + 1];
    }

    int row_size = graph_edge_counts;
    int cuda_aware_mpi = 0;

    Entity *local_data_device;
    checkCuda(hipMalloc((void **) &local_data_device, row_size * sizeof(Entity)));
    hipMemcpy(local_data_device, entity_graph, row_size * sizeof(Entity), hipMemcpyHostToDevice);

    int *send_count;
    checkCuda(hipMalloc((void **) &send_count, nprocs * sizeof(int)));
    int *send_displacements;
    checkCuda(hipMalloc((void **) &send_displacements, nprocs * sizeof(int)));
    int *send_displacements_temp;
    checkCuda(hipMalloc((void **) &send_displacements_temp, nprocs * sizeof(int)));
    get_send_count<<<grid_size, block_size>>>(local_data_device, row_size, send_count, nprocs);
//    int total_send = thrust::reduce(thrust::device, send_count, send_count + nprocs, 0, thrust::plus<int>());
    int total_send = row_size * total_columns;
    // calculate offset
    thrust::exclusive_scan(thrust::device, send_count, send_count + nprocs, send_displacements);
    hipMemcpy(send_displacements_temp, send_displacements, nprocs * sizeof(int), hipMemcpyDeviceToDevice);
    Entity *send_data;
    checkCuda(hipMalloc((void **) &send_data, total_send * sizeof(Entity)));
    get_rank_data<<<grid_size, block_size>>>(local_data_device, row_size, send_displacements_temp, nprocs, send_data);

    int *receive_count;
    checkCuda(hipMalloc((void **) &receive_count, nprocs * sizeof(int)));
    int *receive_displacements;
    checkCuda(hipMalloc((void **) &receive_displacements, nprocs * sizeof(int)));

    if (cuda_aware_mpi) {
        MPI_Alltoall(send_count, 1, MPI_INT, receive_count, 1, MPI_INT, MPI_COMM_WORLD);
    } else {
        int *send_count_host = (int *) malloc(nprocs * sizeof(int));;
        int *receive_count_host = (int *) malloc(nprocs * sizeof(int));;
        hipMemcpy(send_count_host, send_count, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        MPI_Alltoall(send_count_host, 1, MPI_INT, receive_count_host, 1, MPI_INT, MPI_COMM_WORLD);
        hipMemcpy(receive_count, receive_count_host, nprocs * sizeof(int), hipMemcpyHostToDevice);
    }

    int total_receive = thrust::reduce(thrust::device, receive_count, receive_count + nprocs, 0, thrust::plus<int>());
    thrust::exclusive_scan(thrust::device, receive_count, receive_count + nprocs, receive_displacements);
    Entity *receive_data;
    checkCuda(hipMalloc((void **) &receive_data, total_receive * sizeof(Entity)));
    if (cuda_aware_mpi) {
        MPI_Alltoallv(send_data, send_count, send_displacements, MPI_UINT64_T,
                      receive_data, receive_count, receive_displacements, MPI_UINT64_T,
                      MPI_COMM_WORLD);
    } else {
        int *send_count_host = (int *) malloc(nprocs * sizeof(int));;
        int *receive_count_host = (int *) malloc(nprocs * sizeof(int));;
        int *send_displacements_host = (int *) malloc(nprocs * sizeof(int));;
        int *receive_displacements_host = (int *) malloc(nprocs * sizeof(int));;
        Entity *send_data_host = (Entity *) malloc(row_size * total_columns * sizeof(Entity));;
        Entity *receive_data_host = (Entity *) malloc(total_receive * sizeof(Entity));;
        hipMemcpy(send_count_host, send_count, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(receive_count_host, receive_count, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(send_displacements_host, send_displacements, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(receive_displacements_host, receive_displacements, nprocs * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(send_data_host, send_data, total_send * sizeof(Entity), hipMemcpyDeviceToHost);
        MPI_Alltoallv(send_data_host, send_count_host, send_displacements_host, MPI_UINT64_T,
                      receive_data_host, receive_count_host, receive_displacements_host, MPI_UINT64_T,
                      MPI_COMM_WORLD);
        hipMemcpy(receive_data, receive_data_host, total_receive * sizeof(Entity), hipMemcpyHostToDevice);
        show_variable_entity(receive_data_host, total_receive, rank, "receive data");
    }

//    int *send_count;
//    checkCuda(hipMalloc((void **) &send_count, nprocs * sizeof(int)));
//    get_rank_data_size<<<grid_size, block_size>>>(device_entity_graph, graph_edge_counts, send_count, nprocs);
//    // calculate offset
//    thrust::exclusive_scan(thrust::device, rank_data_size, rank_data_size + nprocs, rank_data_size);
//
//    int *rank_data;
//    checkCuda(hipMalloc((void **) &rank_data, graph_edge_counts * total_columns * sizeof(int)));
//    get_rank_data<<<grid_size, block_size>>>(device_graph, graph_edge_counts, rank_data_size, nprocs, rank_data, rank);




//    int *device_entity_graph;
//    checkCuda(hipMalloc((void **) &device_entity_graph, graph_edge_counts * sizeof(Entity)));
//    hipMemcpy(device_entity_graph, entity_graph, graph_edge_counts * sizeof(Entity),
//               hipMemcpyHostToDevice);

//    int *device_graph;
//    checkCuda(hipMalloc((void **) &device_graph, graph_edge_counts * total_columns * sizeof(int)));
//    hipMemcpy(device_graph, raw_graph_data, graph_edge_counts * total_columns * sizeof(int),
//               hipMemcpyHostToDevice);
//
//    int *rank_data_size;
//    checkCuda(hipMalloc((void **) &rank_data_size, nprocs * sizeof(int)));
//    get_rank_data_size<<<grid_size, block_size>>>(device_graph, graph_edge_counts, rank_data_size, nprocs);
//    checkCuda(hipDeviceSynchronize());
//    // calculate offset
//    thrust::exclusive_scan(thrust::device, rank_data_size, rank_data_size + nprocs, rank_data_size);
//    checkCuda(hipDeviceSynchronize());
//
//    int *rank_data;
//    checkCuda(hipMalloc((void **) &rank_data, graph_edge_counts * total_columns * sizeof(int)));
//    get_rank_data<<<grid_size, block_size>>>(device_graph, graph_edge_counts, rank_data_size, nprocs, rank_data, rank);
//    checkCuda(hipDeviceSynchronize());
//
//
//    int *rank_data_size_host = (int *) malloc(nprocs * sizeof(int));
//    hipMemcpy(rank_data_size_host, rank_data_size, nprocs * sizeof(int),
//               hipMemcpyDeviceToHost);
//
//    int *host_graph = (int *) malloc(graph_edge_counts * total_columns * sizeof(int));
//    hipMemcpy(host_graph, device_graph, graph_edge_counts * total_columns * sizeof(int),
//               hipMemcpyDeviceToHost);
//    int *rank_data_host = (int *) malloc(graph_edge_counts * total_columns * sizeof(int));
//    hipMemcpy(rank_data_host, rank_data, graph_edge_counts * total_columns * sizeof(int),
//               hipMemcpyDeviceToHost);
//
//    if (rank == 0) {
//        cout << "----------------" << endl;
//        for (i = 0; i < graph_edge_counts; i++) {
//            cout << "Rank: " << rank << ", " << rank_data_host[i * 2] << " " << rank_data_host[(i * 2) + 1] << endl;
//        }
//    }


//    for (int i = 0; i < graph_edge_counts; i++) {
//        unsigned int hash_value = prefix_hash(raw_graph_data, i * 2, 1);
//        int hash_value_mod = hash_value % nprocs;
//        if (rank == 0) {
//            cout << "Rank " << rank << " key: " << raw_graph_data[i * 2] << " hash_value: " << hash_value
//                 << ", after mod: " << hash_value_mod << endl;
//        }
//    }


    free(raw_graph_data);
    free(raw_reverse_graph_data);
//    hipFree(device_graph);
    MPI_Finalize();
    return 0;
}

// make runcomm DATA_FILE=data/data_10.txt NPROCS=8