#include "hip/hip_runtime.h"
__global__
void get_join_result_size_entity(Entity *hash_table, int hash_table_size,
                                 Entity *t_delta, int t_delta_size,
                                 int *join_result_size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= t_delta_size) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < t_delta_size; i += stride) {
        int key = t_delta[i].key;
        int current_size = 0;
        int position = get_position(key, hash_table_size);
        while (true) {
            if (hash_table[position].key == key) {
                current_size++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_size - 1);
        }
        join_result_size[i] = current_size;
    }
}

__global__
void get_join_result_entity(Entity *hash_table, int hash_table_size,
                            Entity *t_delta, int t_delta_size, int *offset, Entity *join_result) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= t_delta_size) return;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < t_delta_size; i += stride) {
        int key = t_delta[i].key;
        int value = t_delta[i].value;
        int start_index = offset[i];
        int position = get_position(key, hash_table_size);
        printf("i:%d, start_index: %d, position: %d\n", i, start_index, position);
        while (true) {
            if (hash_table[position].key == key) {
                join_result[start_index].key = hash_table[position].value;
                join_result[start_index].value = value;
                start_index++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_size - 1);
        }
    }
}

__global__
void get_join_result_entity_with_rank(Entity *hash_table, int hash_table_size,
                            Entity *t_delta, int t_delta_size, int *offset, Entity *join_result, int join_result_size, int rank) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= t_delta_size) return;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < t_delta_size; i += stride) {
        // if (rank == 1) {
        //     printf(">>>>>>>>>>>>>>>> i: %d, t_delta_size %d \n", i, t_delta_size);
        // }
        if (i >= t_delta_size) {
            printf("rank: %d, i:%d, t_delta_size: %d\n", rank, i, t_delta_size);
        }
        int key = t_delta[i].key;
        int value = t_delta[i].value;
        int start_index = offset[i];
        int position = get_position(key, hash_table_size);
        // if (i >= t_delta_size || start_index >= join_result_size || position >= hash_table_size) {
        //     printf("rank: %d, i:%d, start_index: %d, position: %d\n", rank, i, start_index, position);
        // }
        while (true) {
            if (hash_table[position].key == key) {
                join_result[start_index].key = hash_table[position].value;
                join_result[start_index].value = value;
                start_index++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_size - 1);
        }
    }
}


Entity *get_join(int grid_size, int block_size, Entity *hash_table, int hash_table_size, Entity *relation,
                 int relation_size, int *join_result_size, double *compute_time) {
//    show_variable_generic(hash_table, "hash_table", hash_table_size, "Entity", "device", 0, 0, "hash table in get join", 0);
//    show_variable_generic(relation, "relation", relation_size, "Entity", "device", 0, 0, "", 0);
    double start_time, end_time, elapsed_time;
    start_time = MPI_Wtime();
    Entity *join_result = nullptr;
    if (hash_table_size == 0) {
//        checkCuda(hipMalloc((void **) &join_result, 0));
        *join_result_size = 0;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        *compute_time = elapsed_time;
        return join_result;
    }

    // get current mpi rank
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    int result_size;
    int *join_offset;
    checkCuda(hipMalloc((void **) &join_offset, relation_size * sizeof(int)));
    checkCuda(hipMemset(join_offset, 0, relation_size * sizeof(int)));

    get_join_result_size_entity<<<grid_size, block_size>>>(hash_table, hash_table_size,
                                                           relation, relation_size, join_offset);
    checkCuda(hipDeviceSynchronize());
    result_size = thrust::reduce(thrust::device, join_offset, join_offset + relation_size, 0, thrust::plus<int>());
    int *join_offset_host;
    join_offset_host = (int *) malloc(relation_size * sizeof(int));
    checkCuda(hipMemcpy(join_offset_host, join_offset, relation_size * sizeof(int), hipMemcpyDeviceToHost));
    int totol_reduce_host = join_offset_host[0];
    for (int i = 1; i < relation_size; i++) {
        totol_reduce_host += join_offset_host[i];
        join_offset_host[i] += join_offset_host[i - 1];
    }
    std::cout << "Ranck : " << rank  << " Scanned result : " << join_offset_host[relation_size - 1] << std::endl;
    int step = 90'000'000;
    for (int i = 0; i < relation_size; i += step) {
        int end = i + step;
        if (i + step >= relation_size) {
            end = relation_size - i;
        }
        thrust::exclusive_scan(thrust::device, join_offset + i, join_offset + end, join_offset + i);
    }
    // thrust::exclusive_scan(thrust::device, join_offset, join_offset + relation_size, join_offset);
    thrust::for_each(thrust::device, join_offset, join_offset + relation_size,
        [] __device__(int &x) {
            if (x < 0) {
                printf("x: %d\n", x);
            }
        });
    
#ifdef DEBUG
    std::cout << "rank : " << rank << " >>>>>>>>>>>>>> Join counter: " << debug::debug_counter << std::endl;
    std::cout << "rank : " << rank << ">>>>>>>>>>>>>> Result Size: " << result_size << std::endl;
    std::cout << "rank : " << rank << ">>>>>>>>>>>>>> Relation Size: " << relation_size << std::endl;
    std::cout << "rank : " << rank << ">>>>>>>>>>>>>> Hash Table Size: " << hash_table_size << std::endl;
    std::cout << "rank : " << rank << "result_size * sizeof(Entity): " << result_size * sizeof(Entity) << endl;
    std::cout << "rank : " << rank  << ">>>>>>>>>>>>>> Total Reduce Host: " << totol_reduce_host << std::endl;
#endif

    checkCuda(hipMalloc((void **) &join_result, result_size * sizeof(Entity)));
    get_join_result_entity_with_rank<<<grid_size, block_size>>>(hash_table, hash_table_size,
                                                      relation, relation_size, join_offset, join_result, result_size, rank);
    checkCuda(hipDeviceSynchronize());
    checkCuda(hipFree(join_offset));
#ifdef DEBUG
    debug::debug_counter = debug::debug_counter + 1;
#endif
    *join_result_size = result_size;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    *compute_time = elapsed_time;
    return join_result;
}