#include "hip/hip_runtime.h"
__global__
void get_join_result_size_entity(Entity *hash_table, long int hash_table_size,
                                 Entity *t_delta, long int t_delta_size,
                                 int *join_result_size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= t_delta_size) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < t_delta_size; i += stride) {
        int key = t_delta[i].key;
        int current_size = 0;
        int position = get_position(key, hash_table_size);
        while (true) {
            if (hash_table[position].key == key) {
                current_size++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_size - 1);
        }
        join_result_size[i] = current_size;
    }
}

__global__
void get_join_result_entity(Entity *hash_table, int hash_table_size,
                            Entity *t_delta, int t_delta_size, int *offset, Entity *join_result) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= t_delta_size) return;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < t_delta_size; i += stride) {
        int key = t_delta[i].key;
        int value = t_delta[i].value;
        int start_index = offset[i];
        int position = get_position(key, hash_table_size);
        while (true) {
            if (hash_table[position].key == key) {
                join_result[start_index].key = hash_table[position].value;
                join_result[start_index].value = value;
                start_index++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_size - 1);
        }
    }
}


Entity *get_join(int grid_size, int block_size, Entity *hash_table, int hash_table_size, Entity *relation,
                 int relation_size, int *join_result_size, double *compute_time) {
    double start_time, end_time, elapsed_time;
    start_time = MPI_Wtime();
    if (hash_table_size == 0) {
        Entity *join_result;
        checkCuda(hipMalloc((void **) &join_result, 0 * sizeof(Entity)));
        *join_result_size = 0;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        *compute_time = elapsed_time;
        return join_result;
    }
    int result_size;
    int *join_offset;
    Entity *join_result;
    checkCuda(hipMalloc((void **) &join_offset, relation_size * sizeof(int)));
    checkCuda(hipMemset(join_offset, 0, relation_size * sizeof(int)));

    get_join_result_size_entity<<<grid_size, block_size>>>(hash_table, hash_table_size,
                                                           relation, relation_size, join_offset);
    result_size = thrust::reduce(thrust::device, join_offset, join_offset + relation_size, 0, thrust::plus<int>());
    thrust::exclusive_scan(thrust::device, join_offset, join_offset + relation_size, join_offset);
    checkCuda(hipMalloc((void **) &join_result, result_size * sizeof(Entity)));
    get_join_result_entity<<<grid_size, block_size>>>(hash_table, hash_table_size,
                                                      relation, relation_size, join_offset, join_result);
    hipFree(join_offset);
    *join_result_size = result_size;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    *compute_time = elapsed_time;
    return join_result;
}


__global__
void get_join_result_size_with_reverse_entity(Entity *hash_table, long int hash_table_size,
                                              Entity *t_delta, long int t_delta_size,
                                              int *join_result_size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= t_delta_size) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < t_delta_size; i += stride) {
        int key = t_delta[i].key;
        int current_size = 0;
        int position = get_position(key, hash_table_size);
        while (true) {
            if (hash_table[position].key == key) {
                current_size += 2;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_size - 1);
        }
        join_result_size[i] = current_size;
    }
}

__global__
void get_join_result_with_reverse_entity(Entity *hash_table, int hash_table_size,
                                         Entity *t_delta, int t_delta_size, int *offset, Entity *join_result) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= t_delta_size) return;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < t_delta_size; i += stride) {
        int key = t_delta[i].key;
        int value = t_delta[i].value;
        int start_index = offset[i];
        int position = get_position(key, hash_table_size);
        while (true) {
            if (hash_table[position].key == key) {
                join_result[start_index].key = hash_table[position].value;
                join_result[start_index].value = value;
                join_result[start_index + 1].key = value;
                join_result[start_index + 1].value = hash_table[position].value;
                start_index++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_size - 1);
        }
    }
}


Entity *get_join_with_reverse(int grid_size, int block_size, Entity *hash_table, int hash_table_size, Entity *relation,
                              int relation_size, int *join_result_size, double *compute_time) {
    double start_time, end_time, elapsed_time;
    start_time = MPI_Wtime();
    if (hash_table_size == 0) {
        Entity *join_result;
        checkCuda(hipMalloc((void **) &join_result, 0 * sizeof(Entity)));
        *join_result_size = 0;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        *compute_time = elapsed_time;
        return join_result;
    }
    int result_size;
    int *join_offset;
    Entity *join_result;
    checkCuda(hipMalloc((void **) &join_offset, relation_size * sizeof(int)));
    checkCuda(hipMemset(join_offset, 0, relation_size * sizeof(int)));

    get_join_result_size_entity<<<grid_size, block_size>>>(hash_table, hash_table_size,
                                                           relation, relation_size, join_offset);
    result_size = thrust::reduce(thrust::device, join_offset, join_offset + relation_size, 0, thrust::plus<int>());
    thrust::exclusive_scan(thrust::device, join_offset, join_offset + relation_size, join_offset);
    checkCuda(hipMalloc((void **) &join_result, result_size * sizeof(Entity)));
    get_join_result_entity<<<grid_size, block_size>>>(hash_table, hash_table_size,
                                                      relation, relation_size, join_offset, join_result);
    hipFree(join_offset);
    *join_result_size = result_size;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    *compute_time = elapsed_time;
    return join_result;
}