#include "hip/hip_runtime.h"
__global__
void get_join_result_size_entity(Entity *hash_table, int hash_table_size,
                                 Entity *t_delta, int t_delta_size,
                                 int *join_result_size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= t_delta_size) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < t_delta_size; i += stride) {
        int key = t_delta[i].key;
        int current_size = 0;
        int position = get_position(key, hash_table_size);
        while (true) {
            if (hash_table[position].key == key) {
                current_size++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_size - 1);
        }
        join_result_size[i] = current_size;
    }
}

__global__
void get_join_result_entity(Entity *hash_table, int hash_table_size,
                            Entity *t_delta, int t_delta_size, int *offset, Entity *join_result) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= t_delta_size) return;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < t_delta_size; i += stride) {
        int key = t_delta[i].key;
        int value = t_delta[i].value;
        int start_index = offset[i];
        int position = get_position(key, hash_table_size);
        while (true) {
            if (hash_table[position].key == key) {
                join_result[start_index].key = hash_table[position].value;
                join_result[start_index].value = value;
                start_index++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_size - 1);
        }
    }
}


Entity *get_join(int grid_size, int block_size, Entity *hash_table, int hash_table_size, Entity *relation,
                 int relation_size, int *join_result_size, double *compute_time) {
    double start_time, end_time, elapsed_time;
    start_time = MPI_Wtime();
    Entity *join_result = nullptr;
    if (hash_table_size == 0) {
        *join_result_size = 0;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        *compute_time = elapsed_time;
        return join_result;
    }
    int result_size;
    int *join_offset;
    checkCuda(hipMalloc((void **) &join_offset, relation_size * sizeof(int)));
    checkCuda(hipMemset(join_offset, 0, relation_size * sizeof(int)));

    get_join_result_size_entity<<<grid_size, block_size>>>(hash_table, hash_table_size,
                                                           relation, relation_size, join_offset);
    checkCuda(hipDeviceSynchronize());
    result_size = thrust::reduce(thrust::device, join_offset, join_offset + relation_size, 0, thrust::plus<int>());
//    int step = 90'000'000;
//    for (int i = 0; i < relation_size; i += step) {
//        int end = i + step;
//        if (end > relation_size) {
//            end = relation_size;
//        }
//        thrust::exclusive_scan(thrust::device, join_offset + i, join_offset + end, join_offset + i);
//        checkCuda(hipDeviceSynchronize());
//    }
    thrust::exclusive_scan(thrust::device, join_offset, join_offset + relation_size, join_offset);
#ifdef DEBUG
    cout << "result_size * sizeof(Entity): " << result_size * sizeof(Entity) << endl;
#endif
//    checkCuda(hipDeviceSynchronize());
    checkCuda(hipMalloc((void **) &join_result, result_size * sizeof(Entity)));
    get_join_result_entity<<<grid_size, block_size>>>(hash_table, hash_table_size,
                                                      relation, relation_size, join_offset, join_result);
//    checkCuda(hipDeviceSynchronize());
    hipFree(join_offset);
    *join_result_size = result_size;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    *compute_time = elapsed_time;
    return join_result;
}