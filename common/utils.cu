#include "hip/hip_runtime.h"
#include <string>
#include <chrono>
#include <stdlib.h>

//using namespace std;


struct Entity {
    int key;
    int value;
};

struct Output {
    int block_size;
    int grid_size;
    int total_rank;
    int input_rows;
    int hashtable_rows;
    int iterations;
    long long output_size;
    long long output_size_secondary;
    double load_factor;
    double reverse_time;
    int hashtable_build_rate;
    const char *dataset_name;
    const char *output_file_name;
    double total_time;
    double initialization_time;
    double fileio_time;
    double hashtable_build_time;
    double join_time;
    double buffer_preparation_time;
    double communication_time;
    double merge_time;
    double deduplication_time;
    double finalization_time;
};

struct KernelTimer {
    hipEvent_t start;
    hipEvent_t stop;

    KernelTimer() {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~KernelTimer() {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void start_timer() {
        hipEventRecord(start, 0);
    }

    void stop_timer() {
        hipEventRecord(stop, 0);
    }

    float get_spent_time() {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        elapsed /= 1000.0;
        return elapsed;
    }
};

struct is_equal {
    __host__ __device__
    bool operator()(const Entity &lhs, const Entity &rhs) {
        if ((lhs.key == rhs.key) && (lhs.value == rhs.value))
            return true;
        return false;
    }
};

struct is_equal_key {
    __host__ __device__
    bool operator()(const Entity &lhs, const Entity &rhs) {
        if (lhs.key == rhs.key)
            return true;
        return false;
    }
};

// Predicate to check if key and value are equal
struct is_key_equal_value {
    __host__ __device__
    bool operator()(const Entity &e) {
        return e.key == e.value;
    }
};


struct cmp {
    __host__ __device__
    bool operator()(const Entity &lhs, const Entity &rhs) {
        if (lhs.key < rhs.key)
            return true;
        else if (lhs.key > rhs.key)
            return false;
        else {
            if (lhs.value < rhs.value)
                return true;
            else if (lhs.value > rhs.value)
                return false;
            return true;
        }
    }
};

struct set_cmp {
    __host__ __device__
    bool operator()(const Entity &lhs, const Entity &rhs) {
        if (lhs.key == rhs.key) {
            // If keys are equal, compare values
            return lhs.value < rhs.value;
        }
        return lhs.key < rhs.key;
    }
};

struct minimum_by_value {
    __host__ __device__
    Entity operator()(const Entity &a, const Entity &b) const {
        return (a.value < b.value) ? a : b;
    }
};

struct minimum_value {
    __host__ __device__
    int operator()(const int a, const int b) const {
        return (a < b) ? a : b;
    }
};

// Define a unary operation that extracts the key
struct get_key {
    __host__ __device__
    int operator()(const Entity &e) const {
        return e.key;
    }
};


__device__ int get_position(int key, int hash_table_row_size) {
    key ^= key >> 16;
    key *= 0x85ebca6b;
    key ^= key >> 13;
    key *= 0xc2b2ae35;
    key ^= key >> 16;
    return key & (hash_table_row_size - 1);
}

void show_time_spent(std::string message,
                     std::chrono::high_resolution_clock::time_point time_point_begin,
                     std::chrono::high_resolution_clock::time_point time_point_end) {
    std::chrono::duration<double> time_span = time_point_end - time_point_begin;
    std::cout << message << ": " << time_span.count() << " seconds" << std::endl;
}

double get_time_spent(std::string message,
                      std::chrono::high_resolution_clock::time_point time_point_begin,
                      std::chrono::high_resolution_clock::time_point time_point_end) {
    std::chrono::duration<double> time_span = time_point_end - time_point_begin;
    if (message != "")
        std::cout << message << ": " << time_span.count() << " seconds" << std::endl;
    return time_span.count();
}

void show_relation(int *data, int total_rows,
                   int total_columns, const char *relation_name,
                   int visible_rows, int skip_zero) {
    int count = 0;
    std::cout << "Relation name: " << relation_name << std::endl;
    std::cout << "===================================" << std::endl;
    for (int i = 0; i < total_rows; i++) {
        int skip = 0;
        for (int j = 0; j < total_columns; j++) {
            if ((skip_zero == 1) && (data[(i * total_columns) + j] == 0)) {
                skip = 1;
                continue;
            }
            std::cout << data[(i * total_columns) + j] << " ";
        }
        if (skip == 1)
            continue;
        std::cout << std::endl;
        count++;
        if (count == visible_rows) {
            std::cout << "Result cropped at row " << count << "\n" << std::endl;
            return;
        }

    }
    std::cout << "Result counts " << count << "\n" << std::endl;
    std::cout << "" << std::endl;
}

int *get_relation_from_file(const char *file_path, int total_rows, int total_columns, char separator) {
    int *data = (int *) malloc(total_rows * total_columns * sizeof(int));
    FILE *data_file = fopen(file_path, "r");
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            if (j != (total_columns - 1)) {
                int tmp = fscanf(data_file, "%d%c", &data[(i * total_columns) + j], &separator);
            } else {
                int tmp = fscanf(data_file, "%d", &data[(i * total_columns) + j]);
            }
        }
    }
    return data;
}

void get_relation_from_file_gpu(int *data, const char *file_path, int total_rows, int total_columns, char separator) {
    FILE *data_file = fopen(file_path, "r");
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            if (j != (total_columns - 1)) {
                int tmp = fscanf(data_file, "%d%c", &data[(i * total_columns) + j], &separator);
            } else {
                int tmp = fscanf(data_file, "%d", &data[(i * total_columns) + j]);
            }
        }
    }
}


void get_random_relation(int *data, int total_rows, int total_columns) {
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            data[(i * total_columns) + j] = (rand() % (32767 - 0 + 1)) + 0;
        }
    }
}

void get_string_relation(int *data, int total_rows, int total_columns) {
    int x = 1, y = 2;
    for (int i = 0; i < total_rows; i++) {
        data[(i * total_columns) + 0] = x++;
        data[(i * total_columns) + 1] = y++;
    }
}

void get_reverse_relation_gpu(int *reverse_data, int *data, int total_rows, int total_columns) {
    for (int i = 0; i < total_rows; i++) {
        int pos = total_columns - 1;
        for (int j = 0; j < total_columns; j++) {
            reverse_data[(i * total_columns) + j] = data[(i * total_columns) + pos];
            pos--;
        }
    }
}


void show_hash_table(Entity *hash_table, long int hash_table_row_size, const char *hash_table_name) {
    int count = 0;
    std::cout << "Hashtable name: " << hash_table_name << std::endl;
    std::cout << "===================================" << std::endl;
    for (int i = 0; i < hash_table_row_size; i++) {
        if (hash_table[i].key != -1) {
            std::cout << hash_table[i].key << " " << hash_table[i].value << std::endl;
            count++;
        }
    }
    std::cout << "Row counts " << count << "\n" << std::endl;
    std::cout << "" << std::endl;
}

void show_entity_array(Entity *data, int data_rows, const char *array_name) {
    long int count = 0;
    std::cout << "Entity name: " << array_name << std::endl;
    std::cout << "===================================" << std::endl;
    for (int i = 0; i < data_rows; i++) {
        if (data[i].key != -1) {
            std::cout << data[i].key << " " << data[i].value << std::endl;
            count++;
        }
    }
    std::cout << "Row counts " << count << "\n" << std::endl;
    std::cout << "" << std::endl;
}

long int get_row_size(const char *data_path) {
    long int row_size = 0;
    int base = 1;
    for (int i = strlen(data_path) - 1; i >= 0; i--) {
        if (isdigit(data_path[i])) {
            int digit = (int) data_path[i] - '0';
            row_size += base * digit;
            base *= 10;
        }
    }
    return row_size;
}

void update_reverse_relation(Entity *data, int data_rows, int *reverse_relation) {
    for (int i = 0; i < data_rows; i++) {
        reverse_relation[i * 2] = data[i].key;
        reverse_relation[(i * 2) + 1] = data[i].value;
    }
}

void show_variable(int *host_data, int data_size, int group, int rank, std::string message) {
    std::cout << "Rank " << rank << ": " << message << " ----------------" << std::endl;
    for (int i = 0; i < data_size / group; i++) {
        for (int j = 0; j < group; j++) {
            std::cout << host_data[(i * group) + j] << " ";
        }
        if (data_size <= 20) {
            std::cout << ", ";
        } else {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
}

void show_variable_entity(Entity *host_data, int data_size, int rank, std::string message) {
    std::cout << "Rank " << rank << ", size " << data_size << " : " << message << " ----------------" << std::endl;
    for (int i = 0; i < data_size; i++) {
        std::cout << host_data[i].key << " " << host_data[i].value;
        if (data_size <= 20) {
            std::cout << ", ";
        } else {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
}

void show_device_variable(int *device_data, int device_data_size, int group, int rank, std::string message, int size_only) {
    int *host_data = (int *) malloc(device_data_size * sizeof(int));
    hipMemcpy(host_data, device_data, device_data_size * sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Rank " << rank << ", size " << device_data_size << " : " << message << " ----------------" << std::endl;
    if (size_only != 1) {
        for (int i = 0; i < device_data_size / group; i++) {
            for (int j = 0; j < group; j++) {
                std::cout << host_data[(i * group) + j] << " ";
            }
            if (device_data_size <= 20) {
                std::cout << ", ";
            } else {
                std::cout << std::endl;
            }
        }
        std::cout << std::endl;
    }
    free(host_data);
}

void show_host_vector(const thrust::host_vector<int> &host_vector_data,
                      int group, int rank, const std::string &message, int size_only) {
    std::cout << "Rank " << rank << ", size " << host_vector_data.size()
              << " : " << message << " ----------------" << std::endl;

    if (size_only != 1) {
        for (size_t i = 0; i < host_vector_data.size() / group; i++) {
            for (int j = 0; j < group; j++) {
                std::cout << host_vector_data[(i * group) + j] << " ";
            }
            if (host_vector_data.size() < group) {
                std::cout << ", ";
            } else {
                std::cout << std::endl;
            }
        }
        std::cout << std::endl;
    }
}

void show_host_variable(int *host_data, int data_size, int group, int rank, std::string message, int size_only) {
    std::cout << "Rank " << rank << ", size " << data_size << " : " << message << " ----------------" << std::endl;
    if (size_only != 1) {
        for (int i = 0; i < data_size / group; i++) {
            for (int j = 0; j < group; j++) {
                std::cout << host_data[(i * group) + j] << " ";
            }
            if (data_size <= 20) {
                std::cout << ", ";
            } else {
                std::cout << std::endl;
            }
        }
        std::cout << std::endl;
    }
}

// show_device_entity_variable(hash_table, hash_table_rows, rank, "hash_table");
void show_device_entity_variable(Entity *device_data, int device_data_size, int rank, std::string message, int size_only) {
    Entity *host_data = (Entity *) malloc(device_data_size * sizeof(Entity));
    hipMemcpy(host_data, device_data, device_data_size * sizeof(Entity), hipMemcpyDeviceToHost);
    std::cout << "Rank " << rank << ", size " << device_data_size << " : " << message << " ----------------" << std::endl;
    if (size_only != 1) {
        for (int i = 0; i < device_data_size; i++) {
            std::cout << host_data[i].key << " " << host_data[i].value << std::endl;
        }
        std::cout << std::endl;
    }
    free(host_data);
}

// Function to print variable details and data
void show_variable_generic(void *data, std::string var_name, size_t data_size, std::string data_type,
                           std::string execution_policy, int rank, int iteration,
                           std::string message, int size_only) {
    std::cout << "Rank: " << rank << ", iteration: " << iteration << ", " << var_name << "(" << execution_policy << ")"
         << " size: " << data_size << " : " << message << " ----------------" << std::endl;
    if (size_only == 1) return;
    if (execution_policy == "device") {
        if (data_type == "Entity") {
            Entity *host_data = (Entity *) malloc(data_size * sizeof(Entity));
            hipMemcpy(host_data, data, data_size * sizeof(Entity), hipMemcpyDeviceToHost);
            for (int i = 0; i < data_size; i++) {
                std::cout << host_data[i].key << " " << host_data[i].value << std::endl;
            }
            std::cout << std::endl;
            free(host_data);
        } else {
            int *host_data = (int *) malloc(data_size * sizeof(int));
            hipMemcpy(host_data, data, data_size * sizeof(int), hipMemcpyDeviceToHost);
            for (int i = 0; i < data_size; i++) {
                std::cout << host_data[i] << std::endl;
            }
            std::cout << std::endl;
            free(host_data);
        }
    } else {
        if (data_type == "Entity") {
            Entity *entity_data = static_cast<Entity *>(data);
            for (int i = 0; i < data_size; i++) {
                std::cout << entity_data[i].key << " " << entity_data[i].value << std::endl;
            }
        } else {
            int *int_data = static_cast<int *>(data);
            for (int i = 0; i < data_size; i++) {
                std::cout << int_data[i] << std::endl;
            }
            std::cout << std::endl;
        }
    }
}

std::tuple<double, double, double> calculate_load_metrics(int array_size, int total_rank) {
    // Function to Calculate Load Imbalance Ratio (LIR) and Coefficient of Variation (CV) based on array size
    // LIR = (max_size - min_size) / mean_size,
    // CV = std dev / mean_size
    // Max/min ratio = max_size / min_size
    // LIR near 0: Indicates good load balance, as the difference between max and min loads is minimal.
    // CV: The smaller the CV, the better the load balance. Typically, a CV below 0.1 (10%) suggests reasonable balance, while a CV close to 0 means near-perfect balance.
    // Max_min ratio should be near 1
    int total_size = 0;
    int min_size = 0;
    int max_size = 0;

    // Calculate the total, min, and max array size across all ranks
    MPI_Reduce(&array_size, &total_size, 1, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD);
    MPI_Reduce(&array_size, &min_size, 1, MPI_INT, MPI_MIN, 0, MPI_COMM_WORLD);
    MPI_Reduce(&array_size, &max_size, 1, MPI_INT, MPI_MAX, 0, MPI_COMM_WORLD);

    double lir = 0.0;
    double cv = 0.0;
    double max_min_ratio = 0.0;

    if (total_rank > 0) {
        // Calculate mean size
        double mean_size = static_cast<double>(total_size) / total_rank;

        // Calculate Load Imbalance Ratio (LIR)
        lir = static_cast<double>(max_size - min_size) / mean_size;

        // Calculate local squared difference from the mean
        double local_squared_diff = (array_size - mean_size) * (array_size - mean_size);

        // Calculate the total squared difference across all ranks
        double total_squared_diff = 0.0;
        MPI_Reduce(&local_squared_diff, &total_squared_diff, 1, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);

        // Calculate standard deviation and Coefficient of Variation (CV)
        double std_dev = std::sqrt(total_squared_diff / total_rank);
        cv = std_dev / mean_size;

        // Calculate Max/Min Ratio, ensuring no division by zero
        if (min_size > 0) {
            max_min_ratio = static_cast<double>(max_size) / min_size;
        } else {
            max_min_ratio = std::numeric_limits<double>::infinity(); // Handle division by zero
        }
    }

    return std::make_tuple(lir, cv, max_min_ratio);

}


// show_variable_generic(hash_table, "hash_table", hash_table_rows, "Entity", "device", rank, iterations, "", 0);
// show_device_entity_variable(local_data, local_data_size, rank, "local_data", 1);
// show_device_variable(local_data_temp_device, local_count, 2, rank, "local data temp device", 0);


