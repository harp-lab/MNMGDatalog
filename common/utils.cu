#include <string>
#include <chrono>
#include <stdlib.h>

using namespace std;
struct Entity {
    int key;
    int value;
};

struct Output {
    int block_size;
    int grid_size;
    int total_rank;
    int input_rows;
    int hashtable_rows;
    int iterations;
    int output_size;
    double load_factor;
    double reverse_time;
    int hashtable_build_rate;
    const char *dataset_name;
    const char *output_file_name;
    double total_time;
    double initialization_time;
    double fileio_time;
    double hashtable_build_time;
    double join_time;
    double buffer_preparation_time;
    double communication_time;
    double merge_time;
    double finalization_time;
};

struct KernelTimer {
    hipEvent_t start;
    hipEvent_t stop;

    KernelTimer() {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~KernelTimer() {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void start_timer() {
        hipEventRecord(start, 0);
    }

    void stop_timer() {
        hipEventRecord(stop, 0);
    }

    float get_spent_time() {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        elapsed /= 1000.0;
        return elapsed;
    }
};

struct is_equal {
    __host__ __device__
    bool operator()(const Entity &lhs, const Entity &rhs) {
        if ((lhs.key == rhs.key) && (lhs.value == rhs.value))
            return true;
        return false;
    }
};

// Predicate to check if key and value are equal
struct is_key_equal_value {
    __host__ __device__
    bool operator()(const Entity& e) {
        return e.key == e.value;
    }
};


struct cmp {
    __host__ __device__
    bool operator()(const Entity &lhs, const Entity &rhs) {
        if (lhs.key < rhs.key)
            return true;
        else if (lhs.key > rhs.key)
            return false;
        else {
            if (lhs.value < rhs.value)
                return true;
            else if (lhs.value > rhs.value)
                return false;
            return true;
        }
    }
};

struct set_cmp {
    __host__ __device__
    bool operator()(const Entity &lhs, const Entity &rhs) {
        if (lhs.key == rhs.key) {
            // If keys are equal, compare values
            return lhs.value < rhs.value;
        }
        return lhs.key < rhs.key;
    }
};

__device__ int get_position(int key, int hash_table_row_size) {
    key ^= key >> 16;
    key *= 0x85ebca6b;
    key ^= key >> 13;
    key *= 0xc2b2ae35;
    key ^= key >> 16;
    return key & (hash_table_row_size - 1);
}

void show_time_spent(string message,
                     chrono::high_resolution_clock::time_point time_point_begin,
                     chrono::high_resolution_clock::time_point time_point_end) {
    chrono::duration<double> time_span = time_point_end - time_point_begin;
    cout << message << ": " << time_span.count() << " seconds" << endl;
}

double get_time_spent(string message,
                      chrono::high_resolution_clock::time_point time_point_begin,
                      chrono::high_resolution_clock::time_point time_point_end) {
    chrono::duration<double> time_span = time_point_end - time_point_begin;
    if (message != "")
        cout << message << ": " << time_span.count() << " seconds" << endl;
    return time_span.count();
}

void show_relation(int *data, int total_rows,
                   int total_columns, const char *relation_name,
                   int visible_rows, int skip_zero) {
    int count = 0;
    cout << "Relation name: " << relation_name << endl;
    cout << "===================================" << endl;
    for (int i = 0; i < total_rows; i++) {
        int skip = 0;
        for (int j = 0; j < total_columns; j++) {
            if ((skip_zero == 1) && (data[(i * total_columns) + j] == 0)) {
                skip = 1;
                continue;
            }
            cout << data[(i * total_columns) + j] << " ";
        }
        if (skip == 1)
            continue;
        cout << endl;
        count++;
        if (count == visible_rows) {
            cout << "Result cropped at row " << count << "\n" << endl;
            return;
        }

    }
    cout << "Result counts " << count << "\n" << endl;
    cout << "" << endl;
}

int *get_relation_from_file(const char *file_path, int total_rows, int total_columns, char separator) {
    int *data = (int *) malloc(total_rows * total_columns * sizeof(int));
    FILE *data_file = fopen(file_path, "r");
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            if (j != (total_columns - 1)) {
                fscanf(data_file, "%d%c", &data[(i * total_columns) + j], &separator);
            } else {
                fscanf(data_file, "%d", &data[(i * total_columns) + j]);
            }
        }
    }
    return data;
}

void get_relation_from_file_gpu(int *data, const char *file_path, int total_rows, int total_columns, char separator) {
    FILE *data_file = fopen(file_path, "r");
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            if (j != (total_columns - 1)) {
                fscanf(data_file, "%d%c", &data[(i * total_columns) + j], &separator);
            } else {
                fscanf(data_file, "%d", &data[(i * total_columns) + j]);
            }
        }
    }
}


void get_random_relation(int *data, int total_rows, int total_columns) {
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            data[(i * total_columns) + j] = (rand() % (32767 - 0 + 1)) + 0;
        }
    }
}

void get_string_relation(int *data, int total_rows, int total_columns) {
    int x = 1, y = 2;
    for (int i = 0; i < total_rows; i++) {
        data[(i * total_columns) + 0] = x++;
        data[(i * total_columns) + 1] = y++;
    }
}

void get_reverse_relation_gpu(int *reverse_data, int *data, int total_rows, int total_columns) {
    for (int i = 0; i < total_rows; i++) {
        int pos = total_columns - 1;
        for (int j = 0; j < total_columns; j++) {
            reverse_data[(i * total_columns) + j] = data[(i * total_columns) + pos];
            pos--;
        }
    }
}


void show_hash_table(Entity *hash_table, long int hash_table_row_size, const char *hash_table_name) {
    int count = 0;
    cout << "Hashtable name: " << hash_table_name << endl;
    cout << "===================================" << endl;
    for (int i = 0; i < hash_table_row_size; i++) {
        if (hash_table[i].key != -1) {
            cout << hash_table[i].key << " " << hash_table[i].value << endl;
            count++;
        }
    }
    cout << "Row counts " << count << "\n" << endl;
    cout << "" << endl;
}

void show_entity_array(Entity *data, int data_rows, const char *array_name) {
    long int count = 0;
    cout << "Entity name: " << array_name << endl;
    cout << "===================================" << endl;
    for (int i = 0; i < data_rows; i++) {
        if (data[i].key != -1) {
            cout << data[i].key << " " << data[i].value << endl;
            count++;
        }
    }
    cout << "Row counts " << count << "\n" << endl;
    cout << "" << endl;
}

long int get_row_size(const char *data_path) {
    long int row_size = 0;
    int base = 1;
    for (int i = strlen(data_path) - 1; i >= 0; i--) {
        if (isdigit(data_path[i])) {
            int digit = (int) data_path[i] - '0';
            row_size += base * digit;
            base *= 10;
        }
    }
    return row_size;
}

void update_reverse_relation(Entity *data, int data_rows, int *reverse_relation) {
    for (int i = 0; i < data_rows; i++) {
        reverse_relation[i * 2] = data[i].key;
        reverse_relation[(i * 2) + 1] = data[i].value;
    }
}

void show_variable(int *host_data, int data_size, int group, int rank, string message) {
    cout << "Rank " << rank << ": " << message << " ----------------" << endl;
    for (int i = 0; i < data_size / group; i++) {
        for (int j = 0; j < group; j++) {
            cout << host_data[(i * group) + j] << " ";
        }
        if (data_size <= 20) {
            cout << ", ";
        } else {
            cout << endl;
        }
    }
    cout << endl;
}

void show_variable_entity(Entity *host_data, int data_size, int rank, string message) {
    cout << "Rank " << rank << ", size " << data_size << " : " << message << " ----------------" << endl;
    for (int i = 0; i < data_size; i++) {
        cout << host_data[i].key << " " << host_data[i].value;
        if (data_size <= 20) {
            cout << ", ";
        } else {
            cout << endl;
        }
    }
    cout << endl;
}

void show_device_variable(int *device_data, int device_data_size, int group, int rank, string message, int size_only) {
    int *host_data = (int *) malloc(device_data_size * sizeof(int));
    hipMemcpy(host_data, device_data, device_data_size * sizeof(int), hipMemcpyDeviceToHost);
    cout << "Rank " << rank << ", size " << device_data_size << " : " << message << " ----------------" << endl;
    if(size_only != 1){
        for (int i = 0; i < device_data_size / group; i++) {
            for (int j = 0; j < group; j++) {
                cout << host_data[(i * group) + j] << " ";
            }
            if (device_data_size <= 20) {
                cout << ", ";
            } else {
                cout << endl;
            }
        }
        cout << endl;
    }
    free(host_data);
}

// show_device_entity_variable(hash_table, hash_table_rows, rank, "hash_table");
void show_device_entity_variable(Entity *device_data, int device_data_size, int rank, string message, int size_only) {
    Entity *host_data = (Entity *) malloc(device_data_size * sizeof(Entity));
    hipMemcpy(host_data, device_data, device_data_size * sizeof(Entity), hipMemcpyDeviceToHost);
    cout << "Rank " << rank << ", size " << device_data_size << " : " << message << " ----------------" << endl;
    if(size_only != 1) {
        for (int i = 0; i < device_data_size; i++) {
            cout << host_data[i].key << " " << host_data[i].value << endl;
        }
        cout << endl;
    }
    free(host_data);
}