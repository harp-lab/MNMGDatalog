Entity *get_split_relation_pass_method(int rank, Entity *local_data_device,
                                       int row_size, int total_columns, int total_rank,
                                       int grid_size, int block_size, int cuda_aware_mpi,
                                       int *receive_size,
                                       double *buffer_preparation_time,
                                       double *communication_time, int iterations) {
    double start_time, end_time, elapsed_time;
    double prep_time = 0.0, comm_time = 0.0;
    start_time = MPI_Wtime();
    int *send_count;
    checkCuda(hipMalloc((void **) &send_count, total_rank * sizeof(int)));
    checkCuda(hipMemset(send_count, 0, total_rank * sizeof(int)));
    int *send_displacements;
    checkCuda(hipMalloc((void **) &send_displacements, total_rank * sizeof(int)));
    checkCuda(hipMemset(send_displacements, 0, total_rank * sizeof(int)));
    int *send_displacements_temp;
    checkCuda(hipMalloc((void **) &send_displacements_temp, total_rank * sizeof(int)));
    checkCuda(hipMemset(send_displacements_temp, 0, total_rank * sizeof(int)));
    get_send_count<<<grid_size, block_size>>>(local_data_device, row_size, send_count, total_rank);
    thrust::exclusive_scan(thrust::device, send_count, send_count + total_rank, send_displacements);
    hipMemcpy(send_displacements_temp, send_displacements, total_rank * sizeof(int), hipMemcpyDeviceToDevice);
    Entity *send_data;
    checkCuda(hipMalloc((void **) &send_data, row_size * sizeof(Entity)));
    get_rank_data<<<grid_size, block_size>>>(local_data_device, row_size, send_displacements_temp,
                                             total_rank, send_data);
    int mpi_error;

    int *send_count_host = (int *) malloc(total_rank * sizeof(int));
    int *receive_count_host = (int *) malloc(total_rank * sizeof(int));
    int *send_displacements_host = (int *) malloc(total_rank * sizeof(int));
    int *receive_displacements_host = (int *) malloc(total_rank * sizeof(int));
    memset(send_count_host, 0, total_rank * sizeof(int));
    memset(receive_count_host, 0, total_rank * sizeof(int));
    memset(send_displacements_host, 0, total_rank * sizeof(int));
    memset(receive_displacements_host, 0, total_rank * sizeof(int));
    hipMemcpy(send_count_host, send_count, total_rank * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(send_displacements_host, send_displacements, total_rank * sizeof(int), hipMemcpyDeviceToHost);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    prep_time += elapsed_time;
    start_time = MPI_Wtime();
    mpi_error = MPI_Alltoall(send_count_host, 1, MPI_INT, receive_count_host, 1, MPI_INT, MPI_COMM_WORLD);
    if (mpi_error != MPI_SUCCESS) {
        char error_string[BUFSIZ];
        int length_of_error_string;
        MPI_Error_string(mpi_error, error_string, &length_of_error_string);
        fprintf(stderr, "MPI error on MPI_Alltoall call: %s\n", error_string);
        MPI_Abort(MPI_COMM_WORLD, mpi_error);
    }
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    comm_time += elapsed_time;
    start_time = MPI_Wtime();
    int total_receive = thrust::reduce(thrust::host, receive_count_host, receive_count_host + total_rank, 0,
                                       thrust::plus<int>());

    int global_total_send = 0;
    int global_total_receive = 0;
    MPI_Allreduce(&row_size, &global_total_send, 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
    MPI_Allreduce(&total_receive, &global_total_receive, 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);

    thrust::exclusive_scan(thrust::host, receive_count_host, receive_count_host + total_rank,
                           receive_displacements_host);
    Entity *receive_data;
    checkCuda(hipMalloc((void **) &receive_data, total_receive * sizeof(Entity)));
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    prep_time += elapsed_time;
    start_time = MPI_Wtime();
    if (cuda_aware_mpi) {
        mpi_error = MPI_Alltoallv(send_data, send_count_host, send_displacements_host, MPI_UINT64_T,
                                  receive_data, receive_count_host, receive_displacements_host, MPI_UINT64_T,
                                  MPI_COMM_WORLD);
        if (mpi_error != MPI_SUCCESS) {
            char error_string[BUFSIZ];
            int length_of_error_string;
            MPI_Error_string(mpi_error, error_string, &length_of_error_string);
            fprintf(stderr, "MPI error on CUDA AWARE MPI MPI_Alltoallv call: %s\n", error_string);
            MPI_Abort(MPI_COMM_WORLD, mpi_error);
        }
    } else {
        Entity *send_data_host = (Entity *) malloc(row_size * sizeof(Entity));
        Entity *receive_data_host = (Entity *) malloc(total_receive * sizeof(Entity));
        hipMemcpy(send_data_host, send_data, row_size * sizeof(Entity), hipMemcpyDeviceToHost);
        mpi_error = MPI_Alltoallv(send_data_host, send_count_host, send_displacements_host, MPI_UINT64_T,
                                  receive_data_host, receive_count_host, receive_displacements_host, MPI_UINT64_T,
                                  MPI_COMM_WORLD);
        if (mpi_error != MPI_SUCCESS) {
            char error_string[BUFSIZ];
            int length_of_error_string;
            MPI_Error_string(mpi_error, error_string, &length_of_error_string);
            fprintf(stderr, "MPI error on host MPI_Alltoallv call: %s\n", error_string);
            MPI_Abort(MPI_COMM_WORLD, mpi_error);
        }
        hipMemcpy(receive_data, receive_data_host, total_receive * sizeof(Entity), hipMemcpyHostToDevice);
        free(send_data_host);
        free(receive_data_host);
    }
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    comm_time += elapsed_time;
    start_time = MPI_Wtime();
    *receive_size = total_receive;
    free(send_count_host);
    free(receive_count_host);
    free(send_displacements_host);
    free(receive_displacements_host);
    hipFree(send_count);
    hipFree(send_displacements);
    hipFree(send_displacements_temp);
    hipFree(send_data);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    prep_time += elapsed_time;
    *buffer_preparation_time = prep_time;
    *communication_time = comm_time;
    return receive_data;
}

Entity *get_split_relation_sort_method(int rank, Entity *local_data_device,
                                       int row_size, int total_columns, int total_rank,
                                       int grid_size, int block_size, int cuda_aware_mpi, int *size,
                                       double *buffer_preparation_time,
                                       double *communication_time, int iterations) {
    double start_time, end_time, elapsed_time;
    double prep_time = 0.0, comm_time = 0.0;
    start_time = MPI_Wtime();
    int mpi_error;
    thrust::device_vector<uint8_t> row_mapping(row_size);

    thrust::transform(
            thrust::device, local_data_device,
            local_data_device + row_size, row_mapping.begin(),
    [total_rank = total_rank] __device__(
    const Entity &entity) -> uint8_t{
            return (uint8_t)(get_rank(entity.key, total_rank));
    });

    thrust::stable_sort_by_key(thrust::device, row_mapping.begin(), row_mapping.end(), local_data_device);

    thrust::device_vector<int> unique_rank_row_count(total_rank);
    thrust::device_vector<uint8_t> unique_rank(total_rank);

    auto unique_rank_range = thrust::reduce_by_key(
            thrust::device, row_mapping.begin(), row_mapping.end(),
            thrust::constant_iterator<int>(1), unique_rank.begin(),
            unique_rank_row_count.begin());
    auto total_unique_rank = unique_rank_range.first - unique_rank.begin();
    unique_rank_row_count.resize(total_unique_rank);
    unique_rank.resize(total_unique_rank);
    thrust::host_vector<int> unique_rank_row_count_host(unique_rank_row_count);
    thrust::host_vector<uint8_t> unique_rank_host(unique_rank);
    thrust::host_vector<int> send_count_host(total_rank);
    for (int i = 0; i < total_unique_rank; i++) {
        send_count_host[unique_rank_host[i]] = unique_rank_row_count_host[i];
    }
    thrust::host_vector<int> receive_count_host(total_rank);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    prep_time += elapsed_time;

    start_time = MPI_Wtime();
    mpi_error = MPI_Alltoall(send_count_host.data(), 1, MPI_INT,
                             receive_count_host.data(), 1, MPI_INT, MPI_COMM_WORLD);
    if (mpi_error != MPI_SUCCESS) {
        char error_string[BUFSIZ];
        int length_of_error_string;
        MPI_Error_string(mpi_error, error_string, &length_of_error_string);
        fprintf(stderr, "MPI error on MPI_Alltoall call: %s\n", error_string);
        MPI_Abort(MPI_COMM_WORLD, mpi_error);
    }
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    comm_time += elapsed_time;

    start_time = MPI_Wtime();

    int total_receive = thrust::reduce(receive_count_host.begin(), receive_count_host.end());

    thrust::host_vector<int> send_displacements_host(total_rank);
    thrust::host_vector<int> receive_displacements_host(total_rank);

    thrust::exclusive_scan(send_count_host.begin(), send_count_host.end(), send_displacements_host.begin());
    thrust::exclusive_scan(receive_count_host.begin(), receive_count_host.end(), receive_displacements_host.begin());

    Entity *receive_data;
    checkCuda(hipMalloc((void **) &receive_data, total_receive * sizeof(Entity)));

    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    prep_time += elapsed_time;
    start_time = MPI_Wtime();
    if (cuda_aware_mpi) {
        std::cout << "Rank: " << rank << ", Size: " << total_receive << std::endl;
        mpi_error = MPI_Alltoallv(local_data_device, send_count_host.data(), send_displacements_host.data(),
                                  MPI_UINT64_T,
                                  receive_data, receive_count_host.data(), receive_displacements_host.data(),
                                  MPI_UINT64_T,
                                  MPI_COMM_WORLD);
        if (mpi_error != MPI_SUCCESS) {
            char error_string[BUFSIZ];
            int length_of_error_string;
            MPI_Error_string(mpi_error, error_string, &length_of_error_string);
            fprintf(stderr, "MPI error on CUDA AWARE MPI MPI_Alltoallv call: %s\n", error_string);
            MPI_Abort(MPI_COMM_WORLD, mpi_error);
        }
    } else {
        Entity *send_data_host = (Entity *) malloc(row_size * sizeof(Entity));
        Entity *receive_data_host = (Entity *) malloc(total_receive * sizeof(Entity));
        hipMemcpy(send_data_host, local_data_device, row_size * sizeof(Entity), hipMemcpyDeviceToHost);
        mpi_error = MPI_Alltoallv(send_data_host, send_count_host.data(), send_displacements_host.data(),
                                  MPI_UINT64_T,
                                  receive_data_host, receive_count_host.data(), receive_displacements_host.data(),
                                  MPI_UINT64_T,
                                  MPI_COMM_WORLD);
        if (mpi_error != MPI_SUCCESS) {
            char error_string[BUFSIZ];
            int length_of_error_string;
            MPI_Error_string(mpi_error, error_string, &length_of_error_string);
            fprintf(stderr, "MPI error on host MPI_Alltoallv call: %s\n", error_string);
            MPI_Abort(MPI_COMM_WORLD, mpi_error);
        }
        hipMemcpy(receive_data, receive_data_host, total_receive * sizeof(Entity), hipMemcpyHostToDevice);
        free(send_data_host);
        free(receive_data_host);
    }
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    comm_time += elapsed_time;
    *buffer_preparation_time = prep_time;
    *communication_time = comm_time;
    *size = total_receive;
    return receive_data;
}

Entity *get_split_relation(int rank, Entity *data_device,
                           int data_size, int total_columns, int total_rank,
                           int grid_size, int block_size, int cuda_aware_mpi, int *size, int method,
                           double *buffer_preparation_time,
                           double *communication_time, int iterations) {
    if (method == 0) {
        return get_split_relation_pass_method(rank, data_device, data_size,
                                              total_columns, total_rank, grid_size, block_size, cuda_aware_mpi, size,
                                              buffer_preparation_time, communication_time, iterations);
    } else {
        return get_split_relation_sort_method(rank, data_device, data_size,
                                              total_columns, total_rank, grid_size, block_size, cuda_aware_mpi, size,
                                              buffer_preparation_time, communication_time, iterations);
    }
}