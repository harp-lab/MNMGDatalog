#include "hip/hip_runtime.h"
/*
 * Method that returns position in the hashtable for a key using Murmur3 hash
 * */


__global__
void build_hash_table(Entity *hash_table, long int hash_table_row_size,
                      int *relation, long int relation_rows, int relation_columns) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_rows; i += stride) {
        int key = relation[(i * relation_columns) + 0];
        int value = relation[(i * relation_columns) + 1];
        int position = get_position(key, hash_table_row_size);
        while (true) {
            int existing_key = atomicCAS(&hash_table[position].key, -1, key);
            if (existing_key == -1) {
                hash_table[position].value = value;
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
    }
}

__global__
void build_hash_table_entity(Entity *hash_table, long int hash_table_size,
                             Entity *relation, long int relation_size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_size) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_size; i += stride) {
        int key = relation[i].key;
        int value = relation[i].value;
        int position = get_position(key, hash_table_size);
        while (true) {
            int existing_key = atomicCAS(&hash_table[position].key, -1, key);
            if (existing_key == -1) {
                hash_table[position].value = value;
                break;
            }
            position = (position + 1) & (hash_table_size - 1);
        }
    }
}

__global__
void copy_t_delta(Entity *t_delta, int *reverse_relation, long int reverse_relation_rows, int relation_columns) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= reverse_relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < reverse_relation_rows; i += stride) {
        t_delta[i].key = reverse_relation[(i * relation_columns) + 0];
        t_delta[i].value = reverse_relation[(i * relation_columns) + 1];
    }
}


__global__
void initialize_result_t_delta(Entity *result, Entity *t_delta,
                               int *relation, long int relation_rows, int relation_columns) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_rows; i += stride) {
        t_delta[i].key = result[i].key = relation[(i * relation_columns) + 0];
        t_delta[i].value = result[i].value = relation[(i * relation_columns) + 1];
    }
}

__global__
void copy_struct(Entity *source, long int source_rows, Entity *destination) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= source_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < source_rows; i += stride) {
        destination[i].key = source[i].key;
        destination[i].value = source[i].value;
    }
}

__global__
void negative_fill_struct(Entity *source, long int source_rows) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= source_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < source_rows; i += stride) {
        source[i].key = -1;
        source[i].value = -1;
    }
}

__global__
void get_reverse_relation(int *relation, long int relation_rows, int relation_columns, Entity *t_delta) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (long int i = index; i < relation_rows; i += stride) {
        t_delta[i].key = relation[(i * relation_columns) + 0];
        t_delta[i].value = relation[(i * relation_columns) + 1];
    }
}


__global__
void get_join_result_size(Entity *hash_table, long int hash_table_row_size,
                          int *t_delta, long int reverse_relation_rows,
                          int *join_result_size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= reverse_relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < reverse_relation_rows; i += stride) {
        int key = t_delta[i * 2];
        int current_size = 0;
        int position = get_position(key, hash_table_row_size);
        while (true) {
            if (hash_table[position].key == key) {
                current_size++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
        join_result_size[i] = current_size;
    }
}

__global__
void get_join_result(Entity *hash_table, int hash_table_row_size,
                     int *t_delta, int reverse_relation_rows, int *offset, Entity *join_result) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= reverse_relation_rows) return;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < reverse_relation_rows; i += stride) {
        int key = t_delta[i * 2];
        int value = t_delta[(i * 2) + 1];
        int start_index = offset[i];
        int position = get_position(key, hash_table_row_size);
        while (true) {
            if (hash_table[position].key == key) {
                join_result[start_index].key = hash_table[position].value;
                join_result[start_index].value = value;
                start_index++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
    }
}


__global__
void get_join_result_size_entity(Entity *hash_table, long int hash_table_size,
                          Entity *t_delta, long int t_delta_size,
                          int *join_result_size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= t_delta_size) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < t_delta_size; i += stride) {
        int key = t_delta[i].key;
        int current_size = 0;
        int position = get_position(key, hash_table_size);
        while (true) {
            if (hash_table[position].key == key) {
                current_size++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_size - 1);
        }
        join_result_size[i] = current_size;
    }
}

__global__
void get_join_result_entity(Entity *hash_table, int hash_table_size,
                     Entity *t_delta, int t_delta_size, int *offset, Entity *join_result) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= t_delta_size) return;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < t_delta_size; i += stride) {
        int key = t_delta[i].key;
        int value = t_delta[i].value;
        int start_index = offset[i];
        int position = get_position(key, hash_table_size);
        while (true) {
            if (hash_table[position].key == key) {
                join_result[start_index].key = hash_table[position].value;
                join_result[start_index].value = value;
                start_index++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_size - 1);
        }
    }
}


__global__
void get_join_result_size_ar(Entity *hash_table, long int hash_table_row_size,
                             int *t_delta, long int relation_rows,
                             int *join_result_size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_rows; i += stride) {
        int key = t_delta[(i * 2) + 1];
        int current_size = 0;
        int position = get_position(key, hash_table_row_size);
        while (true) {
            if (hash_table[position].key == key) {
                current_size++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
        join_result_size[i] = current_size;
    }
}

__global__
void get_join_result_ar(Entity *hash_table, int hash_table_row_size,
                        int *t_delta, int relation_rows, int *offset, Entity *join_result) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < relation_rows; i += stride) {
        int key = t_delta[(i * 2) + 1];
        int value = t_delta[i * 2];
        int start_index = offset[i];
        int position = get_position(key, hash_table_row_size);
        while (true) {
            if (hash_table[position].key == key) {
                join_result[start_index].key = value;
                join_result[start_index].value = hash_table[position].value;
                start_index++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
    }
}

/* Semi naive kernels */

__global__ void warm_up_kernel() {}

__host__ __device__ int get_rank(int key, int total_rank) {
    key ^= key >> 16;
    key *= 0x85ebca6b;
    key ^= key >> 13;
    key *= 0xc2b2ae35;
    key ^= key >> 16;
    return key % total_rank;
}

__global__ void get_send_count(Entity *local_data, int local_data_row_count,
                               int *send_count, int total_rank) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= local_data_row_count) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < local_data_row_count; i += stride) {
        int key = local_data[i].key;
        int destination_rank = get_rank(key, total_rank);
        atomicAdd(&send_count[destination_rank], 1);
    }
}

__global__ void get_rank_data(Entity *local_data, int local_data_row_count,
                              int *send_count_offset, int total_rank, Entity *rank_data) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= local_data_row_count) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < local_data_row_count; i += stride) {
        int key = local_data[i].key;
        int value = local_data[i].value;
        int destination_rank = get_rank(key, total_rank);
        int current_position = atomicAdd(&send_count_offset[destination_rank], 1);
        rank_data[current_position].key = key;
        rank_data[current_position].value = value;
    }
}


__global__ void create_entity_ar(Entity *data, int data_rows, int *input_data) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= data_rows) return;

    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < data_rows; i += stride) {
        data[i].key = input_data[i * 2];
        data[i].value = input_data[(i * 2) + 1];
    }
}

__global__ void create_entity_ar_reverse(Entity *data, int data_rows, int *input_data) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= data_rows) return;

    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < data_rows; i += stride) {
        data[i].value = input_data[i * 2];
        data[i].key = input_data[(i * 2) + 1];
    }
}


__global__ void reverse_t_full(int *data, int data_rows, Entity *input_data) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= data_rows) return;

    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < data_rows; i += stride) {
        data[i * 2] = input_data[i].value;
        data[(i * 2) + 1] = input_data[i].key;
    }
}

__global__ void get_int_ar_from_entity_ar(int *data, int data_rows, Entity *input_data) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= data_rows) return;

    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < data_rows; i += stride) {
        data[i * 2] = input_data[i].key;
        data[(i * 2) + 1] = input_data[i].value;
    }
}

__global__ void reverse_entity_ar(Entity *input_data, int data_rows, Entity *output_data) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= data_rows) return;

    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < data_rows; i += stride) {
        int key = input_data[i].key;
        int value = input_data[i].value;
        output_data[i].key = value;
        output_data[i].value = key;
    }
}
