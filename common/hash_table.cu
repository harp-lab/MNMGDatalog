#include "hip/hip_runtime.h"
__global__
void build_hash_table_entity(Entity *hash_table, long int hash_table_size,
                             Entity *relation, long int relation_size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_size) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_size; i += stride) {
        int key = relation[i].key;
        int value = relation[i].value;
        int position = get_position(key, hash_table_size);
        while (true) {
            int existing_key = atomicCAS(&hash_table[position].key, -1, key);
            if (existing_key == -1) {
                hash_table[position].value = value;
                break;
            }
            position = (position + 1) & (hash_table_size - 1);
        }
    }
}

Entity *get_hash_table(int grid_size, int block_size, Entity *edge, int edge_size,
                       int *hash_table_size, double *compute_time) {
    double start_time, end_time, elapsed_time;
    start_time = MPI_Wtime();
    Entity *hash_table = nullptr;

    if (edge_size == 0) {
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        *compute_time = elapsed_time;
        return hash_table;
    }

    double load_factor = 0.6;
    int hash_table_rows = (int) std::ceil(edge_size / load_factor);
    hash_table_rows = 1 << (int) ceil(log2(hash_table_rows));
#ifdef DEBUG
    cout << "hash_table_rows * sizeof(Entity): " << hash_table_rows * sizeof(Entity) << endl;
#endif
    checkCuda(hipMalloc((void **) &hash_table, hash_table_rows * sizeof(Entity)));
    Entity negative_entity;
    negative_entity.key = -1;
    negative_entity.value = -1;
    thrust::fill(thrust::device, hash_table, hash_table + hash_table_rows, negative_entity);
    build_hash_table_entity<<<grid_size, block_size>>>(hash_table, hash_table_rows, edge,
                                                       edge_size);
    *hash_table_size = hash_table_rows;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    *compute_time = elapsed_time;
    return hash_table;
}