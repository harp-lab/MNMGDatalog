#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
// #include "common/error_handler.cu"
#include "../include/exception.cuh"
// #include "common/utils.cu"
#include "../include/utils.cuh"
// #include "common/kernels.cu"
#include "../include/gkernel.cuh"

Output output;

#define BLOCK_START(process_id, total_process, n) ((process_id)*(n)/(total_process))
#define BLOCK_SIZE(process_id, total_process, n) \
    (BLOCK_START(process_id + 1, total_process, n) - BLOCK_START(process_id, total_process, n))

std::pair<int *, long int> get_split_relation(int rank, int *local_data, long int local_count,
                                         int total_columns, int nprocs) {
    int i, j;
    // Array of vectors where we need to push to integers to proper rank based on hash
    std::vector<int> rank_data[nprocs];
    // Count the number of data to be sent to destination processor
    int *send_count = (int *) calloc(nprocs, sizeof(int));
    int *receive_count = (int *) calloc(nprocs, sizeof(int));

    int *send_displacements = (int *) calloc(nprocs, sizeof(int));
    for (i = 0; i < local_count; i += total_columns) {
        int destination_rank = local_data[i] % nprocs;
        send_count[destination_rank] += total_columns;
        rank_data[destination_rank].push_back(local_data[i]);       // key
        rank_data[destination_rank].push_back(local_data[i + 1]);   // value
    }

    // Calculate the displacements for each process for current process
    long int total_send = send_count[0];
    for (i = 1; i < nprocs; i++) {
        send_displacements[i] = send_displacements[i - 1] + send_count[i - 1];
        total_send += send_count[i];
    }

    // Create the send data buffer for each process from the array of vectors
    int *send_data = (int *) calloc(total_send, sizeof(int));
    for (i = 0; i < nprocs; i++) {
        int pos = 0;
        for (j = send_displacements[i]; j < send_displacements[i] + rank_data[i].size(); j++) {
            send_data[j] = rank_data[i][pos++];
        }
    }

    // Send total number of items for each process from current process
    // Send 1 integer to each process from send buffer
    MPI_Alltoall(send_count, 1, MPI_INT, receive_count, 1, MPI_INT, MPI_COMM_WORLD);

    // Calculate the displacements for receive buffer
    int *receive_displacements = (int *) calloc(nprocs, sizeof(int));
    long int total_receive = receive_count[0];
    for (i = 1; i < nprocs; i++) {
        receive_displacements[i] = receive_displacements[i - 1] + receive_count[i - 1];
        total_receive += receive_count[i];
    }

    // Set the receive data buffer for each processor
    int *receive_data = (int *) calloc(total_receive, sizeof(int));
    MPI_Alltoallv(send_data, send_count, send_displacements, MPI_INT,
                  receive_data, receive_count, receive_displacements, MPI_INT,
                  MPI_COMM_WORLD);
    std::set<std::pair<int, int>> unique_rows;
    for (i = 0; i < total_receive; i += total_columns) {
        unique_rows.insert(std::make_pair(receive_data[i], receive_data[i + 1]));
    }

    long int total_unique_receive = unique_rows.size() * total_columns;
    int *unique_receive_data = (int *) calloc(total_unique_receive, sizeof(int));
    long int count = 0;
    for (auto p: unique_rows) {
        unique_receive_data[count++] = p.first;
        unique_receive_data[count++] = p.second;
    }

    free(receive_data);
    free(send_count);
    free(send_displacements);
    free(send_data);
    free(receive_count);
    free(receive_displacements);
    return std::make_pair(unique_receive_data, total_unique_receive);
}

int *get_reverse_relation(int *relation, long int relation_size, int total_columns) {
    int *reverse_relation = (int *) malloc(relation_size * sizeof(int));
    for (int i = 0; i < relation_size; i += total_columns) {
        reverse_relation[i] = relation[i + 1];
        reverse_relation[i + 1] = relation[i];
    }
    return reverse_relation;
}


std::pair<int *, long int> update_t_full(int *t_full, long int t_full_size, int *facts, long int facts_size,
                                    int total_columns) {
    std::set<std::pair<int, int>> unique_rows;
    for (int i = 0; i < t_full_size; i += total_columns) {
        unique_rows.insert(std::make_pair(t_full[i], t_full[i + 1]));
    }
    for (int i = 0; i < facts_size; i += total_columns) {
        unique_rows.insert(std::make_pair(facts[i], facts[i + 1]));
    }
    long int total_unique_rows = unique_rows.size() * total_columns;
    int *unique_data = (int *) calloc(total_unique_rows, sizeof(int));
    long int count = 0;
    for (auto p: unique_rows) {
        unique_data[count++] = p.first;
        unique_data[count++] = p.second;
    }
    return std::make_pair(unique_data, count);
}


int main(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);

    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    setlocale(LC_ALL, "");
    double elapsed_time = -MPI_Wtime();
    double max_time = 0.0;
    int nprocs, rank;
    int i, j, k;
    MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    // Should pass the input filename in command line argument
    const char *input_file;
    if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "hipc_2019.bin";
    }
    // READ THE FILE IN PARALLEL
    // Reading filesize in bytes
    struct stat filestats;
    stat(input_file, &filestats);
    off_t filesize = filestats.st_size;

    // Calculating the current rank's starting row and number of rows
    // Scatter larger blocks among processes (non uniform)
    int total_columns = 2;
    long int total_rows = filesize / (sizeof(int) * total_columns);
    long int row_start = BLOCK_START(rank, nprocs, total_rows);
    long int row_size = BLOCK_SIZE(rank, nprocs, total_rows);
    long int local_count = row_size * total_columns;

    // Reading specific portion from the file as char in parallel
    long int offset = row_start * total_columns * sizeof(int);
    int *local_data = (int *) malloc(local_count * sizeof(int));
    MPI_File mpi_file_buffer;
    if (MPI_File_open(MPI_COMM_WORLD, input_file, MPI_MODE_RDONLY,
                      MPI_INFO_NULL, &mpi_file_buffer) != MPI_SUCCESS) {
        printf("Error opening file %s", input_file);
        MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    }
    MPI_File_read_at(mpi_file_buffer, offset, local_data, local_count,
                     MPI_INT, MPI_STATUS_IGNORE);
    MPI_File_close(&mpi_file_buffer);

    int *local_data_reverse = get_reverse_relation(local_data, local_count, total_columns);
    std::pair<int *, long int> input_relation_data = get_split_relation(rank, local_data, local_count,
                                                                   total_columns, nprocs);
    int *input_relation = input_relation_data.first;
    long int input_relation_size = input_relation_data.second;


    long int global_t_full_size, global_tc_size;
    std::pair<int *, long int> reverse_relation_data = get_split_relation(rank, local_data_reverse, local_count,
                                                                     total_columns, nprocs);
    int *reverse_relation = reverse_relation_data.first;
    long int reverse_relation_size = reverse_relation_data.second;

    long int input_relation_rows = input_relation_size / total_columns;
    long int reverse_relation_rows = reverse_relation_size / total_columns;

    // T_FULL is reverse_relation with first column as key
    int *t_full = (int *) malloc(reverse_relation_size * sizeof(int));
    // Copy the contents of the input_relation array to the new memory location
    std::memcpy(t_full, reverse_relation, reverse_relation_size * sizeof(int));
    long int t_full_size = reverse_relation_size;

    MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_LONG, MPI_SUM,
                  MPI_COMM_WORLD);


    int relation_columns = 2;
    long int join_result_rows;
    int block_size, grid_size;
    int *relation, *t_delta_host;
    Entity *hash_table, *result;
    Entity *result_host;
    Entity *hash_table_host;
    double load_factor = 0.4;
    long int hash_table_rows = (long int) input_relation_rows / load_factor;
    hash_table_rows = pow(2, ceil(log(hash_table_rows) / log(2)));
    checkCuda(hipMalloc((void **) &relation, input_relation_rows * relation_columns * sizeof(int)));
    checkCuda(hipMalloc((void **) &hash_table, hash_table_rows * sizeof(Entity)));
    block_size = 512;
    grid_size = 32 * number_of_sm;

    hipMemcpy(relation, input_relation, input_relation_rows * relation_columns * sizeof(int),
               hipMemcpyHostToDevice);

    Entity negative_entity;
    negative_entity.key = -1;
    negative_entity.value = -1;
    thrust::fill(thrust::device, hash_table, hash_table + hash_table_rows, negative_entity);
    build_hash_table<<<grid_size, block_size>>>
            (hash_table, hash_table_rows,
             relation, input_relation_rows,
             relation_columns);
    checkCuda(hipDeviceSynchronize());
    long int iterations = 0;

    int *t_delta;
    checkCuda(hipMalloc((void **) &t_delta, reverse_relation_rows * relation_columns * sizeof(int)));
    hipMemcpy(t_delta, reverse_relation, reverse_relation_rows * relation_columns * sizeof(int),
               hipMemcpyHostToDevice);

    while (true) {

        long int projection_rows = 0;
        int *join_offset;
        Entity *join_result;
        Entity *join_result_host;

        checkCuda(hipMalloc((void **) &join_offset, reverse_relation_rows * sizeof(int)));

        get_join_result_size<<<grid_size, block_size>>>(hash_table, hash_table_rows,
                                                        t_delta, reverse_relation_rows, join_offset);
        checkCuda(hipDeviceSynchronize());

        join_result_rows = thrust::reduce(thrust::device, join_offset, join_offset + reverse_relation_rows, 0);
        thrust::exclusive_scan(thrust::device, join_offset, join_offset + reverse_relation_rows, join_offset);

        checkCuda(hipMalloc((void **) &join_result, join_result_rows * sizeof(Entity)));
        get_join_result<<<grid_size, block_size>>>(hash_table, hash_table_rows,
                                                   t_delta, reverse_relation_rows, join_offset, join_result);
        checkCuda(hipDeviceSynchronize());

        thrust::stable_sort(thrust::device, join_result, join_result + join_result_rows, cmp());
        projection_rows = (thrust::unique(thrust::device,
                                          join_result, join_result + join_result_rows,
                                          is_equal())) - join_result;
        join_result_host = (Entity *) malloc(projection_rows * sizeof(Entity));
        hipMemcpy(join_result_host, join_result, projection_rows * sizeof(Entity),
                   hipMemcpyDeviceToHost);
//         Update the reverse relation which will be used in next iteration
        reverse_relation = (int *) realloc(reverse_relation, projection_rows * total_columns * sizeof(int));
        update_reverse_relation(join_result_host, projection_rows, reverse_relation);

        free(join_result_host);
        hipFree(join_offset);
        hipFree(join_result);


        // Scatter the new facts among relevant processes
        std::pair<int *, long int> scatter_fact_data = get_split_relation(rank, reverse_relation,
                                                                     projection_rows * total_columns,
                                                                     total_columns, nprocs);
        reverse_relation_size = scatter_fact_data.second;
        reverse_relation_rows = reverse_relation_size / total_columns;
        // Update the reverse relation which will be used in next iteration
        reverse_relation = (int *) realloc(reverse_relation, reverse_relation_rows * total_columns * sizeof(int));
        reverse_relation = scatter_fact_data.first;
        checkCuda(hipMalloc((void **) &t_delta, reverse_relation_rows * relation_columns * sizeof(int)));
        hipMemcpy(t_delta, reverse_relation, reverse_relation_rows * total_columns * sizeof(int),
                   hipMemcpyHostToDevice);

        // Update t full for current process
        std::pair<int *, long int> t_full_data = update_t_full(t_full, t_full_size,
                                                          reverse_relation,
                                                          reverse_relation_size,
                                                          total_columns);
        t_full_size = t_full_data.second;
        t_full = (int *) realloc(t_full, t_full_size * sizeof(int));
        t_full = t_full_data.first;

        // Check if the global t full size has changed in this iteration
        long int old_global_t_full_size = global_t_full_size;
        MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_LONG, MPI_SUM,
                      MPI_COMM_WORLD);
        iterations++;
        if (old_global_t_full_size == global_t_full_size) {
            break;
        }

    }

    // Reverse the t_full as we stored it in reverse order initially
    t_full = get_reverse_relation(t_full, t_full_size, total_columns);
    global_tc_size = global_t_full_size;

    // List the t full counts for each process and calculate the displacements in the final result
    long int *t_full_counts = (long int *) calloc(nprocs, sizeof(long int));
    MPI_Allgather(&t_full_size, 1, MPI_LONG,
                  t_full_counts, 1, MPI_LONG, MPI_COMM_WORLD);

    long int *t_full_displacements = (long int *) calloc(nprocs, sizeof(long int));
    for (i = 1; i < nprocs; i++) {
        t_full_displacements[i] = t_full_displacements[i - 1] + t_full_counts[i - 1];
    }

    // Write the t full to a offset of the output file
    MPI_File fh;
    std::string output_file = std::string(input_file) + "_tc.bin";
    const char *output_file_name = output_file.c_str();
    MPI_File_open(MPI_COMM_WORLD, output_file_name, MPI_MODE_WRONLY | MPI_MODE_CREATE, MPI_INFO_NULL, &fh);
    long int file_offset = t_full_displacements[rank] * sizeof(int);
    MPI_File_write_at(fh, file_offset, t_full, t_full_size, MPI_INT, MPI_STATUS_IGNORE);
    // Close the file and clean up
    MPI_File_close(&fh);


    free(local_data);
    free(local_data_reverse);
    free(input_relation);
    free(reverse_relation);
    free(t_full);
    free(t_full_counts);
    free(t_full_displacements);
    hipFree(hash_table);
    elapsed_time += MPI_Wtime();
    MPI_Allreduce(&elapsed_time, &max_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    if (rank == 0) {
        global_tc_size /= total_columns;
        printf("Total iterations %ld, TC size %ld, generated file %s\n",
               iterations, global_tc_size, output_file_name);
        printf("Total time: %.4lf seconds\n\n", max_time);
        printf("| # Input | # Process | # Iterations | # TC | Time (s) |\n");
        printf("| --- | --- | --- | --- | --- |\n");
        printf("| %'ld | %'d | %'ld | %'ld | %'8.4lf |\n",
               total_rows, nprocs, iterations, global_tc_size, max_time);
    }
    MPI_Finalize();
    return 0;
}