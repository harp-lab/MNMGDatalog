#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/iterator/transform_iterator.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/kernels.cu"
#include "common/comm.cu"
#include "common/join.cu"

using namespace std;


void benchmark(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    Output output;
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    setlocale(LC_ALL, "");
    double start_time, end_time, elapsed_time;
    double initialization_time = 0.0, max_initialization_time = 0.0;
    double finalization_time = 0.0, max_finalization_time = 0.0;
    double max_fileio_time = 0.0, max_hashtable_build_time = 0.0;
    double max_join_time = 0.0, max_merge_time = 0.0;
    double max_buffer_preparation_time = 0.0, max_communication_time = 0.0;
    double buffer_preparation_time = 0.0, communication_time = 0.0;
    double buffer_preparation_time_temp = 0.0, communication_time_temp = 0.0;
    double join_time = 0.0, merge_time = 0.0;
    double deduplication_time = 0.0, max_deduplication_time = 0.0;;
    double file_io_time = 0.0;
    double total_time = 0.0, max_total_time = 0.0;
    int total_rank, rank;
    int i;
    MPI_Comm_size(MPI_COMM_WORLD, &total_rank);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    warm_up_kernel<<<1, 1>>>();
    // Should pass the input filename in command line argument
    const char *input_file;
    int comm_method = 0;
    int job_run = 0;
    int cuda_aware_mpi = 0;

    if (argc == 5) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
        job_run = atoi(argv[4]);
    } else if (argc == 4) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
    } else if (argc == 3) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
    } else if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "hipc_2019.bin";
    }
    string output_file = string(input_file) + "_cc.bin";
    const char *output_file_name = output_file.c_str();

    // READ THE FILE IN PARALLEL
    // Reading filesize in bytes
    start_time = MPI_Wtime();
    struct stat filestats{};
    stat(input_file, &filestats);
    off_t filesize = filestats.st_size;

    // Calculating the current rank's starting row and number of rows
    // Scatter larger blocks among processes (non-uniform)
    int total_columns = 2;
    int total_rows = filesize / (sizeof(int) * total_columns);
    int row_start = BLOCK_START(rank, total_rank, total_rows);
    int row_size = BLOCK_SIZE(rank, total_rank, total_rows);
    int local_count = row_size * total_columns;

    // Reading specific portion from the file as char in parallel
    int offset = row_start * total_columns * sizeof(int);
    int *local_data_host = (int *) malloc(local_count * sizeof(int));
    MPI_File mpi_file_buffer;
    if (MPI_File_open(MPI_COMM_WORLD, input_file, MPI_MODE_RDONLY,
                      MPI_INFO_NULL, &mpi_file_buffer) != MPI_SUCCESS) {
        printf("Error opening file %s", input_file);
        MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    }
    MPI_File_read_at(mpi_file_buffer, offset, local_data_host, local_count, MPI_INT, MPI_STATUS_IGNORE);
    MPI_File_close(&mpi_file_buffer);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    file_io_time += elapsed_time;
    start_time = MPI_Wtime();
    int *local_data_temp_device;
    checkCuda(hipMalloc((void **) &local_data_temp_device, local_count * sizeof(int)));
    hipMemcpy(local_data_temp_device, local_data_host, local_count * sizeof(int), hipMemcpyHostToDevice);
    // Ensure edges are bidirectional by adding reverse edges
    int *local_data_reverse_temp_device;
    checkCuda(hipMalloc((void **) &local_data_reverse_temp_device, local_count * sizeof(int)));
    get_reverse_ar<<<grid_size, block_size>>>(local_data_temp_device, local_count, local_data_reverse_temp_device);

    // Create Entity array from GPU buffers
    Entity *local_data;
    int local_data_size = row_size * 2;
    checkCuda(hipMalloc((void **) &local_data, local_data_size * sizeof(Entity)));
    create_entity_ar_with_offset<<<grid_size, block_size>>>(local_data_temp_device, row_size, local_data, 0);
    create_entity_ar_with_offset<<<grid_size, block_size>>>(local_data_reverse_temp_device, row_size,
                                                            local_data, row_size);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    // Deduplicate local data
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, local_data, local_data + local_data_size, set_cmp());
    local_data_size = (thrust::unique(thrust::device,
                                      local_data, local_data + local_data_size,
                                      is_equal())) - local_data;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    // Create reverse entity array from local data
    start_time = MPI_Wtime();
    Entity *local_data_reverse;
    int local_data_reverse_size = local_data_size;
    checkCuda(hipMalloc((void **) &local_data_reverse, local_data_reverse_size * sizeof(Entity)));
    reverse_entity_ar<<<grid_size, block_size>>>(local_data, local_data_reverse_size, local_data_reverse);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    // Deduplicate local data reverse
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, local_data_reverse, local_data_reverse + local_data_reverse_size, set_cmp());
    local_data_reverse_size = (thrust::unique(thrust::device,
                                              local_data_reverse, local_data_reverse + local_data_reverse_size,
                                              is_equal())) - local_data_reverse;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    int iterations = 0;
    int input_relation_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    Entity *input_relation = get_split_relation(rank, local_data,
                                                local_data_size, total_columns, total_rank,
                                                grid_size, block_size, cuda_aware_mpi,
                                                &input_relation_size, comm_method,
                                                &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;

    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    int t_delta_size;
    Entity *t_delta = get_split_relation(rank, local_data_reverse,
                                         local_data_reverse_size, total_columns, total_rank,
                                         grid_size, block_size, cuda_aware_mpi, &t_delta_size, comm_method,
                                         &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;

    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, t_delta, t_delta + t_delta_size, set_cmp());
    t_delta_size = (thrust::unique(thrust::device,
                                   t_delta, t_delta + t_delta_size,
                                   is_equal())) - t_delta;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;
    start_time = MPI_Wtime();
    // T_FULL is t delta with first column as key
    Entity *t_full;
    checkCuda(hipMalloc((void **) &t_full, t_delta_size * sizeof(Entity)));
    hipMemcpy(t_full, t_delta, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);

    long long global_t_full_size;
    long long t_full_size = t_delta_size;
    MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_LONG_LONG, MPI_SUM, MPI_COMM_WORLD);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    merge_time += elapsed_time;
    start_time = MPI_Wtime();
    Entity *hash_table;
    double load_factor = 0.4;
    int hash_table_rows = (int) input_relation_size / load_factor;
    hash_table_rows = pow(2, ceil(log(hash_table_rows) / log(2)));
    checkCuda(hipMalloc((void **) &hash_table, hash_table_rows * sizeof(Entity)));
    Entity negative_entity;
    negative_entity.key = -1;
    negative_entity.value = -1;
    thrust::fill(thrust::device, hash_table, hash_table + hash_table_rows, negative_entity);
    build_hash_table_entity<<<grid_size, block_size>>>(hash_table, hash_table_rows, input_relation,
                                                       input_relation_size);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    MPI_Allreduce(&elapsed_time, &max_hashtable_build_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    while (true) {
        Entity *new_t_full;
        double temp_join_time = 0.0;
        int join_result_size = 0;
        Entity *join_result = get_join(grid_size, block_size, hash_table, hash_table_rows,
                                       t_delta, t_delta_size,
                                       &join_result_size, &temp_join_time);

        join_time += temp_join_time;
        // Scatter the new facts among relevant processes
        buffer_preparation_time_temp = 0.0;
        communication_time_temp = 0.0;
        Entity *t_delta_temp = get_split_relation(rank, join_result,
                                                  join_result_size, total_columns, total_rank,
                                                  grid_size, block_size, cuda_aware_mpi, &t_delta_size,
                                                  comm_method,
                                                  &buffer_preparation_time_temp, &communication_time_temp);
        buffer_preparation_time += buffer_preparation_time_temp;
        communication_time += communication_time_temp;
        start_time = MPI_Wtime();
        // Deduplicate scattered facts
        thrust::stable_sort(thrust::device, t_delta_temp, t_delta_temp + t_delta_size, set_cmp());
        t_delta_size = (thrust::unique(thrust::device,
                                       t_delta_temp, t_delta_temp + t_delta_size,
                                       is_equal())) - t_delta_temp;
        hipFree(t_delta);
        checkCuda(hipMalloc((void **) &t_delta, t_delta_size * sizeof(Entity)));
        hipMemcpy(t_delta, t_delta_temp, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        deduplication_time += elapsed_time;

        start_time = MPI_Wtime();
        // Update t delta which is the only new facts which are not in t full and will be used in next iteration
        t_delta_size = thrust::set_difference(thrust::device,
                                              t_delta, t_delta + t_delta_size,
                                              t_full, t_full + t_full_size,
                                              t_delta, set_cmp()) - t_delta;

        // set union of two sets (sorted t full and t delta)
        int new_t_full_size = t_delta_size + t_full_size;
        checkCuda(hipMalloc((void **) &new_t_full, new_t_full_size * sizeof(Entity)));
        new_t_full_size = thrust::set_union(thrust::device,
                                            t_full, t_full + t_full_size,
                                            t_delta, t_delta + t_delta_size,
                                            new_t_full, set_cmp()) - new_t_full;
        hipFree(t_full);
        checkCuda(hipMalloc((void **) &t_full, new_t_full_size * sizeof(Entity)));
        hipMemcpy(t_full, new_t_full, new_t_full_size * sizeof(Entity), hipMemcpyDeviceToDevice);
        t_full_size = new_t_full_size;
        hipFree(join_result);
        hipFree(new_t_full);
        hipFree(t_delta_temp);
        // Check if the global t full size has changed in this iteration
        long long old_global_t_full_size = global_t_full_size;
        MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_LONG_LONG, MPI_SUM, MPI_COMM_WORLD);
        iterations++;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;
        if (old_global_t_full_size == global_t_full_size) {
            break;
        }
    }

    start_time = MPI_Wtime();
    // Reverse the t_full as we stored it in reverse order initially
    reverse_entity_ar<<<grid_size, block_size>>>(t_full, t_full_size, t_full);
    // show_device_entity_variable(t_full, t_full_size, rank, "t_full", 0);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    // Scatter t_full among relevant processes
    int t_full_distributed_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    Entity *t_full_distributed = get_split_relation(rank, t_full,
                                                    t_full_size, total_columns, total_rank,
                                                    grid_size, block_size, cuda_aware_mpi, &t_full_distributed_size,
                                                    comm_method,
                                                    &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;
    start_time = MPI_Wtime();
    // Deduplicate scattered t full
    thrust::stable_sort(thrust::device, t_full_distributed, t_full_distributed + t_full_distributed_size, set_cmp());
    t_full_distributed_size = (thrust::unique(thrust::device,
                                              t_full_distributed, t_full_distributed + t_full_distributed_size,
                                              is_equal())) - t_full_distributed;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    start_time = MPI_Wtime();
    int *keys, *values;
    checkCuda(hipMalloc((void **) &keys, t_full_distributed_size * sizeof(int)));
    checkCuda(hipMalloc((void **) &values, t_full_distributed_size * sizeof(int)));

    int *keys_reduced, *values_reduced;
    checkCuda(hipMalloc((void **) &keys_reduced, t_full_distributed_size * sizeof(int)));
    checkCuda(hipMalloc((void **) &values_reduced, t_full_distributed_size * sizeof(int)));

    // Extract keys from the t_full_distributed array
    thrust::transform(thrust::device,
                      t_full_distributed, t_full_distributed + t_full_distributed_size, keys,
                      [] __device__(const Entity &e) { return e.key; });

    // Extract values from the t_full_distributed array
    thrust::transform(thrust::device,
                      t_full_distributed, t_full_distributed + t_full_distributed_size, values,
                      [] __device__(const Entity &e) { return e.value; });

    thrust::equal_to<int> binary_pred;

    // Reduce by key
    auto reduce_end = thrust::reduce_by_key(thrust::device,
                                            keys, keys + t_full_distributed_size,  // Keys input range
                                            values,                                // Values input range
                                            keys_reduced,                          // Keys output range
                                            values_reduced,                        // Values output range
                                            binary_pred,                           // Binary predicate for key comparison
                                            minimum_value()                        // Binary operator for value reduction
    );

    // Calculate the number of reduced values, reduce by minimum
    int number_of_values = thrust::distance(values_reduced, reduce_end.second);

    Entity *component;
    checkCuda(hipMalloc((void **) &component, number_of_values * sizeof(Entity)));

    // Create Entity array from values
    get_valueless_entity_ar_from_int_ar<<<grid_size, block_size>>>(values_reduced, number_of_values, component);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    // Scatter component among relevant processes
    int component_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    Entity *component_distributed = get_split_relation(rank, component,
                                                       number_of_values, total_columns, total_rank,
                                                       grid_size, block_size, cuda_aware_mpi, &component_size,
                                                       comm_method,
                                                       &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;
    start_time = MPI_Wtime();
    // Deduplicate components
    thrust::stable_sort(thrust::device, component_distributed, component_distributed + component_size, set_cmp());
    component_size = (thrust::unique(thrust::device,
                                     component_distributed, component_distributed + component_size,
                                     is_equal())) - component_distributed;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    start_time = MPI_Wtime();
    long long global_component_size = 0;
    long long current_component_size = component_size;
    MPI_Allreduce(&current_component_size, &global_component_size, 1, MPI_LONG_LONG, MPI_SUM, MPI_COMM_WORLD);


    int *component_ar;
    checkCuda(hipMalloc((void **) &component_ar, component_size * total_columns * sizeof(int)));
    get_int_ar_from_entity_ar<<<grid_size, block_size>>>(component_distributed, component_size, component_ar);

    // Copy component ar to host for file write
    int *component_ar_host = (int *) malloc(component_size * total_columns * sizeof(int));
    hipMemcpy(component_ar_host, component_ar, component_size * total_columns * sizeof(int), hipMemcpyDeviceToHost);

    // List the t full counts for each process and calculate the displacements in the final result
    int *component_counts = (int *) calloc(total_rank, sizeof(int));
    MPI_Allgather(&component_size, 1, MPI_INT, component_counts, 1, MPI_INT, MPI_COMM_WORLD);

    int *component_displacements = (int *) calloc(total_rank, sizeof(int));
    for (i = 1; i < total_rank; i++) {
        component_displacements[i] = component_displacements[i - 1] + (component_counts[i - 1] * total_columns);
    }
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    if (job_run == 0) {
        // Write the t full to an offset of the output file
        start_time = MPI_Wtime();
        MPI_File fh;
        MPI_File_open(MPI_COMM_WORLD, output_file_name, MPI_MODE_WRONLY | MPI_MODE_CREATE, MPI_INFO_NULL, &fh);
        int file_offset = component_displacements[rank] * sizeof(int);
        MPI_File_write_at(fh, file_offset, component_ar_host, component_size * total_columns, MPI_INT, MPI_STATUS_IGNORE);
        // Close the file and clean up
        MPI_File_close(&fh);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        file_io_time += elapsed_time;
    }

    start_time = MPI_Wtime();
    hipFree(input_relation);
    hipFree(local_data_reverse_temp_device);
    hipFree(local_data_temp_device);
    hipFree(local_data);
    hipFree(local_data_reverse);
    hipFree(t_full);
    hipFree(t_delta);
    hipFree(component_ar);
    hipFree(hash_table);
    free(component_ar_host);
    free(component_counts);
    free(component_displacements);
    free(local_data_host);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;
    MPI_Allreduce(&initialization_time, &max_initialization_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&deduplication_time, &max_deduplication_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&join_time, &max_join_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&merge_time, &max_merge_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&buffer_preparation_time, &max_buffer_preparation_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&communication_time, &max_communication_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&file_io_time, &max_fileio_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&finalization_time, &max_finalization_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    total_time = max_initialization_time + max_hashtable_build_time + max_join_time +
                 max_buffer_preparation_time + max_communication_time + max_deduplication_time + max_merge_time +
                 max_finalization_time;
    MPI_Allreduce(&total_time, &max_total_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    if (rank == 0) {
        output.block_size = block_size;
        output.grid_size = grid_size;
        output.input_rows = total_rows;
        output.total_rank = total_rank;
        output.iterations = iterations;
        output.output_file_name = output_file_name;
        output.output_size = global_component_size;

        output.total_time = max_total_time;
        output.initialization_time = max_initialization_time;
        output.fileio_time = max_fileio_time;
        output.hashtable_build_time = max_hashtable_build_time;
        output.join_time = max_join_time;
        output.buffer_preparation_time = max_buffer_preparation_time;
        output.communication_time = max_communication_time;
        output.deduplication_time = max_deduplication_time;
        output.merge_time = max_merge_time;
        output.finalization_time = max_finalization_time;
        if (job_run == 0) {
            printf("| # Input | # Process | # Iterations | # CC | Total Time ");
            printf("| Initialization | File I/O | Hashtable | Join | Buffer preparation | Communication | Deduplication | Merge | Finalization | Output |\n");
            printf("| --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- |\n");
        }
        printf("| %'d | %'d | %'d | %'lld | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %s |\n",
               output.input_rows, output.total_rank, output.iterations,
               output.output_size, output.total_time,
               output.initialization_time, output.fileio_time, output.hashtable_build_time, output.join_time,
               output.buffer_preparation_time, output.communication_time, output.deduplication_time, output.merge_time,
               output.finalization_time,
               output.output_file_name);
    }
    MPI_Finalize();
}

int main(int argc, char **argv) {
    benchmark(argc, argv);
    return 0;
}
// METHOD 0 = two pass method, 1 = sorting method
// make runcc DATA_FILE=data/dummy.bin NPROCS=1 CUDA_AWARE_MPI=0 METHOD=0
// make runcc DATA_FILE=data/dummy.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/data_10.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=1
