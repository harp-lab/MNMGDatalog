#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/iterator/transform_iterator.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/kernels.cu"
#include "common/comm.cu"
#include "common/join.cu"

using namespace std;

/*
Base rule:
edge(x,y) ← edge(y,x).
cc(n, n) ← edge(n,_).
t_delta(x, y) ← cc(x, y)


Recursive rule:
join_result(z, x) ← t_delta(y, z), edge(x, y).
join_result(z, x) ← join_result(x, z).
cc(y, min(z)) ← cc(y, z) U join_result(z, x)
t_delta(x, y) ← cc(y, z) - cc_old(y, z)

Final rule:
cc_representative_node(n) ← cc(_ , n).
*/
void benchmark(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    Output output;
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    setlocale(LC_ALL, "");
    double start_time, end_time, elapsed_time;
    double initialization_time = 0.0, max_initialization_time = 0.0;
    double finalization_time = 0.0, max_finalization_time = 0.0;
    double file_io_time = 0.0, max_fileio_time = 0.0;
    double max_join_time = 0.0, max_merge_time = 0.0;
    double max_buffer_preparation_time = 0.0, max_communication_time = 0.0;
    double buffer_preparation_time = 0.0, communication_time = 0.0;
    double buffer_preparation_time_temp = 0.0, communication_time_temp = 0.0;
    double join_time = 0.0, merge_time = 0.0;
    double deduplication_time = 0.0, max_deduplication_time = 0.0;;
    double hashtable_build_time = 0.0, max_hashtable_build_time = 0.0;

    double total_time = 0.0, max_total_time = 0.0;
    int total_rank, rank;
    int i;
    MPI_Comm_size(MPI_COMM_WORLD, &total_rank);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    warm_up_kernel<<<1, 1>>>();
    int iterations = 0;
    // Should pass the input filename in command line argument
    const char *input_file;
    int comm_method = 0;
    int job_run = 0;
    int cuda_aware_mpi = 0;

    if (argc == 5) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
        job_run = atoi(argv[4]);
    } else if (argc == 4) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
    } else if (argc == 3) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
    } else if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "hipc_2019.bin";
    }
    string output_file = string(input_file) + "_cc.bin";
    const char *output_file_name = output_file.c_str();

    // READ THE FILE IN PARALLEL
    // Reading filesize in bytes
    start_time = MPI_Wtime();
    struct stat filestats{};
    stat(input_file, &filestats);
    off_t filesize = filestats.st_size;

    // Calculating the current rank's starting row and number of rows
    // Scatter larger blocks among processes (non-uniform)
    int total_columns = 2;
    int total_rows = filesize / (sizeof(int) * total_columns);
    int row_start = BLOCK_START(rank, total_rank, total_rows);
    int row_size = BLOCK_SIZE(rank, total_rank, total_rows);
    int local_count = row_size * total_columns;

    // Reading specific portion from the file as char in parallel
    int offset = row_start * total_columns * sizeof(int);
    int *local_data_host = (int *) malloc(local_count * sizeof(int));
    MPI_File mpi_file_buffer;
    if (MPI_File_open(MPI_COMM_WORLD, input_file, MPI_MODE_RDONLY,
                      MPI_INFO_NULL, &mpi_file_buffer) != MPI_SUCCESS) {
        printf("Error opening file %s", input_file);
        MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    }
    MPI_File_read_at(mpi_file_buffer, offset, local_data_host, local_count, MPI_INT, MPI_STATUS_IGNORE);
    MPI_File_close(&mpi_file_buffer);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    file_io_time += elapsed_time;
    start_time = MPI_Wtime();
    int *local_data_temp_device;
    checkCuda(hipMalloc((void **) &local_data_temp_device, local_count * sizeof(int)));
    hipMemcpy(local_data_temp_device, local_data_host, local_count * sizeof(int), hipMemcpyHostToDevice);
    // Ensure edges are bidirectional by adding reverse edges
    int *local_data_reverse_temp_device;
    checkCuda(hipMalloc((void **) &local_data_reverse_temp_device, local_count * sizeof(int)));
    get_reverse_ar<<<grid_size, block_size>>>(local_data_temp_device, row_size, local_data_reverse_temp_device);

    // Create Entity array from GPU buffers, edge + reverse_edge
    // edge(x, y) :- edge(y, x)
    Entity *local_data;
    int local_data_size = local_count;
    checkCuda(hipMalloc((void **) &local_data, local_data_size * sizeof(Entity)));
    create_entity_ar_with_offset<<<grid_size, block_size>>>(local_data_temp_device, row_size, local_data, 0);
    create_entity_ar_with_offset<<<grid_size, block_size>>>(local_data_reverse_temp_device, row_size,
                                                            local_data, row_size);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    // Deduplicate local data
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, local_data, local_data + local_data_size, set_cmp());
    local_data_size = (thrust::unique(thrust::device,
                                      local_data, local_data + local_data_size,
                                      is_equal())) - local_data;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    // Create cc from local data
    // cc(x, x) :- edge(x, _)
    start_time = MPI_Wtime();
    Entity *cc_base;
    int cc_base_size = local_data_size;
    checkCuda(hipMalloc((void **) &cc_base, cc_base_size * sizeof(Entity)));
    same_key_value_entity_ar<<<grid_size, block_size>>>(local_data, cc_base_size, cc_base);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    // Deduplicate cc
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, cc_base, cc_base + cc_base_size, set_cmp());
    cc_base_size = (thrust::unique(thrust::device,
                                   cc_base, cc_base + cc_base_size,
                                   is_equal())) - cc_base;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    // Distribute edge
    int input_relation_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    Entity *input_relation = get_split_relation(rank, local_data,
                                                local_data_size, total_columns, total_rank,
                                                grid_size, block_size, cuda_aware_mpi,
                                                &input_relation_size, comm_method,
                                                &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;

    // Distribute CC
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    int t_delta_size = 0;
    Entity *t_delta = get_split_relation(rank, cc_base,
                                         cc_base_size, total_columns, total_rank,
                                         grid_size, block_size, cuda_aware_mpi, &t_delta_size, comm_method,
                                         &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;

    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, t_delta, t_delta + t_delta_size, set_cmp());
    t_delta_size = (thrust::unique(thrust::device,
                                   t_delta, t_delta + t_delta_size,
                                   is_equal())) - t_delta;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    // cc = t delta, key-value pair: node - component id
    start_time = MPI_Wtime();
    Entity *cc;
    long long cc_size = t_delta_size;
    checkCuda(hipMalloc((void **) &cc, cc_size * sizeof(Entity)));
    hipMemcpy(cc, t_delta, cc_size * sizeof(Entity), hipMemcpyDeviceToDevice);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    start_time = MPI_Wtime();
    long long global_t_delta_size = 0;
    MPI_Allreduce(&t_delta_size, &global_t_delta_size, 1, MPI_LONG_LONG, MPI_SUM, MPI_COMM_WORLD);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    merge_time += elapsed_time;

    // Hash table is Edge
    start_time = MPI_Wtime();
    Entity *hash_table;
    double load_factor = 0.4;
    int hash_table_rows = (int) input_relation_size / load_factor;
    hash_table_rows = pow(2, ceil(log(hash_table_rows) / log(2)));
    checkCuda(hipMalloc((void **) &hash_table, hash_table_rows * sizeof(Entity)));
    Entity negative_entity;
    negative_entity.key = -1;
    negative_entity.value = -1;
    thrust::fill(thrust::device, hash_table, hash_table + hash_table_rows, negative_entity);
    build_hash_table_entity<<<grid_size, block_size>>>(hash_table, hash_table_rows, input_relation,
                                                       input_relation_size);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    hashtable_build_time += elapsed_time;

    while (true) {
        Entity *new_cc;
        double temp_join_time = 0.0;
        int join_result_size = 0;
        Entity *join_result = get_join(grid_size, block_size, hash_table, hash_table_rows,
                                       t_delta, t_delta_size,
                                       &join_result_size, &temp_join_time);
        join_time += temp_join_time;

        // Add the reverse of join results
        start_time = MPI_Wtime();
        Entity *join_result_with_reverse;
        int join_result_with_reverse_size = join_result_size * 2;
        checkCuda(hipMalloc((void **) &join_result_with_reverse, join_result_with_reverse_size * sizeof(Entity)));
        duplicate_entity_with_reverse<<<grid_size, block_size>>>(join_result, join_result_size,
                                                                 join_result_with_reverse);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        join_time += elapsed_time;


        // Scatter the join result with reverse among relevant processes
        buffer_preparation_time_temp = 0.0;
        communication_time_temp = 0.0;
        int distributed_join_result_size = 0;
        Entity *distributed_join_result = get_split_relation(rank, join_result_with_reverse,
                                                             join_result_with_reverse_size, total_columns, total_rank,
                                                             grid_size, block_size, cuda_aware_mpi,
                                                             &distributed_join_result_size,
                                                             comm_method,
                                                             &buffer_preparation_time_temp, &communication_time_temp);
        buffer_preparation_time += buffer_preparation_time_temp;
        communication_time += communication_time_temp;


        // Deduplicate distributed join result with reverse
        start_time = MPI_Wtime();
        thrust::stable_sort(thrust::device, distributed_join_result,
                            distributed_join_result + distributed_join_result_size, set_cmp());
        distributed_join_result_size = (thrust::unique(thrust::device,
                                                       distributed_join_result,
                                                       distributed_join_result + distributed_join_result_size,
                                                       is_equal())) - distributed_join_result;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        deduplication_time += elapsed_time;


        // Set union of two sets (sorted t full and distributed join result)
        start_time = MPI_Wtime();
        int new_cc_size = distributed_join_result_size + cc_size;
        checkCuda(hipMalloc((void **) &new_cc, new_cc_size * sizeof(Entity)));
        new_cc_size = thrust::set_union(thrust::device,
                                        cc, cc + cc_size,
                                        distributed_join_result,
                                            distributed_join_result + distributed_join_result_size,
                                        new_cc, set_cmp()) - new_cc;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;

        // Deduplicate new t full by keeping only the minimum component ID for each node
        start_time = MPI_Wtime();
        new_cc_size = (thrust::unique(thrust::device,
                                      new_cc,
                                      new_cc + new_cc_size,
                                      is_equal_key())) - new_cc;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        deduplication_time += elapsed_time;

        // Update t delta which is the only new facts which are not in t full and will be used in next iteration
        start_time = MPI_Wtime();
        t_delta_size = thrust::set_difference(thrust::device,
                                              new_cc, new_cc + new_cc_size,
                                              cc, cc + cc_size,
                                              t_delta, set_cmp()) - t_delta;
        // Update t full
        cc_size = new_cc_size;
        hipMemcpy(cc, new_cc, new_cc_size * sizeof(Entity), hipMemcpyDeviceToDevice);
        long long old_global_t_delta_size = global_t_delta_size;
        MPI_Allreduce(&t_delta_size, &global_t_delta_size, 1, MPI_LONG_LONG, MPI_SUM, MPI_COMM_WORLD);
        iterations++;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        hipFree(distributed_join_result);
        hipFree(new_cc);
        hipFree(join_result_with_reverse);
        merge_time += elapsed_time;
        if (old_global_t_delta_size == global_t_delta_size) {
            break;
        }
    }

    // We are interested only the unique component ID, thus we make the component ID as key and got rid of node
    start_time = MPI_Wtime();
    replace_key_by_value<<<grid_size, block_size>>>(cc, cc_size, cc);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    // Scatter component IDs among relevant processes
    int cc_distributed_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    Entity *cc_distributed = get_split_relation(rank, cc,
                                                cc_size, total_columns, total_rank,
                                                grid_size, block_size, cuda_aware_mpi, &cc_distributed_size,
                                                comm_method,
                                                &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;

    // Deduplicate scattered component IDs
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, cc_distributed, cc_distributed + cc_distributed_size, set_cmp());
    cc_distributed_size = (thrust::unique(thrust::device,
                                          cc_distributed, cc_distributed + cc_distributed_size,
                                          is_equal())) - cc_distributed;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    start_time = MPI_Wtime();
    long long global_component_size = 0;
    long long current_component_size = cc_distributed_size;
    MPI_Allreduce(&current_component_size, &global_component_size, 1, MPI_LONG_LONG, MPI_SUM, MPI_COMM_WORLD);

    int *component_ar;
    checkCuda(hipMalloc((void **) &component_ar, cc_distributed_size * total_columns * sizeof(int)));
    get_int_ar_from_entity_ar<<<grid_size, block_size>>>(cc_distributed, cc_distributed_size, component_ar);

    // Copy component ar to host for file write
    int *component_ar_host = (int *) malloc(cc_distributed_size * total_columns * sizeof(int));
    hipMemcpy(component_ar_host, component_ar, cc_distributed_size * total_columns * sizeof(int),
               hipMemcpyDeviceToHost);

    // List the t full counts for each process and calculate the displacements in the final result
    int *component_counts = (int *) calloc(total_rank, sizeof(int));
    MPI_Allgather(&cc_distributed_size, 1, MPI_INT, component_counts, 1, MPI_INT, MPI_COMM_WORLD);

    int *component_displacements = (int *) calloc(total_rank, sizeof(int));
    for (i = 1; i < total_rank; i++) {
        component_displacements[i] = component_displacements[i - 1] + (component_counts[i - 1] * total_columns);
    }
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    if (job_run == 0) {
        // Write the t full to an offset of the output file
        start_time = MPI_Wtime();
        MPI_File fh;
        MPI_File_open(MPI_COMM_WORLD, output_file_name, MPI_MODE_WRONLY | MPI_MODE_CREATE, MPI_INFO_NULL, &fh);
        int file_offset = component_displacements[rank] * sizeof(int);
        MPI_File_write_at(fh, file_offset, component_ar_host, cc_distributed_size * total_columns, MPI_INT,
                          MPI_STATUS_IGNORE);
        // Close the file and clean up
        MPI_File_close(&fh);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        file_io_time += elapsed_time;
    }

    start_time = MPI_Wtime();
    hipFree(input_relation);
    hipFree(local_data_reverse_temp_device);
    hipFree(local_data_temp_device);
    hipFree(local_data);
    hipFree(cc_base);
    hipFree(cc);
    hipFree(t_delta);
    hipFree(component_ar);
    hipFree(hash_table);
    hipFree(cc_distributed);
    free(component_ar_host);
    free(component_counts);
    free(component_displacements);
    free(local_data_host);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;
    MPI_Allreduce(&initialization_time, &max_initialization_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&hashtable_build_time, &max_hashtable_build_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&deduplication_time, &max_deduplication_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&join_time, &max_join_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&merge_time, &max_merge_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&buffer_preparation_time, &max_buffer_preparation_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&communication_time, &max_communication_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&file_io_time, &max_fileio_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&finalization_time, &max_finalization_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    total_time = max_initialization_time + max_hashtable_build_time + max_join_time +
                 max_buffer_preparation_time + max_communication_time + max_deduplication_time + max_merge_time +
                 max_finalization_time;
    MPI_Allreduce(&total_time, &max_total_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    if (rank == 0) {
        output.block_size = block_size;
        output.grid_size = grid_size;
        output.input_rows = total_rows;
        output.total_rank = total_rank;
        output.iterations = iterations;
        output.output_file_name = output_file_name;
        output.output_size = global_component_size;
        output.total_time = max_total_time;
        output.initialization_time = max_initialization_time;
        output.fileio_time = max_fileio_time;
        output.hashtable_build_time = max_hashtable_build_time;
        output.join_time = max_join_time;
        output.buffer_preparation_time = max_buffer_preparation_time;
        output.communication_time = max_communication_time;
        output.deduplication_time = max_deduplication_time;
        output.merge_time = max_merge_time;
        output.finalization_time = max_finalization_time;
        if (job_run == 0) {
            printf("| # Input | # Process | # Iterations | # CC | Total Time ");
            printf("| Initialization | File I/O | Hashtable | Join | Buffer preparation | Communication | Deduplication | Merge | Finalization | Output |\n");
            printf("| --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- |\n");
        }
        printf("| %'d | %'d | %'d | %'lld | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %s |\n",
               output.input_rows, output.total_rank, output.iterations,
               output.output_size, output.total_time,
               output.initialization_time, output.fileio_time, output.hashtable_build_time, output.join_time,
               output.buffer_preparation_time, output.communication_time, output.deduplication_time, output.merge_time,
               output.finalization_time,
               output.output_file_name);
    }
    MPI_Finalize();
}

int main(int argc, char **argv) {
    benchmark(argc, argv);
    return 0;
}
// METHOD 0 = two pass method, 1 = sorting method
// make runcc DATA_FILE=data/dummy.bin NPROCS=1 CUDA_AWARE_MPI=0 METHOD=0
// make runcc DATA_FILE=data/dummy.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runcc DATA_FILE=data/data_214078.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runcc DATA_FILE=data/data_214078.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=1