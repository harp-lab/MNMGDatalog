#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/iterator/transform_iterator.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/kernels.cu"
#include "common/comm.cu"
#include "common/join.cu"

using namespace std;

/*
Base rule:
edge(x,y) ← edge(y,x).
cc(n, n) ← edge(n,_).
t_delta(x, y) ← cc(x, y)

Recursive rule:
join_result(z, x) ← t_delta(y, z), edge(x, y).
join_result(z, x) ← join_result(x, z).
cc_new(y, min(z)) ← cc(y, z) U join_result(z, x)
t_delta(x, y) ← cc_new(y, z) - cc_old(y, z)
cc(x, y) ← cc_new(x, y)

Final rule:
cc_representative_node(n) ← cc(_ , n).
*/
void benchmark(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    Output output;
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    setlocale(LC_ALL, "");
    double start_time, end_time, elapsed_time;
    double initialization_time = 0.0, max_initialization_time = 0.0;
    double finalization_time = 0.0, max_finalization_time = 0.0;
    double file_io_time = 0.0, max_fileio_time = 0.0;
    double max_join_time = 0.0, max_merge_time = 0.0;
    double max_buffer_preparation_time = 0.0, max_communication_time = 0.0;
    double buffer_preparation_time = 0.0, communication_time = 0.0;
    double buffer_preparation_time_temp = 0.0, communication_time_temp = 0.0;
    double join_time = 0.0, merge_time = 0.0;
    double deduplication_time = 0.0, max_deduplication_time = 0.0;;
    double hashtable_build_time = 0.0, max_hashtable_build_time = 0.0;

    double total_time = 0.0, max_total_time = 0.0;
    int total_rank, rank;
    int i;
    MPI_Comm_size(MPI_COMM_WORLD, &total_rank);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    warm_up_kernel<<<1, 1>>>();
    int iterations = 0;
    // Should pass the input filename in command line argument
    const char *input_file;
    int comm_method = 0;
    int job_run = 0;
    int cuda_aware_mpi = 0;

    if (argc == 5) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
        job_run = atoi(argv[4]);
    } else if (argc == 4) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
    } else if (argc == 3) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
    } else if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "hipc_2019.bin";
    }
    string output_file = string(input_file) + "_cc.bin";
    const char *output_file_name = output_file.c_str();

    // READ THE FILE IN PARALLEL
    // Reading filesize in bytes
    start_time = MPI_Wtime();
    struct stat filestats{};
    stat(input_file, &filestats);
    off_t filesize = filestats.st_size;

    // Calculating the current rank's starting row and number of rows
    // Scatter larger blocks among processes (non-uniform)
    int total_columns = 2;
    unsigned long total_rows = filesize / (sizeof(int) * total_columns);
    int row_start = BLOCK_START(rank, total_rank, total_rows);
    int row_size = BLOCK_SIZE(rank, total_rank, total_rows);
    int local_count = row_size * total_columns;

    // Reading specific portion from the file as char in parallel
    long long offset = row_start * total_columns * sizeof(int);
    int *edge_host = (int *) malloc(local_count * sizeof(int));
    MPI_File mpi_file_buffer;
    if (MPI_File_open(MPI_COMM_WORLD, input_file, MPI_MODE_RDONLY,
                      MPI_INFO_NULL, &mpi_file_buffer) != MPI_SUCCESS) {
        printf("Error opening file %s", input_file);
        MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    }
    MPI_File_read_at(mpi_file_buffer, offset, edge_host, local_count, MPI_INT, MPI_STATUS_IGNORE);
    MPI_File_close(&mpi_file_buffer);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    file_io_time += elapsed_time;
    start_time = MPI_Wtime();
    int *edge_temp_device;
    checkCuda(hipMalloc((void **) &edge_temp_device, local_count * sizeof(int)));
    hipMemcpy(edge_temp_device, edge_host, local_count * sizeof(int), hipMemcpyHostToDevice);
    // Ensure edges are bidirectional by adding reverse edges
    int *edge_reverse_temp_device;
    checkCuda(hipMalloc((void **) &edge_reverse_temp_device, local_count * sizeof(int)));
    get_reverse_ar<<<grid_size, block_size>>>(edge_temp_device, row_size, edge_reverse_temp_device);

    // Create Entity array from GPU buffers, edge + reverse_edge
    // edge(x, y) :- edge(y, x)
    Entity *edge;
    long edge_size = local_count;
    checkCuda(hipMalloc((void **) &edge, edge_size * sizeof(Entity)));
    create_entity_ar_with_offset<<<grid_size, block_size>>>(edge_temp_device, row_size, edge, 0);
    create_entity_ar_with_offset<<<grid_size, block_size>>>(edge_reverse_temp_device, row_size,
                                                            edge, row_size);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    // Deduplicate local data
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, edge, edge + edge_size, set_cmp());
    edge_size = (thrust::unique(thrust::device,
                                edge, edge + edge_size,
                                is_equal())) - edge;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    // Create cc from edge where node, component_id = node, node
    // cc(x, x) :- edge(x, _)
    start_time = MPI_Wtime();
    Entity *cc_base;
    long cc_base_size = edge_size;
    checkCuda(hipMalloc((void **) &cc_base, cc_base_size * sizeof(Entity)));
    same_key_value_entity_ar<<<grid_size, block_size>>>(edge, cc_base_size, cc_base);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    // Deduplicate cc
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, cc_base, cc_base + cc_base_size, set_cmp());
    cc_base_size = (thrust::unique(thrust::device,
                                   cc_base, cc_base + cc_base_size,
                                   is_equal_key())) - cc_base;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    // Distribute edge
    int distributed_edge_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    Entity *distributed_edge = get_split_relation(rank, edge,
                                                  edge_size, total_columns, total_rank,
                                                  grid_size, block_size, cuda_aware_mpi,
                                                  &distributed_edge_size, comm_method,
                                                  &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;

    // Distribute cc created from base rule
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    int t_delta_size = 0;
    Entity *t_delta = get_split_relation(rank, cc_base,
                                         cc_base_size, total_columns, total_rank,
                                         grid_size, block_size, cuda_aware_mpi, &t_delta_size, comm_method,
                                         &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;


    // Deduplicate distributed cc
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, t_delta, t_delta + t_delta_size, set_cmp());
    t_delta_size = (thrust::unique(thrust::device,
                                   t_delta, t_delta + t_delta_size,
                                   is_equal())) - t_delta;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    // cc = t delta, key-value pair: node - component id
    start_time = MPI_Wtime();
    Entity *cc;
    long long cc_size = t_delta_size;
    checkCuda(hipMalloc((void **) &cc, cc_size * sizeof(Entity)));
    hipMemcpy(cc, t_delta, cc_size * sizeof(Entity), hipMemcpyDeviceToDevice);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    start_time = MPI_Wtime();
    long long global_t_delta_size = 0;
    MPI_Allreduce(&t_delta_size, &global_t_delta_size, 1, MPI_LONG_LONG, MPI_SUM, MPI_COMM_WORLD);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    // Hash table is Edge
    start_time = MPI_Wtime();
    Entity *hash_table;
    double load_factor = 0.4;
    int hash_table_rows = (int) distributed_edge_size / load_factor;
    hash_table_rows = pow(2, ceil(log(hash_table_rows) / log(2)));
    checkCuda(hipMalloc((void **) &hash_table, hash_table_rows * sizeof(Entity)));
    Entity negative_entity;
    negative_entity.key = -1;
    negative_entity.value = -1;
    thrust::fill(thrust::device, hash_table, hash_table + hash_table_rows, negative_entity);
    build_hash_table_entity<<<grid_size, block_size>>>(hash_table, hash_table_rows, distributed_edge,
                                                       distributed_edge_size);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    hashtable_build_time += elapsed_time;

    while (true) {
        Entity *new_cc;
        double temp_join_time = 0.0;
        int join_result_with_reverse_size = 0;
        Entity *join_result_with_reverse = get_join_with_reverse(grid_size, block_size, hash_table, hash_table_rows,
                                       t_delta, t_delta_size,
                                       &join_result_with_reverse_size, &temp_join_time);
        join_time += temp_join_time;

        // Scatter the join result with reverse among relevant processes
        buffer_preparation_time_temp = 0.0;
        communication_time_temp = 0.0;
        int distributed_join_result_size = 0;
        Entity *distributed_join_result = get_split_relation(rank, join_result_with_reverse,
                                                             join_result_with_reverse_size, total_columns, total_rank,
                                                             grid_size, block_size, cuda_aware_mpi,
                                                             &distributed_join_result_size,
                                                             comm_method,
                                                             &buffer_preparation_time_temp, &communication_time_temp);
        buffer_preparation_time += buffer_preparation_time_temp;
        communication_time += communication_time_temp;


        // Deduplicate distributed join result with reverse
        start_time = MPI_Wtime();
        thrust::stable_sort(thrust::device, distributed_join_result,
                            distributed_join_result + distributed_join_result_size, set_cmp());
        distributed_join_result_size = (thrust::unique(thrust::device,
                                                       distributed_join_result,
                                                       distributed_join_result + distributed_join_result_size,
                                                       is_equal())) - distributed_join_result;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        deduplication_time += elapsed_time;


        // Set union of two sets (sorted cc and distributed join result)
        start_time = MPI_Wtime();
        int new_cc_size = distributed_join_result_size + cc_size;
        checkCuda(hipMalloc((void **) &new_cc, new_cc_size * sizeof(Entity)));
        new_cc_size = thrust::set_union(thrust::device,
                                        cc, cc + cc_size,
                                        distributed_join_result,
                                        distributed_join_result + distributed_join_result_size,
                                        new_cc, set_cmp()) - new_cc;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;

        // Deduplicate new cc by keeping only the minimum component ID for each node
        start_time = MPI_Wtime();
        new_cc_size = (thrust::unique(thrust::device,
                                      new_cc,
                                      new_cc + new_cc_size,
                                      is_equal_key())) - new_cc;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        deduplication_time += elapsed_time;

        // Update t delta which is the only new facts which are not in cc and will be used in next iteration
        start_time = MPI_Wtime();
        Entity *t_delta_temp;
        checkCuda(hipMalloc((void **) &t_delta_temp, new_cc_size * sizeof(Entity)));

        t_delta_size = thrust::set_difference(thrust::device,
                                              new_cc, new_cc + new_cc_size,
                                              cc, cc + cc_size,
                                              t_delta_temp, set_cmp()) - t_delta_temp;
        hipFree(t_delta);
        checkCuda(hipMalloc((void **) &t_delta, t_delta_size * sizeof(Entity)));
        hipMemcpy(t_delta, t_delta_temp, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);
        // Update cc
        cc_size = new_cc_size;
        hipMemcpy(cc, new_cc, cc_size * sizeof(Entity), hipMemcpyDeviceToDevice);
        long long old_global_t_delta_size = global_t_delta_size;
        MPI_Allreduce(&t_delta_size, &global_t_delta_size, 1, MPI_LONG_LONG, MPI_SUM, MPI_COMM_WORLD);
        iterations++;
        hipFree(distributed_join_result);
        hipFree(new_cc);
        hipFree(join_result_with_reverse);
        hipFree(t_delta_temp);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;
        if (old_global_t_delta_size == global_t_delta_size) {
            break;
        }
    }

    // We are interested only the unique component ID, thus we make the component ID as key and got rid of node
    start_time = MPI_Wtime();
    replace_key_by_value<<<grid_size, block_size>>>(cc, cc_size, cc);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    // Scatter component IDs among relevant processes
    int cc_distributed_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    Entity *cc_distributed = get_split_relation(rank, cc,
                                                cc_size, total_columns, total_rank,
                                                grid_size, block_size, cuda_aware_mpi, &cc_distributed_size,
                                                comm_method,
                                                &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;

    // Sort scattered component IDs
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, cc_distributed, cc_distributed + cc_distributed_size, set_cmp());
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;


    // Calculate Nodes in largest WCC
    int *component_ids;
    checkCuda(hipMalloc((void **) &component_ids, cc_distributed_size * sizeof(int)));
    int *unique_component_ids, *component_sizes;
    checkCuda(hipMalloc((void **) &unique_component_ids, cc_distributed_size * sizeof(int)));
    checkCuda(hipMalloc((void **) &component_sizes, cc_distributed_size * sizeof(int)));
    // Extract keys from the cc_distributed array
    thrust::transform(thrust::device,
                      cc_distributed, cc_distributed + cc_distributed_size, component_ids,
                      [] __device__(const Entity &e) { return e.key; });
    thrust::equal_to<int> binary_pred;
    // Count the occurrences of each component ID
    auto reduce_end = thrust::reduce_by_key(thrust::device,
                                            component_ids, component_ids + cc_distributed_size,
                                            thrust::constant_iterator<int>(1),
                                            unique_component_ids,
                                            component_sizes, binary_pred);
    // Calculate the number of total unique compoennt
    long long total_unique_component = thrust::distance(component_sizes, reduce_end.second);
    // Find the largest component size
    long long max_component_size_current_rank = thrust::reduce(thrust::device,
                                                               component_sizes, component_sizes + total_unique_component, -1,
                                                               thrust::maximum<int>());
    long long max_component_size = 0;
    MPI_Allreduce(&max_component_size_current_rank, &max_component_size, 1, MPI_LONG_LONG, MPI_MAX, MPI_COMM_WORLD);

    // Deduplicate component IDs
    start_time = MPI_Wtime();
    cc_distributed_size = (thrust::unique(thrust::device,
                                          cc_distributed, cc_distributed + cc_distributed_size,
                                          is_equal_key())) - cc_distributed;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    start_time = MPI_Wtime();
    long long global_component_size = 0;
    long long current_component_size = cc_distributed_size;
    MPI_Allreduce(&current_component_size, &global_component_size, 1, MPI_LONG_LONG, MPI_SUM, MPI_COMM_WORLD);

    int *component_ar;
    checkCuda(hipMalloc((void **) &component_ar, cc_distributed_size * total_columns * sizeof(int)));
    get_int_ar_from_entity_ar<<<grid_size, block_size>>>(cc_distributed, cc_distributed_size, component_ar);

    // Copy component ar to host for file write
    int *component_ar_host = (int *) malloc(cc_distributed_size * total_columns * sizeof(int));
    hipMemcpy(component_ar_host, component_ar, cc_distributed_size * total_columns * sizeof(int),
               hipMemcpyDeviceToHost);

    // List the cc counts for each process and calculate the displacements in the final result
    int *component_counts = (int *) calloc(total_rank, sizeof(int));
    MPI_Allgather(&cc_distributed_size, 1, MPI_INT, component_counts, 1, MPI_INT, MPI_COMM_WORLD);

    int *component_displacements = (int *) calloc(total_rank, sizeof(int));
    for (i = 1; i < total_rank; i++) {
        component_displacements[i] = component_displacements[i - 1] + (component_counts[i - 1] * total_columns);
    }
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    if (job_run == 0) {
        // Write the cc to an offset of the output file
        start_time = MPI_Wtime();
        MPI_File fh;
        MPI_File_open(MPI_COMM_WORLD, output_file_name, MPI_MODE_WRONLY | MPI_MODE_CREATE, MPI_INFO_NULL, &fh);
        int file_offset = component_displacements[rank] * sizeof(int);
        MPI_File_write_at(fh, file_offset, component_ar_host, cc_distributed_size * total_columns, MPI_INT,
                          MPI_STATUS_IGNORE);
        // Close the file and clean up
        MPI_File_close(&fh);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        file_io_time += elapsed_time;
    }

    start_time = MPI_Wtime();

    hipFree(distributed_edge);
    hipFree(edge_reverse_temp_device);
    hipFree(edge_temp_device);
    hipFree(edge);
    hipFree(cc_base);
    hipFree(cc);
    hipFree(t_delta);
    hipFree(component_ar);
    hipFree(hash_table);
    hipFree(cc_distributed);
    hipFree(component_ids);
    hipFree(unique_component_ids);
    hipFree(component_sizes);
    free(component_ar_host);
    free(component_counts);
    free(component_displacements);
    free(edge_host);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;
    MPI_Allreduce(&initialization_time, &max_initialization_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&hashtable_build_time, &max_hashtable_build_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&deduplication_time, &max_deduplication_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&join_time, &max_join_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&merge_time, &max_merge_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&buffer_preparation_time, &max_buffer_preparation_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&communication_time, &max_communication_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&file_io_time, &max_fileio_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&finalization_time, &max_finalization_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    total_time = max_initialization_time + max_hashtable_build_time + max_join_time +
                 max_buffer_preparation_time + max_communication_time + max_deduplication_time + max_merge_time +
                 max_finalization_time;
    MPI_Allreduce(&total_time, &max_total_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    if (rank == 0) {
        output.block_size = block_size;
        output.grid_size = grid_size;
        output.input_rows = total_rows;
        output.total_rank = total_rank;
        output.iterations = iterations;
        output.output_file_name = output_file_name;
        output.output_size = global_component_size;
        output.output_size_secondary = max_component_size;
        output.total_time = max_total_time;
        output.initialization_time = max_initialization_time;
        output.fileio_time = max_fileio_time;
        output.hashtable_build_time = max_hashtable_build_time;
        output.join_time = max_join_time;
        output.buffer_preparation_time = max_buffer_preparation_time;
        output.communication_time = max_communication_time;
        output.deduplication_time = max_deduplication_time;
        output.merge_time = max_merge_time;
        output.finalization_time = max_finalization_time;
        if (job_run == 0) {
            printf("| # Input | # Process | # Iterations | # CC (# Nodes in largest WCC) | Total Time ");
            printf("| Initialization | File I/O | Hashtable | Join | Buffer preparation | Communication | Deduplication | Merge | Finalization | Output |\n");
            printf("| --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- |\n");
        }
        printf("| %'lu | %'d | %'d | %'lld (%'lld) | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %s |\n",
               output.input_rows, output.total_rank, output.iterations,
               output.output_size, output.output_size_secondary, output.total_time,
               output.initialization_time, output.fileio_time, output.hashtable_build_time, output.join_time,
               output.buffer_preparation_time, output.communication_time, output.deduplication_time, output.merge_time,
               output.finalization_time,
               output.output_file_name);
    }
    MPI_Finalize();
}

int main(int argc, char **argv) {
    benchmark(argc, argv);
    return 0;
}
// METHOD 0 = two pass method, 1 = sorting method
// make runcc DATA_FILE=data/dummy.bin NPROCS=1 CUDA_AWARE_MPI=0 METHOD=0
// make runcc DATA_FILE=data/dummy.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runcc DATA_FILE=data/flickr.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runcc DATA_FILE=data/data_214078.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runcc DATA_FILE=data/data_214078.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=1
// make runcc DATA_FILE=data/web-Stanford.bin NPROCS=1 CUDA_AWARE_MPI=0 METHOD=0