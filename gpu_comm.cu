#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <fstream>
#include <iostream>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/pair.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/unique.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/kernels.cu"

using namespace std;

int get_edge_count(const char *data_path) {
    std::ifstream f;
    f.open(data_path);
    char c;
    int i = 0;
    while (f.get(c)) {
        if (c == '\n') {
            i++;
        }
    }
    f.close();
    return i;
}

void *get_data_from_file(const char *file_path, int *data, int total_rows,
                         int total_columns, char separator) {
    FILE *data_file = fopen(file_path, "r");
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            if (j != (total_columns - 1)) {
                fscanf(data_file, "%d%c", &data[(i * total_columns) + j], &separator);
            } else {
                fscanf(data_file, "%d", &data[(i * total_columns) + j]);
            }
        }
    }
}

/**
 * @brief fnv1-a hash used in original slog backend
 *
 * @param start_ptr
 * @param prefix_len
 * @return __host__ __device__
 */
__host__ __device__ inline unsigned int prefix_hash(int *data, int start_index,
                                                    int prefix_len) {
    const unsigned int base = 2166136261U;
    const unsigned int prime = 16777619U;

    unsigned int hash = base;
    for (int i = 0; i < prefix_len; i++) {
        int chunk = data[start_index + i];
        hash ^= chunk & 255U;
        hash *= prime;
        for (int j = 0; j < 3; j++) {
            chunk = chunk >> 8;
            hash ^= chunk & 255U;
            hash *= prime;
        }
    }
    return hash;
}

__global__
void get_rank_data_size(int *local_data, int local_data_row_count,
                        int *rank_data_size, int nprocs) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= local_data_row_count) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < local_data_row_count; i += stride) {
        int key = local_data[i * 2];
        int destination_rank = key % nprocs;
        atomicAdd(&rank_data_size[destination_rank], 2);
    }
}

__global__
void get_rank_data(int *local_data, int local_data_row_count,
                   int *rank_data_size, int nprocs, int *rank_data, int rank) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= local_data_row_count) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < local_data_row_count; i += stride) {
        int key = local_data[i * 2];
        int value = local_data[(i * 2) + 1];
        int destination_rank = key % nprocs;
//        int current_position = rank_data_size[destination_rank];
        int current_position = atomicAdd(&rank_data_size[destination_rank], 2);
//        if (rank == 0) {
//            printf("Hello from %d, key %d, value %d, destination rank %d, pos %d\n",
//                   i, key, value, destination_rank, current_position);
//        }
        rank_data[current_position] = key;
        rank_data[current_position + 1] = value;
//        atomicExch(&rank_data[current_position - 2], key);
//        atomicExch(&rank_data[current_position - 1], value);
//        atomicAdd(&rank_data_size[destination_rank], 2);
//        rank_data[current_position] = key;
//        rank_data[current_position + 1] = value;
//        atomicCAS(&rank_data[current_position], 0, key);
//        atomicCAS(&rank_data[current_position+1], 0, value);

//        atomicExch(&rank_data[current_position], key);
//        atomicExch(&rank_data[current_position + 1], value);
//        rank_data[rank_data_size[destination_rank] - 2] = key;
//        rank_data[rank_data_size[destination_rank] - 1] = value;
    }
}





//5 rows, 2 cols
//10 data local_data
//hash_values 5
//
//
//__global__ void get_split_relation_gpu(int *local_data, int *hash_values, int edge_count,
//                                       int total_columns, int rank) {
//    int id = threadIdx.x * threadIdx.y;
//    for (int i = 0; i < graph_edge_counts; i++) {
//        unsigned int hash_value = prefix_hash(local_data, id * 2, 1);
//        int hash_value_mod = hash_value % nprocs;
//        hash_values
//    }
//}


int main(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    setlocale(LC_ALL, "");
    double elapsed_time = -MPI_Wtime();
    double max_time = 0.0;
    int nprocs, rank;
    MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    int i, j, k;
    int total_columns = 2;
    // Should pass the input filename in command line argument
    const char *input_file;
    if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "data/hipc_2019.txt";
    }

    // load the raw graph
    int graph_edge_counts = get_edge_count(input_file);
    if (rank == 0) {
        std::cout << "Input graph edge counts: " << graph_edge_counts << std::endl;
    }
    int *raw_graph_data = (int *) malloc(graph_edge_counts * total_columns * sizeof(int));
    get_data_from_file(input_file, raw_graph_data, graph_edge_counts, total_columns, '\t');
    int *raw_reverse_graph_data = (int *) malloc(graph_edge_counts * total_columns * sizeof(int));

    for (i = 0; i < graph_edge_counts; i++) {
        raw_reverse_graph_data[i * 2 + 1] = raw_graph_data[i * total_columns];
        raw_reverse_graph_data[i * 2] = raw_graph_data[i * total_columns + 1];
    }
    int *device_graph;
    checkCuda(hipMalloc((void **) &device_graph, graph_edge_counts * total_columns * sizeof(int)));
    hipMemcpy(device_graph, raw_graph_data, graph_edge_counts * total_columns * sizeof(int),
               hipMemcpyHostToDevice);

    int *rank_data_size;
    checkCuda(hipMalloc((void **) &rank_data_size, nprocs * sizeof(int)));
    get_rank_data_size<<<grid_size, block_size>>>(device_graph, graph_edge_counts, rank_data_size, nprocs);
    checkCuda(hipDeviceSynchronize());
    // calculate offset
    thrust::exclusive_scan(thrust::device, rank_data_size, rank_data_size + nprocs, rank_data_size);
    checkCuda(hipDeviceSynchronize());

    int *rank_data;
    checkCuda(hipMalloc((void **) &rank_data, graph_edge_counts * total_columns * sizeof(int)));
    get_rank_data<<<grid_size, block_size>>>(device_graph, graph_edge_counts, rank_data_size, nprocs, rank_data, rank);
    checkCuda(hipDeviceSynchronize());


    int *rank_data_size_host = (int *) malloc(nprocs * sizeof(int));
    hipMemcpy(rank_data_size_host, rank_data_size, nprocs * sizeof(int),
               hipMemcpyDeviceToHost);

    int *host_graph = (int *) malloc(graph_edge_counts * total_columns * sizeof(int));
    hipMemcpy(host_graph, device_graph, graph_edge_counts * total_columns * sizeof(int),
               hipMemcpyDeviceToHost);
    int *rank_data_host = (int *) malloc(graph_edge_counts * total_columns * sizeof(int));
    hipMemcpy(rank_data_host, rank_data, graph_edge_counts * total_columns * sizeof(int),
               hipMemcpyDeviceToHost);
//    for (i = 0; i < nprocs; i++) {
//        cout << "Rank: " << rank << ", Destination Rank: " << i << ", data_size: " << rank_data_size_host[i] << endl;
//    }
//    cout << endl;

    if (rank == 0) {
//        int *rank_data_size_host_verification = (int *) malloc(nprocs * sizeof(int));
//
//        for (i = 0; i < graph_edge_counts; i++) {
//            rank_data_size_host_verification[raw_graph_data[i*2]%nprocs]++;
//        }
//        for (i = 0; i < nprocs; i++) {
//            cout << "Verify Rank " << i << ", data_size: " << rank_data_size_host_verification[i] << endl;
//        }
//        for (i = 0; i < nprocs; i++) {
//            cout << "Rank " << i << ", data_size: " << rank_data_size_host[i] << endl;
//        }
//        for (i = 0; i < graph_edge_counts; i++) {
//            cout << host_graph[i * 2] << " " << host_graph[(i * 2) + 1] << endl;
//        }
        cout << "----------------" << endl;
        for (i = 0; i < graph_edge_counts; i++) {
            cout << "Rank: " << rank << ", " << rank_data_host[i * 2] << " " << rank_data_host[(i * 2) + 1] << endl;
        }
    }

//    int *hash_values;
//    checkCuda(hipMalloc((void **) &hash_values, graph_edge_counts * sizeof(int)));


//    getting_Hash<<<512, 1024>>>(device_graph, hash_values, graph_edge_counts, total_columns, rank);
//    sort_by_Hash_gpu_buffer<<<512, 1024>>>(device_graph, hash_values, graph_edge_counts, total_columns, rank);
//    // cpu code to compute the displacement and size
//    // mpi all 2 all
//    // cpu code receive buffer
//    //


//    for (int i = 0; i < graph_edge_counts; i++) {
//        cout << raw_graph_data[i * total_columns] << ", " << raw_graph_data[(i * total_columns) + 1] << endl;
//    }

//    int *reverse_graph;
//    checkCuda(hipMalloc((void **) &reverse_graph, graph_edge_counts * 2 * sizeof(int)));

//    for (int i = 0; i < graph_edge_counts; i++) {
//        unsigned int hash_value = prefix_hash(raw_graph_data, i * 2, 1);
//        int hash_value_mod = hash_value % nprocs;
//        if (rank == 0) {
//            cout << "Rank " << rank << " key: " << raw_graph_data[i * 2] << " hash_value: " << hash_value
//                 << ", after mod: " << hash_value_mod << endl;
//        }
//    }


    free(raw_graph_data);
    free(raw_reverse_graph_data);
    hipFree(device_graph);
    MPI_Finalize();
    return 0;
}

// make runcomm DATA_FILE=data/data_10.txt NPROCS=8