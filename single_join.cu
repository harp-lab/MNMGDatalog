#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/kernels.cu"
#include "common/parallel_io.cu"
#include "common/comm.cu"
#include "common/hash_table.cu"
#include "common/join.cu"

using namespace std;


void benchmark(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    Output output;
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    setlocale(LC_ALL, "");
    double start_time, end_time, elapsed_time;
    double initialization_time = 0.0, max_initialization_time = 0.0;
    double finalization_time = 0.0, max_finalization_time = 0.0;
    double file_io_time = 0.0, max_fileio_time = 0.0;
    double max_join_time = 0.0, max_merge_time = 0.0;
    double max_buffer_preparation_time_before_join = 0.0, max_communication_time_before_join = 0.0;
    double max_buffer_preparation_time_after_join = 0.0, max_communication_time_after_join = 0.0;
    double buffer_preparation_time_before_join = 0.0, communication_time_before_join = 0.0;
    double buffer_preparation_time_after_join = 0.0, communication_time_after_join = 0.0;
    double buffer_preparation_time_temp = 0.0, communication_time_temp = 0.0;
    double join_time = 0.0, merge_time = 0.0;
    double deduplication_time = 0.0, max_deduplication_time = 0.0;;
    double hashtable_build_time = 0.0, max_hashtable_build_time = 0.0;
    double clear_time = 0.0, max_clear_time = 0.0;;
    double copy_to_host_time = 0.0, max_copy_to_host_time = 0.0;;

    double total_time = 0.0, max_total_time = 0.0;
    int total_rank, rank;
    int i;
    MPI_Comm_size(MPI_COMM_WORLD, &total_rank);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    warm_up_kernel<<<1, 1>>>();
    int iterations = 1;
    // Should pass the input filename in command line argument
    const char *input_file;
    int comm_method = 0;
    int job_run = 1;
    int cuda_aware_mpi = 0;
    int rand_range = 1000000;
    if (argc == 5) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
        rand_range = atoi(argv[4]);
    } else if (argc == 4) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
    } else if (argc == 3) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
    } else if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "hipc_2019.bin";
    }
    string output_file = string(input_file) + "_singlejoin.bin";
    const char *output_file_name = output_file.c_str();
    int total_rows = atoi(input_file);
    // if total_rows > 10M then perform strong scaling

    // Read file in parallel
    int total_columns = 2;
    double temp_file_io_time = 0.0;
    long long row_size = 0;
    int *local_data_host = parallel_generate(total_rank, rank, total_rows, total_columns, rand_range,
                                             &row_size, &temp_file_io_time);

    long long local_count = row_size * total_columns;
    long long global_row_size = 0;
    MPI_Allreduce(&row_size, &global_row_size, 1, MPI_LONG_LONG_INT, MPI_SUM,
                  MPI_COMM_WORLD);
#ifdef DEBUG
    cout << "Rank: " << rank << ", Local count: " << local_count << endl;
#endif
    file_io_time += temp_file_io_time;

    start_time = MPI_Wtime();
    int *local_data_device;
    checkCuda(hipMalloc((void **) &local_data_device, local_count * sizeof(int)));
    hipMemcpy(local_data_device, local_data_host, local_count * sizeof(int), hipMemcpyHostToDevice);
    Entity *local_data;
    checkCuda(hipMalloc((void **) &local_data, row_size * sizeof(Entity)));
    Entity *local_data_reverse;
    checkCuda(hipMalloc((void **) &local_data_reverse, row_size * sizeof(Entity)));
    create_entity_ar<<<grid_size, block_size>>>(local_data, row_size, local_data_device);
    create_entity_ar_reverse<<<grid_size, block_size>>>(local_data_reverse, row_size, local_data_device);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    int input_relation_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    Entity *input_relation = get_split_relation(rank, local_data,
                                                row_size, total_columns, total_rank,
                                                grid_size, block_size, cuda_aware_mpi,
                                                &input_relation_size, comm_method,
                                                &buffer_preparation_time_temp, &communication_time_temp, iterations);
    buffer_preparation_time_before_join += buffer_preparation_time_temp;
    communication_time_before_join += communication_time_temp;
#ifdef DEBUG
    cout << "Rank: " << rank << ", input_relation_size: " << input_relation_size << endl;
#endif

    start_time = MPI_Wtime();
    thrust::sort(thrust::device, input_relation, input_relation + input_relation_size, set_cmp());
    input_relation_size = (thrust::unique(thrust::device,
                                          input_relation, input_relation + input_relation_size,
                                          is_equal())) - input_relation;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;
#ifdef DEBUG
    cout << "Rank: " << rank << ", input_relation_size after deduplication: " << input_relation_size << endl;
#endif


    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    int reverse_relation_size = 0;
    Entity *reverse_relation = get_split_relation(rank, local_data_reverse,
                                                  row_size, total_columns, total_rank,
                                                  grid_size, block_size, cuda_aware_mpi, &reverse_relation_size,
                                                  comm_method,
                                                  &buffer_preparation_time_temp, &communication_time_temp, iterations);
    buffer_preparation_time_before_join += buffer_preparation_time_temp;
    communication_time_before_join += communication_time_temp;
#ifdef DEBUG
    cout << "Rank: " << rank << ", reverse_relation_size: " << reverse_relation_size << endl;
#endif
    start_time = MPI_Wtime();
    thrust::sort(thrust::device, reverse_relation, reverse_relation + reverse_relation_size, set_cmp());
    reverse_relation_size = (thrust::unique(thrust::device,
                                            reverse_relation, reverse_relation + reverse_relation_size,
                                            is_equal())) - reverse_relation;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;
#ifdef DEBUG
    cout << "Rank: " << rank << ", reverse_relation_size after deduplication: " << reverse_relation_size << endl;
#endif

    // Hash table is Edge
    double temp_hashtable_build_time = 0.0;
    int hash_table_rows = 0;
    Entity *hash_table = get_hash_table(grid_size, block_size, input_relation, input_relation_size,
                                        &hash_table_rows, &temp_hashtable_build_time);
    hashtable_build_time += temp_hashtable_build_time;
#ifdef DEBUG
    cout << "Rank: " << rank << ", hash_table_rows: " << hash_table_rows << endl;
#endif

    double temp_join_time = 0.0;
    int join_result_size = 0;
    Entity *join_result = get_join(grid_size, block_size, hash_table, hash_table_rows,
                                   reverse_relation, reverse_relation_size,
                                   &join_result_size, &temp_join_time);

    join_time += temp_join_time;
#ifdef DEBUG
    cout << "Rank: " << rank << ", join_result_size: " << join_result_size << endl;
#endif
    // Scatter the join result among relevant processes
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    int distributed_join_result_size = 0;
    Entity *distributed_join_result = get_split_relation(rank, join_result,
                                                         join_result_size, total_columns, total_rank,
                                                         grid_size, block_size, cuda_aware_mpi,
                                                         &distributed_join_result_size,
                                                         comm_method,
                                                         &buffer_preparation_time_temp, &communication_time_temp,
                                                         iterations);
    buffer_preparation_time_after_join += buffer_preparation_time_temp;
    communication_time_after_join += communication_time_temp;
#ifdef DEBUG
    cout << "Rank: " << rank << ", distributed_join_result_size: " << distributed_join_result_size << endl;
#endif

    // Deduplicate distributed join result
    start_time = MPI_Wtime();
    thrust::sort(thrust::device, distributed_join_result,
                 distributed_join_result + distributed_join_result_size, set_cmp());
    distributed_join_result_size = (thrust::unique(thrust::device,
                                                   distributed_join_result,
                                                   distributed_join_result + distributed_join_result_size,
                                                   is_equal())) - distributed_join_result;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;
#ifdef DEBUG
    cout << "Rank: " << rank << ", distributed_join_result_size after deduplication: " << distributed_join_result_size << endl;
#endif

    start_time = MPI_Wtime();
    long long global_join_result_size = 0;
    long long distributed_join_result_size_temp = distributed_join_result_size;
    MPI_Allreduce(&distributed_join_result_size_temp, &global_join_result_size, 1, MPI_LONG_LONG_INT, MPI_SUM,
                  MPI_COMM_WORLD);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;


    start_time = MPI_Wtime();
    // Create int ar from entity ar
    int *distributed_join_result_ar;
    checkCuda(hipMalloc((void **) &distributed_join_result_ar,
                         distributed_join_result_size * total_columns * sizeof(int)));
    get_reverse_int_ar_from_entity_ar<<<grid_size, block_size>>>(distributed_join_result, distributed_join_result_size,
                                                                 distributed_join_result_ar);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    // Copy to host for file write
    start_time = MPI_Wtime();
    int *distributed_join_result_ar_host = (int *) malloc(distributed_join_result_size * total_columns * sizeof(int));
    hipMemcpy(distributed_join_result_ar_host, distributed_join_result_ar,
               distributed_join_result_size * total_columns * sizeof(int), hipMemcpyDeviceToHost);

    // List the t full counts for each process and calculate the displacements in the final result
    int *join_result_counts = (int *) calloc(total_rank, sizeof(int));
    MPI_Allgather(&distributed_join_result_size, 1, MPI_INT,
                  join_result_counts, 1, MPI_INT, MPI_COMM_WORLD);

    int *join_result_displacements = (int *) calloc(total_rank, sizeof(int));
    for (i = 1; i < total_rank; i++) {
        join_result_displacements[i] = join_result_displacements[i - 1] + (join_result_counts[i - 1] * total_columns);
    }
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    copy_to_host_time += elapsed_time;

    if (job_run == 0) {
        // Write the t full to an offset of the output file
        double temp_file_write_time = 0.0;
        parallel_write(rank, total_rank, output_file_name, distributed_join_result_ar_host, join_result_displacements,
                       total_columns, distributed_join_result_size, &temp_file_write_time);
        file_io_time += temp_file_write_time;
    }

    start_time = MPI_Wtime();
    hipFree(local_data_device);
    hipFree(input_relation);
    hipFree(local_data);
    hipFree(local_data_reverse);
    hipFree(join_result);
    hipFree(distributed_join_result);
    hipFree(distributed_join_result_ar);
    hipFree(hash_table);

    free(distributed_join_result_ar_host);
    free(join_result_counts);
    free(join_result_displacements);
    free(local_data_host);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    clear_time += elapsed_time;

    total_time = initialization_time + hashtable_build_time + join_time +
                 buffer_preparation_time_before_join + communication_time_before_join +
                 buffer_preparation_time_after_join + communication_time_after_join +
                 deduplication_time + merge_time + clear_time + finalization_time + copy_to_host_time;
    MPI_Allreduce(&total_time, &max_total_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    // Breakdown time is the breakdown times of the slowest process
    if (total_time == max_total_time) {
        max_initialization_time = initialization_time;
        max_deduplication_time = deduplication_time;
        max_join_time = join_time;
        max_merge_time = merge_time;
        max_buffer_preparation_time_before_join = buffer_preparation_time_before_join;
        max_communication_time_before_join = communication_time_before_join;
        max_buffer_preparation_time_after_join = buffer_preparation_time_after_join;
        max_communication_time_after_join = communication_time_after_join;
        max_hashtable_build_time = hashtable_build_time;
        max_fileio_time = file_io_time;
        max_clear_time = clear_time;
        max_finalization_time = finalization_time;
        max_copy_to_host_time = copy_to_host_time;
        output.block_size = block_size;
        output.grid_size = grid_size;
        output.input_rows = global_row_size;
        output.total_rank = total_rank;
        output.iterations = iterations;
        output.output_file_name = output_file_name;
        output.output_size = global_join_result_size;

        output.total_time = max_total_time;
        output.initialization_time = max_initialization_time;
        output.fileio_time = max_fileio_time;
        output.hashtable_build_time = max_hashtable_build_time;
        output.join_time = max_join_time;
        output.deduplication_time = max_deduplication_time;
        output.merge_time = max_merge_time;
        output.finalization_time = max_finalization_time;
        if (job_run == 0) {
            printf("| # Input | # Process | # Iterations | # Join | Total Time ");
            printf("| Initialization | File I/O | Hashtable | Join | Buffer preparation (before) | Communication (before) | Buffer preparation (after) | Communication (after) | Deduplication | Clear | Copy | Finalization | Output |\n");
            printf("| --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- |\n");
        }
        printf("| %'d | %'d | %'d | %'lld | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %s |\n",
               output.input_rows, output.total_rank, output.iterations,
               output.output_size, output.total_time,
               output.initialization_time, output.fileio_time, output.hashtable_build_time, output.join_time,
               max_buffer_preparation_time_before_join, max_communication_time_before_join,
               max_buffer_preparation_time_after_join, max_communication_time_after_join,
               output.deduplication_time, max_clear_time, max_copy_to_host_time,
               output.finalization_time,
               output.output_file_name);
    }
    MPI_Finalize();
}

int main(int argc, char **argv) {
    benchmark(argc, argv);
    return 0;
}
// METHOD 0 = two pass method, 1 = sorting method
// DATA_FILE>10000000, strong scaling, otherwise weak scaling;
// make runsinglejoin DATA_FILE=100 NPROCS=4 CUDA_AWARE_MPI=0 METHOD=0 RAND_RANGE=100
// make runsinglejoin DATA_FILE=10 NPROCS=4 CUDA_AWARE_MPI=0 METHOD=0 RAND_RANGE=100
// DATA_FILE>10M, strong scaling, otherwise weak scaling;
// make runsinglejoin DATA_FILE=10000000 NPROCS=4 CUDA_AWARE_MPI=0 METHOD=0 RAND_RANGE=100000
// make runsinglejoin DATA_FILE=20000000 NPROCS=4 CUDA_AWARE_MPI=0 METHOD=0 RAND_RANGE=1000000

// Exp 1: control output size linearly
// make runsinglejoin DATA_FILE=5000000 NPROCS=1 CUDA_AWARE_MPI=0 METHOD=0 RAND_RANGE=1000000
// mpirun -np 1 ./single_join.out 5000000 0 0 1000000
// mpirun -np 2 ./single_join.out 5000000 0 0 2000000
// mpirun -np 4 ./single_join.out 5000000 0 0 4000000
// mpirun -np 8 ./single_join.out 5000000 0 0 8000000
// make runsinglejoin DATA_FILE=5000000 NPROCS=2 CUDA_AWARE_MPI=0 METHOD=0 RAND_RANGE=1000000
// make runsinglejoin DATA_FILE=5000000 NPROCS=1 CUDA_AWARE_MPI=0 METHOD=0 RAND_RANGE=500000

// Exp 2: do not control output size


// Exp 3: Smaller range


// Polaris exp 1
// mpirun -np 1 ./single_join.out 10000000 0 0 1000000
// mpirun -np 2 ./single_join.out 10000000 0 0 2000000