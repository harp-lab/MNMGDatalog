#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/remove.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/kernels.cu"
#include "common/comm.cu"
#include "common/hash_table.cu"
#include "common/join.cu"

using namespace std;


void benchmark(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    Output output;
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    setlocale(LC_ALL, "");
    double start_time, end_time, elapsed_time;
    double max_fileio_time = 0.0, max_initialization_time = 0.0, max_finalization_time = 0.0;
    double max_join_time = 0.0, max_merge_time = 0.0, max_deduplication_time = 0.0;
    double max_buffer_preparation_time = 0.0, max_communication_time = 0.0;
    double buffer_preparation_time = 0.0, communication_time = 0.0;
    double buffer_preparation_time_temp = 0.0, communication_time_temp = 0.0;
    double join_time = 0.0, merge_time = 0.0, deduplication_time = 0.0;
    double initialization_time = 0.0, finalization_time = 0.0;
    double file_io_time = 0.0;
    double hashtable_build_time = 0.0, max_hashtable_build_time = 0.0;
    double total_time = 0.0, max_total_time = 0.0;
    int total_rank, rank;
    int i;
    MPI_Comm_size(MPI_COMM_WORLD, &total_rank);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    warm_up_kernel<<<1, 1>>>();
    // Should pass the input filename in command line argument
    const char *input_file;
    int comm_method = 0;
    int job_run = 0;
    int cuda_aware_mpi = 0;

    if (argc == 5) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
        job_run = atoi(argv[4]);
    } else if (argc == 4) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
    } else if (argc == 3) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
    } else if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "hipc_2019.bin";
    }
    string output_file = string(input_file) + "_sg.bin";
    const char *output_file_name = output_file.c_str();

    // READ THE FILE IN PARALLEL
    // Reading filesize in bytes
    start_time = MPI_Wtime();
    struct stat filestats;
    stat(input_file, &filestats);
    off_t filesize = filestats.st_size;

    // Calculating the current rank's starting row and number of rows
    // Scatter larger blocks among processes (non-uniform)
    int total_columns = 2;
    int total_rows = filesize / (sizeof(int) * total_columns);
    int row_start = BLOCK_START(rank, total_rank, total_rows);
    int row_size = BLOCK_SIZE(rank, total_rank, total_rows);
    int local_count = row_size * total_columns;

    // Reading specific portion from the file as char in parallel
    int offset = row_start * total_columns * sizeof(int);
    int *local_data_host = (int *) malloc(local_count * sizeof(int));
    memset(local_data_host, 0, local_count * sizeof(int));
    MPI_File mpi_file_buffer;
    if (MPI_File_open(MPI_COMM_WORLD, input_file, MPI_MODE_RDONLY,
                      MPI_INFO_NULL, &mpi_file_buffer) != MPI_SUCCESS) {
        printf("Error opening file %s", input_file);
        MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    }
    MPI_File_read_at(mpi_file_buffer, offset, local_data_host, local_count, MPI_INT, MPI_STATUS_IGNORE);
    MPI_File_close(&mpi_file_buffer);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    file_io_time = elapsed_time;
    start_time = MPI_Wtime();
    int *local_data_device;
    checkCuda(hipMalloc((void **) &local_data_device, local_count * sizeof(int)));
    hipMemcpy(local_data_device, local_data_host, local_count * sizeof(int), hipMemcpyHostToDevice);
    Entity *local_data;
    checkCuda(hipMalloc((void **) &local_data, row_size * sizeof(Entity)));
    create_entity_ar<<<grid_size, block_size>>>(local_data, row_size, local_data_device);
    int iterations = 0;
    end_time = MPI_Wtime();
    initialization_time = end_time - start_time;
    int input_relation_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    Entity *input_relation = get_split_relation(rank, local_data,
                                                row_size, total_columns, total_rank,
                                                grid_size, block_size, cuda_aware_mpi,
                                                &input_relation_size, comm_method,
                                                &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;

    start_time = MPI_Wtime();
    Entity *t_delta;
    int t_delta_size = input_relation_size;
    checkCuda(hipMalloc((void **) &t_delta, t_delta_size * sizeof(Entity)));
    hipMemcpy(t_delta, input_relation, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, t_delta, t_delta + t_delta_size, set_cmp());
    t_delta_size = (thrust::unique(thrust::device,
                                   t_delta, t_delta + t_delta_size,
                                   is_equal())) - t_delta;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;


    // Hash table is t_delta
    double temp_hashtable_build_time = 0.0;
    int hash_table_rows = 0;
    Entity *hash_table = get_hash_table(grid_size, block_size, t_delta, t_delta_size,
                                        &hash_table_rows, &temp_hashtable_build_time);
    hashtable_build_time += temp_hashtable_build_time;

    // T_FULL is input relation join t delta projected by key != value
    // sg(x, y): - edge(p, x), edge(p, y), x != y.
    double base_join_time = 0.0;
    int base_join_size = 0;
    Entity *base_join_result = get_join(grid_size, block_size, hash_table, hash_table_rows, t_delta, t_delta_size,
                                        &base_join_size,
                                        &base_join_time);
    join_time += base_join_time;

    start_time = MPI_Wtime();
    Entity *same_key_value_removed = thrust::remove_if(thrust::device, base_join_result,
                                        base_join_result + base_join_size,
                                        is_key_equal_value());
    base_join_size = same_key_value_removed - base_join_result;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    int t_delta_size_temp = 0;
    Entity *t_delta_temp_base = get_split_relation(rank, base_join_result,
                                                   base_join_size, total_columns, total_rank,
                                                   grid_size, block_size, cuda_aware_mpi,
                                                   &t_delta_size_temp, comm_method,
                                                   &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;

    start_time = MPI_Wtime();
    t_delta_size = t_delta_size_temp;
    hipFree(t_delta);
    checkCuda(hipMalloc((void **) &t_delta, t_delta_size * sizeof(Entity)));
    hipMemcpy(t_delta, t_delta_temp_base, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    start_time = MPI_Wtime();
    // Deduplicate scattered facts
    thrust::stable_sort(thrust::device, t_delta, t_delta + t_delta_size, set_cmp());
    t_delta_size = (thrust::unique(thrust::device, t_delta, t_delta + t_delta_size, is_equal())) - t_delta;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    start_time = MPI_Wtime();
    Entity *t_full;
    checkCuda(hipMalloc((void **) &t_full, t_delta_size * sizeof(Entity)));
    hipMemcpy(t_full, t_delta, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);

    long long global_t_full_size;
    long long t_full_size = t_delta_size;
    MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    merge_time += elapsed_time;


    while (true) {
        // tmp(b, x): - edge(a, x), sg(a, b).
        double first_join_time = 0.0;
        int first_join_size = 0;
        Entity *first_join_result = get_join(grid_size, block_size, hash_table, hash_table_rows, t_delta, t_delta_size,
                                             &first_join_size,
                                             &first_join_time);
        join_time += first_join_time;
        start_time = MPI_Wtime();
        reverse_entity_ar<<<grid_size, block_size>>>(first_join_result, first_join_size, first_join_result);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        join_time += elapsed_time;

        // Scatter the new facts among relevant processes
        buffer_preparation_time_temp = 0.0;
        communication_time_temp = 0.0;
        int distributed_first_join_size = 0;
        Entity *distributed_first_join_result = get_split_relation(rank, first_join_result,
                                                                   first_join_size, total_columns, total_rank,
                                                                   grid_size, block_size, cuda_aware_mpi,
                                                                   &distributed_first_join_size,
                                                                   comm_method,
                                                                   &buffer_preparation_time_temp,
                                                                   &communication_time_temp);
        buffer_preparation_time += buffer_preparation_time_temp;
        communication_time += communication_time_temp;

        start_time = MPI_Wtime();
        // Deduplicate scattered facts
        thrust::stable_sort(thrust::device, distributed_first_join_result,
                            distributed_first_join_result + distributed_first_join_size, set_cmp());
        distributed_first_join_size = (thrust::unique(thrust::device,
                                                      distributed_first_join_result,
                                                      distributed_first_join_result + distributed_first_join_size,
                                                      is_equal())) - distributed_first_join_result;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        deduplication_time += elapsed_time;

        // sg(x, y): - tmp(b, x), edge(b, y).
        double second_join_time = 0.0;
        int second_join_size = 0;
        Entity *second_join_result = get_join(grid_size, block_size, hash_table, hash_table_rows,
                                              distributed_first_join_result, distributed_first_join_size,
                                              &second_join_size, &second_join_time);

        join_time += second_join_time;
        start_time = MPI_Wtime();
        reverse_entity_ar<<<grid_size, block_size>>>(second_join_result, second_join_size, second_join_result);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        join_time += elapsed_time;

        // Scatter the new facts among relevant processes
        buffer_preparation_time_temp = 0.0;
        communication_time_temp = 0.0;
        int distributed_second_join_size = 0;
        Entity *distributed_second_join_result = get_split_relation(rank, second_join_result,
                                                                    second_join_size, total_columns, total_rank,
                                                                    grid_size, block_size, cuda_aware_mpi,
                                                                    &distributed_second_join_size,
                                                                    comm_method,
                                                                    &buffer_preparation_time_temp,
                                                                    &communication_time_temp);
        buffer_preparation_time += buffer_preparation_time_temp;
        communication_time += communication_time_temp;
        start_time = MPI_Wtime();
        // Deduplicate scattered facts
        thrust::stable_sort(thrust::device, distributed_second_join_result,
                            distributed_second_join_result + distributed_second_join_size, set_cmp());
        distributed_second_join_size = (thrust::unique(thrust::device,
                                                       distributed_second_join_result,
                                                       distributed_second_join_result + distributed_second_join_size,
                                                       is_equal())) - distributed_second_join_result;

        t_delta_size = distributed_second_join_size;
        hipFree(t_delta);
        checkCuda(hipMalloc((void **) &t_delta, t_delta_size * sizeof(Entity)));
        hipMemcpy(t_delta, distributed_second_join_result, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        deduplication_time += elapsed_time;
        start_time = MPI_Wtime();
        // Update t delta which is the only new facts which are not in t full and will be used in next iteration
        t_delta_size = thrust::set_difference(thrust::device,
                                              t_delta, t_delta + t_delta_size,
                                              t_full, t_full + t_full_size,
                                              t_delta, set_cmp()) - t_delta;

        // set union of two sets (sorted t full and t delta)
        Entity *new_t_full;
        int new_t_full_size = t_delta_size + t_full_size;
        checkCuda(hipMalloc((void **) &new_t_full, new_t_full_size * sizeof(Entity)));
        new_t_full_size = thrust::set_union(thrust::device,
                                            t_full, t_full + t_full_size,
                                            t_delta, t_delta + t_delta_size,
                                            new_t_full, set_cmp()) - new_t_full;
        hipFree(t_full);
        checkCuda(hipMalloc((void **) &t_full, new_t_full_size * sizeof(Entity)));
        hipMemcpy(t_full, new_t_full, new_t_full_size * sizeof(Entity), hipMemcpyDeviceToDevice);
        t_full_size = new_t_full_size;
        // Check if the global t full size has changed in this iteration
        long long old_global_t_full_size = global_t_full_size;
        MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);
        iterations++;
        hipFree(distributed_first_join_result);
        hipFree(distributed_second_join_result);
        hipFree(new_t_full);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;
        if (old_global_t_full_size == global_t_full_size) {
            break;
        }
    }

    MPI_Allreduce(&initialization_time, &max_initialization_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&join_time, &max_join_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&deduplication_time, &max_deduplication_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&merge_time, &max_merge_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&buffer_preparation_time, &max_buffer_preparation_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&communication_time, &max_communication_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&hashtable_build_time, &max_hashtable_build_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    start_time = MPI_Wtime();
    // Reverse the t_full as we stored it in reverse order initially
    int *t_full_ar;
    checkCuda(hipMalloc((void **) &t_full_ar, t_full_size * total_columns * sizeof(int)));
    get_int_ar_from_entity_ar<<<grid_size, block_size>>>(t_full, t_full_size, t_full_ar);

    // Copy t full to host for file write
    int *t_full_ar_host = (int *) malloc(t_full_size * total_columns * sizeof(int));
    hipMemcpy(t_full_ar_host, t_full_ar, t_full_size * total_columns * sizeof(int), hipMemcpyDeviceToHost);

    // List the t full counts for each process and calculate the displacements in the final result
    int *t_full_counts = (int *) calloc(total_rank, sizeof(int));
    MPI_Allgather(&t_full_size, 1, MPI_INT,
                  t_full_counts, 1, MPI_INT, MPI_COMM_WORLD);

    int *t_full_displacements = (int *) calloc(total_rank, sizeof(int));
    for (i = 1; i < total_rank; i++) {
        t_full_displacements[i] = t_full_displacements[i - 1] + (t_full_counts[i - 1] * total_columns);
    }
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    if(job_run == 0){
        // Comment out file write for polaris benchmark
        // Write the t full to an offset of the output file
        start_time = MPI_Wtime();
        MPI_File fh;
        MPI_File_open(MPI_COMM_WORLD, output_file_name, MPI_MODE_WRONLY | MPI_MODE_CREATE, MPI_INFO_NULL, &fh);
        int file_offset = t_full_displacements[rank] * sizeof(int);
        MPI_File_write_at(fh, file_offset, t_full_ar_host, t_full_size * total_columns, MPI_INT, MPI_STATUS_IGNORE);
        // Close the file and clean up
        MPI_File_close(&fh);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        file_io_time += elapsed_time;
    }
    MPI_Allreduce(&file_io_time, &max_fileio_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    start_time = MPI_Wtime();
    hipFree(local_data_device);
    hipFree(input_relation);
    hipFree(local_data);
    hipFree(t_full);
    hipFree(t_delta);
    hipFree(t_full_ar);
    hipFree(hash_table);
    hipFree(base_join_result);

    free(t_full_ar_host);
    free(t_full_counts);
    free(t_full_displacements);
    free(local_data_host);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;
    MPI_Allreduce(&finalization_time, &max_finalization_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    total_time = max_initialization_time + max_hashtable_build_time + max_join_time +
                 max_buffer_preparation_time + max_communication_time + max_merge_time + max_deduplication_time +
                 max_finalization_time;
    MPI_Allreduce(&total_time, &max_total_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    if (rank == 0) {
        output.block_size = block_size;
        output.grid_size = grid_size;
        output.input_rows = total_rows;
        output.total_rank = total_rank;
        output.iterations = iterations;
        output.output_file_name = output_file_name;
        output.output_size = global_t_full_size;

        output.total_time = max_total_time;
        output.initialization_time = max_initialization_time;
        output.fileio_time = max_fileio_time;
        output.hashtable_build_time = max_hashtable_build_time;
        output.join_time = max_join_time;
        output.buffer_preparation_time = max_buffer_preparation_time;
        output.communication_time = max_communication_time;
        output.merge_time = max_merge_time;
        output.deduplication_time = max_deduplication_time;
        output.finalization_time = max_finalization_time;
        if(job_run == 0){
            // Comment out next 3 prints for polaris benchmark
            printf("| # Input | # Process | # Iterations | # SG | Total Time ");
            printf("| Initialization | File I/O | Hashtable | Join | Buffer preparation | Communication | Deduplication | Merge | Finalization | Output |\n");
            printf("| --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- |\n");
        }
        printf("| %'d | %'d | %'d | %'lld | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %s |\n",
               output.input_rows, output.total_rank, output.iterations,
               output.output_size, output.total_time,
               output.initialization_time, output.fileio_time, output.hashtable_build_time, output.join_time,
               output.buffer_preparation_time, output.communication_time, output.deduplication_time, output.merge_time,
               output.finalization_time,
               output.output_file_name);
    }
    MPI_Finalize();
}

int main(int argc, char **argv) {
    benchmark(argc, argv);
    return 0;
}
// METHOD 0 = two pass method, 1 = sorting method
// make runsg DATA_FILE=data/data_10.bin NPROCS=1 CUDA_AWARE_MPI=0 METHOD=0
// make runsg DATA_FILE=data/hipc_2019.bin NPROCS=1 CUDA_AWARE_MPI=0 METHOD=0
// make runsg DATA_FILE=data/data_7035.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runsg DATA_FILE=data/data_7035.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=1
