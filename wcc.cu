#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <stdio.h>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/iterator/transform_iterator.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/parallel_io.cu"
#include "common/kernels.cu"
#include "common/comm.cu"
#include "common/hash_table.cu"
#include "common/join.cu"

using namespace std;

/*
Base rule:
edge(x,y) ← edge(y,x).
cc(n, n) ← edge(n,_).
t_delta(x, y) ← cc(x, y)

Recursive rule:
join_result(z, x) ← t_delta(y, z), edge(x, y).
join_result(z, x) ← join_result(x, z).
cc_new(y, min(z)) ← cc(y, z) U join_result(z, x)
t_delta(x, y) ← cc_new(y, z) - cc_old(y, z)
cc(x, y) ← cc_new(x, y)

Final rule:
cc_representative_node(n) ← cc(_ , n).
*/
void benchmark(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    Output output;
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    setlocale(LC_ALL, "");
    double start_time, end_time, elapsed_time;
    double initialization_time = 0.0, max_initialization_time = 0.0;
    double finalization_time = 0.0, max_finalization_time = 0.0;
    double file_io_time = 0.0, max_fileio_time = 0.0;
    double max_join_time = 0.0, max_merge_time = 0.0;
    double max_buffer_preparation_time = 0.0, max_communication_time = 0.0;
    double buffer_preparation_time = 0.0, communication_time = 0.0;
    double buffer_preparation_time_temp = 0.0, communication_time_temp = 0.0;
    double join_time = 0.0, merge_time = 0.0;
    double deduplication_time = 0.0, max_deduplication_time = 0.0;
    double hashtable_build_time = 0.0, max_hashtable_build_time = 0.0;

    double total_time = 0.0, max_total_time = 0.0;
    int total_rank, rank;
    int i;
    MPI_Comm_size(MPI_COMM_WORLD, &total_rank);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    warm_up_kernel<<<1, 1>>>();
    int iterations = 0;
    // Should pass the input filename in command line argument
    const char *input_file;
    int comm_method = 0;
    int job_run = 0;
    int cuda_aware_mpi = 0;

    if (argc == 5) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
        job_run = atoi(argv[4]);
    } else if (argc == 4) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
    } else if (argc == 3) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
    } else if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "data/dummy.bin";
    }
    string output_file = string(input_file) + "_cc.bin";
    const char *output_file_name = output_file.c_str();

    // Read file in parallel
    int total_columns = 2;
    double temp_file_io_time = 0.0;
    int row_size = 0;
    int total_rows = 0;
    int *edge_host = parallel_read(rank, total_rank, input_file, total_columns,
                                   &row_size, &total_rows, &temp_file_io_time);
    int local_count = row_size * total_columns;
    file_io_time += temp_file_io_time;

    start_time = MPI_Wtime();
    int *edge_temp_device;
    checkCuda(hipMalloc((void **) &edge_temp_device, local_count * sizeof(int)));
    hipMemcpy(edge_temp_device, edge_host, local_count * sizeof(int), hipMemcpyHostToDevice);
    // Ensure edges are bidirectional by adding reverse edges
    int *edge_reverse_temp_device;
    checkCuda(hipMalloc((void **) &edge_reverse_temp_device, local_count * sizeof(int)));
    get_reverse_ar<<<grid_size, block_size>>>(edge_temp_device, row_size, edge_reverse_temp_device);

    // Create Entity array from GPU buffers, edge + reverse_edge
    // edge(x, y) :- edge(y, x)
    Entity *edge;
    int edge_size = local_count;
    checkCuda(hipMalloc((void **) &edge, edge_size * sizeof(Entity)));
    create_entity_ar_with_offset<<<grid_size, block_size>>>(edge_temp_device, row_size, edge, 0);
    create_entity_ar_with_offset<<<grid_size, block_size>>>(edge_reverse_temp_device, row_size,
                                                            edge, row_size);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    // Deduplicate local data
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, edge, edge + edge_size, set_cmp());
    edge_size = (thrust::unique(thrust::device,
                                edge, edge + edge_size,
                                is_equal())) - edge;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    // Distribute edge
    int distributed_edge_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    Entity *distributed_edge = get_split_relation(rank, edge,
                                                  edge_size, total_columns, total_rank,
                                                  grid_size, block_size, cuda_aware_mpi,
                                                  &distributed_edge_size, comm_method,
                                                  &buffer_preparation_time_temp, &communication_time_temp, iterations);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;

    // Deduplicate distributed edge
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, distributed_edge, distributed_edge + distributed_edge_size, set_cmp());
    distributed_edge_size = (thrust::unique(thrust::device,
                                            distributed_edge, distributed_edge + distributed_edge_size,
                                            is_equal())) - distributed_edge;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    // Create cc from edge where node, component_id = node, node
    // cc(x, x) :- edge(x, _)
    start_time = MPI_Wtime();
    Entity *cc;
    int cc_size = distributed_edge_size;
    checkCuda(hipMalloc((void **) &cc, cc_size * sizeof(Entity)));
    same_key_value_entity_ar<<<grid_size, block_size>>>(distributed_edge, cc_size, cc);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    // Deduplicate cc
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, cc, cc + cc_size, set_cmp());
    cc_size = (thrust::unique(thrust::device,
                              cc, cc + cc_size,
                              is_equal_key())) - cc;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    // t_delta = cc, key-value pair: node - component id
    start_time = MPI_Wtime();
    Entity *t_delta;
    int t_delta_size = cc_size;
    checkCuda(hipMalloc((void **) &t_delta, t_delta_size * sizeof(Entity)));
    hipMemcpy(t_delta, cc, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    start_time = MPI_Wtime();
    long long global_t_delta_size = 0;
    long long t_delta_size_temp = t_delta_size;
    MPI_Allreduce(&t_delta_size_temp, &global_t_delta_size, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    // Hash table is Edge
    double temp_hashtable_build_time = 0.0;
    int hash_table_rows = 0;
    Entity *hash_table = get_hash_table(grid_size, block_size, distributed_edge, distributed_edge_size,
                                        &hash_table_rows, &temp_hashtable_build_time);
    hashtable_build_time += temp_hashtable_build_time;


    while (true) {
        Entity *new_cc;
        double temp_join_time = 0.0;
        int join_result_size = 0;
        Entity *join_result = get_join(grid_size, block_size, hash_table, hash_table_rows,
                                       t_delta, t_delta_size,
                                       &join_result_size, &temp_join_time);
        join_time += temp_join_time;

        // Scatter the join result with reverse among relevant processes
        buffer_preparation_time_temp = 0.0;
        communication_time_temp = 0.0;
        int distributed_join_result_size = 0;
        Entity *distributed_join_result = get_split_relation(rank, join_result,
                                                             join_result_size, total_columns, total_rank,
                                                             grid_size, block_size, cuda_aware_mpi,
                                                             &distributed_join_result_size,
                                                             comm_method,
                                                             &buffer_preparation_time_temp, &communication_time_temp,
                                                             iterations);
        buffer_preparation_time += buffer_preparation_time_temp;
        communication_time += communication_time_temp;


        // Deduplicate distributed join result with reverse
        start_time = MPI_Wtime();
        thrust::stable_sort(thrust::device, distributed_join_result,
                            distributed_join_result + distributed_join_result_size, set_cmp());
        distributed_join_result_size = (thrust::unique(thrust::device,
                                                       distributed_join_result,
                                                       distributed_join_result + distributed_join_result_size,
                                                       is_equal())) - distributed_join_result;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        deduplication_time += elapsed_time;

        // Set union of two sets (sorted cc and distributed join result)
        start_time = MPI_Wtime();
        int new_cc_size = distributed_join_result_size + cc_size;
        checkCuda(hipMalloc((void **) &new_cc, new_cc_size * sizeof(Entity)));
        new_cc_size = thrust::set_union(thrust::device,
                                        cc, cc + cc_size,
                                        distributed_join_result,
                                        distributed_join_result + distributed_join_result_size,
                                        new_cc, set_cmp()) - new_cc;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;

        // Deduplicate new cc by keeping only the minimum component ID for each node
        start_time = MPI_Wtime();
        new_cc_size = (thrust::unique(thrust::device,
                                      new_cc,
                                      new_cc + new_cc_size,
                                      is_equal_key())) - new_cc;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        deduplication_time += elapsed_time;

        // Update t delta which is the only new facts which are not in cc and will be used in next iteration
        start_time = MPI_Wtime();
        Entity *t_delta_temp;
        checkCuda(hipMalloc((void **) &t_delta_temp, new_cc_size * sizeof(Entity)));

        t_delta_size = thrust::set_difference(thrust::device,
                                              new_cc, new_cc + new_cc_size,
                                              cc, cc + cc_size,
                                              t_delta_temp, set_cmp()) - t_delta_temp;
        hipFree(t_delta);
        checkCuda(hipMalloc((void **) &t_delta, t_delta_size * sizeof(Entity)));
        hipMemcpy(t_delta, t_delta_temp, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);
        // Update cc
        cc_size = new_cc_size;
        hipMemcpy(cc, new_cc, cc_size * sizeof(Entity), hipMemcpyDeviceToDevice);


        long long t_delta_size_temp_loop = t_delta_size;
        long long old_global_t_delta_size = global_t_delta_size;
        MPI_Allreduce(&t_delta_size_temp_loop, &global_t_delta_size, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);
        iterations++;
        hipFree(distributed_join_result);
        hipFree(new_cc);
        hipFree(join_result);
        hipFree(t_delta_temp);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;
        if (old_global_t_delta_size == global_t_delta_size) {
            break;
        }
    }

    // We are interested only the unique component ID, thus we make the component ID as key and got rid of node
    start_time = MPI_Wtime();
    replace_key_by_value<<<grid_size, block_size>>>(cc, cc_size, cc);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    // Scatter component IDs among relevant processes
    int cc_distributed_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    Entity *cc_distributed = get_split_relation(rank, cc,
                                                cc_size, total_columns, total_rank,
                                                grid_size, block_size, cuda_aware_mpi, &cc_distributed_size,
                                                comm_method,
                                                &buffer_preparation_time_temp, &communication_time_temp, iterations);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;

    // Sort scattered component IDs
    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, cc_distributed, cc_distributed + cc_distributed_size, set_cmp());
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;


    // Calculate Nodes in largest WCC
    int *component_ids;
    checkCuda(hipMalloc((void **) &component_ids, cc_distributed_size * sizeof(int)));
    int *unique_component_ids, *component_sizes;
    checkCuda(hipMalloc((void **) &unique_component_ids, cc_distributed_size * sizeof(int)));
    checkCuda(hipMalloc((void **) &component_sizes, cc_distributed_size * sizeof(int)));
    // Extract keys from the cc_distributed array
    thrust::transform(thrust::device,
                      cc_distributed, cc_distributed + cc_distributed_size, component_ids,
                      [] __device__(const Entity &e) { return e.key; });
    thrust::equal_to<int> binary_pred;
    // Count the occurrences of each component ID
    auto reduce_end = thrust::reduce_by_key(thrust::device,
                                            component_ids, component_ids + cc_distributed_size,
                                            thrust::constant_iterator<int>(1),
                                            unique_component_ids,
                                            component_sizes, binary_pred);
    // Calculate the number of total unique compoennt
    long long total_unique_component = thrust::distance(component_sizes, reduce_end.second);
    // Find the largest component size
    long long max_component_size_current_rank = thrust::reduce(thrust::device,
                                                               component_sizes,
                                                               component_sizes + total_unique_component, -1,
                                                               thrust::maximum<int>());
    long long max_component_size = 0;
    MPI_Allreduce(&max_component_size_current_rank, &max_component_size, 1, MPI_LONG_LONG_INT, MPI_MAX, MPI_COMM_WORLD);

    // Deduplicate component IDs
    start_time = MPI_Wtime();
    cc_distributed_size = (thrust::unique(thrust::device,
                                          cc_distributed, cc_distributed + cc_distributed_size,
                                          is_equal_key())) - cc_distributed;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;

    start_time = MPI_Wtime();
    long long global_component_size = 0;
    long long current_component_size = cc_distributed_size;
    MPI_Allreduce(&current_component_size, &global_component_size, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);

    int *component_ar;
    checkCuda(hipMalloc((void **) &component_ar, cc_distributed_size * total_columns * sizeof(int)));
    get_int_ar_from_entity_ar<<<grid_size, block_size>>>(cc_distributed, cc_distributed_size, component_ar);

    // Copy component ar to host for file write
    int *component_ar_host = (int *) malloc(cc_distributed_size * total_columns * sizeof(int));
    hipMemcpy(component_ar_host, component_ar, cc_distributed_size * total_columns * sizeof(int),
               hipMemcpyDeviceToHost);

    // List the cc counts for each process and calculate the displacements in the final result
    int *component_counts = (int *) calloc(total_rank, sizeof(int));
    MPI_Allgather(&cc_distributed_size, 1, MPI_INT, component_counts, 1, MPI_INT, MPI_COMM_WORLD);

    int *component_displacements = (int *) calloc(total_rank, sizeof(int));
    for (i = 1; i < total_rank; i++) {
        component_displacements[i] = component_displacements[i - 1] + (component_counts[i - 1] * total_columns);
    }
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    if (job_run == 0) {
        // Write the cc to an offset of the output file
        double temp_file_write_time = 0.0;
        parallel_write(rank, total_rank, output_file_name, component_ar_host, component_displacements,
                       total_columns, cc_distributed_size, &temp_file_write_time);
        file_io_time += temp_file_write_time;
    }

    start_time = MPI_Wtime();

    hipFree(distributed_edge);
    hipFree(edge_reverse_temp_device);
    hipFree(edge_temp_device);
    hipFree(edge);
    hipFree(cc);
    hipFree(t_delta);
    hipFree(component_ar);
    hipFree(hash_table);
    hipFree(cc_distributed);
    hipFree(component_ids);
    hipFree(unique_component_ids);
    hipFree(component_sizes);
    free(component_ar_host);
    free(component_counts);
    free(component_displacements);
    free(edge_host);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;
    MPI_Allreduce(&initialization_time, &max_initialization_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&hashtable_build_time, &max_hashtable_build_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&deduplication_time, &max_deduplication_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&join_time, &max_join_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&merge_time, &max_merge_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&buffer_preparation_time, &max_buffer_preparation_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&communication_time, &max_communication_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&file_io_time, &max_fileio_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&finalization_time, &max_finalization_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    total_time = initialization_time + hashtable_build_time + join_time +
                 buffer_preparation_time + communication_time + deduplication_time + merge_time +
                 finalization_time;
    MPI_Allreduce(&total_time, &max_total_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    if (rank == 0) {
        output.block_size = block_size;
        output.grid_size = grid_size;
        output.input_rows = total_rows;
        output.total_rank = total_rank;
        output.iterations = iterations;
        output.output_file_name = output_file_name;
        output.output_size = global_component_size;
        output.output_size_secondary = max_component_size;
        output.total_time = max_total_time;
        output.initialization_time = max_initialization_time;
        output.fileio_time = max_fileio_time;
        output.hashtable_build_time = max_hashtable_build_time;
        output.join_time = max_join_time;
        output.buffer_preparation_time = max_buffer_preparation_time;
        output.communication_time = max_communication_time;
        output.deduplication_time = max_deduplication_time;
        output.merge_time = max_merge_time;
        output.finalization_time = max_finalization_time;
        if (job_run == 0) {
            printf("| # Input | # Process | # Iterations | # CC (# Nodes in largest WCC) | Total Time ");
            printf("| Initialization | File I/O | Hashtable | Join | Buffer preparation | Communication | Deduplication | Merge | Finalization | Output |\n");
            printf("| --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- |\n");
        }
        printf("| %'d | %'d | %'d | %'lld (%'lld) | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %s |\n",
               output.input_rows, output.total_rank, output.iterations,
               output.output_size, output.output_size_secondary, output.total_time,
               output.initialization_time, output.fileio_time, output.hashtable_build_time, output.join_time,
               output.buffer_preparation_time, output.communication_time, output.deduplication_time, output.merge_time,
               output.finalization_time,
               output.output_file_name);
    }
    MPI_Finalize();
}

int main(int argc, char **argv) {
    benchmark(argc, argv);
    return 0;
}
// METHOD 0 = two pass method, 1 = sorting method
// make runwcc DATA_FILE=data/dummy.bin NPROCS=1 CUDA_AWARE_MPI=0 METHOD=0
// make runwcc DATA_FILE=data/dummy.bin NPROCS=2 CUDA_AWARE_MPI=0 METHOD=0
// make runwcc DATA_FILE=data/dummy.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runwcc DATA_FILE=data/flickr.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runwcc DATA_FILE=data/data_214078.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runwcc DATA_FILE=data/data_214078.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=1
// make runwcc DATA_FILE=data/web-Stanford.bin NPROCS=1 CUDA_AWARE_MPI=0 METHOD=0
// make runwcc DATA_FILE=data/roadNet-CA.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0

// /opt/nvidia/hpc_sdk/Linux_x86_64/24.1/comm_libs/hpcx/bin/mpirun -np 8 ./cc.out data/roadNet-CA.bin 1 0