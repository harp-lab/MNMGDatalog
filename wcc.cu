#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <stdio.h>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/iterator/transform_iterator.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/parallel_io.cu"
#include "common/kernels.cu"
#include "common/comm.cu"
#include "common/hash_table.cu"
#include "common/join.cu"

using namespace std;

/*
Base rule:
edge(x,y) ← edge(y,x).
cc(n, n) ← edge(n,_).
t_delta(x, y) ← cc(x, y)

Recursive rule:
join_result(z, x) ← t_delta(y, z), edge(x, y).
join_result(z, x) ← join_result(x, z).
cc_new(y, min(z)) ← cc(y, z) U join_result(z, x)
t_delta(x, y) ← cc_new(y, z) - cc_old(y, z)
cc(x, y) ← cc_new(x, y)

Final rule:
cc_representative_node(n) ← cc(_ , n).
*/
void benchmark(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    Output output;
    KernelTimer timer;
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    warm_up_kernel<<<1, 1>>>();

    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    setlocale(LC_ALL, "");
    double start_time, end_time, elapsed_time, kernel_time;
    start_time = MPI_Wtime();
    double initialization_time = 0.0;
    double finalization_time = 0.0;
    double file_io_time = 0.0;
    double buffer_preparation_time = 0.0, communication_time = 0.0, memory_clear_time = 0.0;
    double buffer_preparation_time_temp = 0.0, communication_time_temp = 0.0, buffer_memory_clear_time_temp = 0.0;
    double join_time = 0.0, merge_time = 0.0;
    double deduplication_time = 0.0;
    double hashtable_build_time = 0.0;

    double total_time = 0.0, max_total_time = 0.0;
    int total_rank, rank;
    int i;
    MPI_Comm_size(MPI_COMM_WORLD, &total_rank);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    int iterations = 0;
    // Should pass the input filename in command line argument
    const char *input_file;
    int comm_method = 0;
    int job_run = 0;
    int cuda_aware_mpi = 0;

    if (argc == 5) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
        job_run = atoi(argv[4]);
    } else if (argc == 4) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
    } else if (argc == 3) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
    } else if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "data/dummy.bin";
    }
    string output_file = string(input_file) + "_cc.bin";
    const char *output_file_name = output_file.c_str();
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    // Read file in parallel
    int total_columns = 2;
    double temp_file_io_time = 0.0;
    int row_size = 0;
    int total_rows = 0;
    int *edge_host = parallel_read(rank, total_rank, input_file, total_columns,
                                   &row_size, &total_rows, &temp_file_io_time);
    int local_count = row_size * total_columns;
    file_io_time += temp_file_io_time;

    start_time = MPI_Wtime();
    int *edge_temp_device;
    checkCuda(hipMalloc((void **) &edge_temp_device, local_count * sizeof(int)));
    hipMemcpy(edge_temp_device, edge_host, local_count * sizeof(int), hipMemcpyHostToDevice);
    // Ensure edges are bidirectional by adding reverse edges
    int *edge_reverse_temp_device;
    checkCuda(hipMalloc((void **) &edge_reverse_temp_device, local_count * sizeof(int)));
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;
    timer.start_timer();
    get_reverse_ar<<<grid_size, block_size>>>(edge_temp_device, row_size, edge_reverse_temp_device);
    timer.stop_timer();
    kernel_time = timer.get_spent_time();
    initialization_time += kernel_time;

    // Create Entity array from GPU buffers, edge + reverse_edge
    // edge(x, y) :- edge(y, x)
    start_time = MPI_Wtime();
    Entity *edge;
    int edge_size = local_count;
    checkCuda(hipMalloc((void **) &edge, edge_size * sizeof(Entity)));
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;
    timer.start_timer();
    create_entity_ar_with_offset<<<grid_size, block_size>>>(edge_temp_device, row_size, edge, 0);
    create_entity_ar_with_offset<<<grid_size, block_size>>>(edge_reverse_temp_device, row_size,
                                                            edge, row_size);
    timer.stop_timer();
    kernel_time = timer.get_spent_time();
    initialization_time += kernel_time;

    // Deduplicate local data
    timer.start_timer();
    thrust::sort(thrust::device, edge, edge + edge_size, set_cmp());
    edge_size = (thrust::unique(thrust::device,
                                edge, edge + edge_size,
                                is_equal())) - edge;
    timer.stop_timer();
    kernel_time = timer.get_spent_time();
    deduplication_time += kernel_time;

#ifdef DEBUG
     show_device_entity_variable(edge, edge_size, rank, "edge", 0);
#endif
    // Distribute edge
    int distributed_edge_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    buffer_memory_clear_time_temp = 0.0;

    Entity *distributed_edge = get_split_relation(rank, edge,
                                                  edge_size, total_columns, total_rank,
                                                  grid_size, block_size, cuda_aware_mpi,
                                                  &distributed_edge_size, comm_method,
                                                  &buffer_preparation_time_temp, &communication_time_temp,
                                                  &buffer_memory_clear_time_temp, iterations);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;
    memory_clear_time += buffer_memory_clear_time_temp;

    // Deduplicate distributed edge
    timer.start_timer();
    thrust::sort(thrust::device, distributed_edge, distributed_edge + distributed_edge_size, set_cmp());
    distributed_edge_size = (thrust::unique(thrust::device,
                                            distributed_edge, distributed_edge + distributed_edge_size,
                                            is_equal())) - distributed_edge;
    timer.stop_timer();
    kernel_time = timer.get_spent_time();
    deduplication_time += kernel_time;

    // Create cc from edge where node, component_id = node, node
    // cc(x, x) :- edge(x, _)
    start_time = MPI_Wtime();
    Entity *cc;
    int cc_size = distributed_edge_size;
    checkCuda(hipMalloc((void **) &cc, cc_size * sizeof(Entity)));
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;
    timer.start_timer();
    same_key_value_entity_ar<<<grid_size, block_size>>>(distributed_edge, cc_size, cc);
    timer.stop_timer();
    kernel_time = timer.get_spent_time();
    initialization_time += kernel_time;

    // Deduplicate cc
    timer.start_timer();
    thrust::sort(thrust::device, cc, cc + cc_size, set_cmp());
    cc_size = (thrust::unique(thrust::device,
                              cc, cc + cc_size,
                              is_equal_key())) - cc;
    timer.stop_timer();
    kernel_time = timer.get_spent_time();
    deduplication_time += kernel_time;

    // t_delta = cc, key-value pair: node - component id
    start_time = MPI_Wtime();
    Entity *t_delta;
    int t_delta_size = cc_size;
    checkCuda(hipMalloc((void **) &t_delta, t_delta_size * sizeof(Entity)));
    hipMemcpy(t_delta, cc, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    start_time = MPI_Wtime();
    long long global_t_delta_size = 0;
    long long t_delta_size_temp = t_delta_size;
    MPI_Allreduce(&t_delta_size_temp, &global_t_delta_size, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    communication_time += elapsed_time;

    // Hash table is Edge
    double temp_hashtable_build_time = 0.0;
    int hash_table_rows = 0;
    Entity *hash_table = get_hash_table(grid_size, block_size, distributed_edge, distributed_edge_size,
                                        &hash_table_rows, &temp_hashtable_build_time);
    hashtable_build_time += temp_hashtable_build_time;

    Entity *new_cc;
    while (true) {
        double temp_join_time = 0.0;
        int join_result_size = 0;
        Entity *join_result = get_join(grid_size, block_size, hash_table, hash_table_rows,
                                       t_delta, t_delta_size,
                                       &join_result_size, &temp_join_time);
        join_time += temp_join_time;
#ifdef DEBUG
//        show_device_entity_variable(join_result, join_result_size, rank, "join_result", 0);
#endif

        // Scatter the join result with reverse among relevant processes
        buffer_preparation_time_temp = 0.0;
        communication_time_temp = 0.0;
        buffer_memory_clear_time_temp = 0.0;
        int distributed_join_result_size = 0;
        Entity *distributed_join_result = get_split_relation(rank, join_result,
                                                             join_result_size, total_columns, total_rank,
                                                             grid_size, block_size, cuda_aware_mpi,
                                                             &distributed_join_result_size,
                                                             comm_method,
                                                             &buffer_preparation_time_temp, &communication_time_temp,
                                                             &buffer_memory_clear_time_temp,
                                                             iterations);
        buffer_preparation_time += buffer_preparation_time_temp;
        communication_time += communication_time_temp;
        memory_clear_time += buffer_memory_clear_time_temp;


        // Deduplicate distributed join result with reverse
        timer.start_timer();
        thrust::sort(thrust::device, distributed_join_result,
                     distributed_join_result + distributed_join_result_size, set_cmp());
        distributed_join_result_size = (thrust::unique(thrust::device,
                                                       distributed_join_result,
                                                       distributed_join_result + distributed_join_result_size,
                                                       is_equal())) - distributed_join_result;
        timer.stop_timer();
        kernel_time = timer.get_spent_time();
        deduplication_time += kernel_time;

#ifdef DEBUG
//        show_device_entity_variable(distributed_join_result, distributed_join_result_size, rank, "distributed_join_result deduplicated", 0);
#endif

        // Set union of two sets (sorted cc and distributed join result)
        start_time = MPI_Wtime();
        int new_cc_size = distributed_join_result_size + cc_size;
        checkCuda(hipMalloc((void **) &new_cc, new_cc_size * sizeof(Entity)));
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;
        timer.start_timer();
        thrust::merge(thrust::device,
                      cc, cc + cc_size,
                      distributed_join_result,
                      distributed_join_result + distributed_join_result_size,
                      new_cc, set_cmp());
        timer.stop_timer();
        kernel_time = timer.get_spent_time();
        merge_time += kernel_time;
        // show_device_entity_variable(new_cc, new_cc_size, rank, "new_cc merged dedpulicated", 0);

        // Deduplicate new cc by keeping only the minimum component ID for each node
        timer.start_timer();
        new_cc_size = (thrust::unique(thrust::device,
                                      new_cc,
                                      new_cc + new_cc_size,
                                      is_equal_key())) - new_cc;
        timer.stop_timer();
        kernel_time = timer.get_spent_time();
        deduplication_time += kernel_time;

        // Update t delta which is the only new facts which are not in cc and will be used in next iteration
        start_time = MPI_Wtime();
        Entity *t_delta_temp;
        checkCuda(hipMalloc((void **) &t_delta_temp, new_cc_size * sizeof(Entity)));
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;
        timer.start_timer();
        t_delta_size = thrust::set_difference(thrust::device,
                                              new_cc, new_cc + new_cc_size,
                                              cc, cc + cc_size,
                                              t_delta_temp, set_cmp()) - t_delta_temp;
        timer.stop_timer();
        kernel_time = timer.get_spent_time();
        merge_time += kernel_time;
        start_time = MPI_Wtime();
        hipFree(t_delta);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        memory_clear_time += elapsed_time;

        start_time = MPI_Wtime();
        checkCuda(hipMalloc((void **) &t_delta, t_delta_size * sizeof(Entity)));
        hipMemcpy(t_delta, t_delta_temp, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;

        // Update cc
        start_time = MPI_Wtime();
        cc_size = new_cc_size;
        hipFree(cc);
        cc = new_cc;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        memory_clear_time += elapsed_time;
//        show_device_entity_variable(cc, cc_size, rank, "cc", 0);
        start_time = MPI_Wtime();
        long long t_delta_size_temp_loop = t_delta_size;
        long long old_global_t_delta_size = global_t_delta_size;
        MPI_Allreduce(&t_delta_size_temp_loop, &global_t_delta_size, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);
        iterations++;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        communication_time += elapsed_time;

        start_time = MPI_Wtime();
        hipFree(distributed_join_result);
        hipFree(join_result);
        hipFree(t_delta_temp);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        memory_clear_time += elapsed_time;
        if (old_global_t_delta_size == global_t_delta_size) {
            break;
        }
#ifdef DEBUG
        cout << "Iteration " << iterations << " ends" << endl;
#endif
    }

    // We are interested only the unique component ID, thus we make the component ID as key and got rid of node
    timer.start_timer();
    replace_key_by_value<<<grid_size, block_size>>>(cc, cc_size, cc);
    timer.stop_timer();
    kernel_time = timer.get_spent_time();
    finalization_time += kernel_time;

    // Scatter component IDs among relevant processes
    int cc_distributed_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    buffer_memory_clear_time_temp = 0.0;
    Entity *cc_distributed = get_split_relation(rank, cc,
                                                cc_size, total_columns, total_rank,
                                                grid_size, block_size, cuda_aware_mpi, &cc_distributed_size,
                                                comm_method,
                                                &buffer_preparation_time_temp, &communication_time_temp,
                                                &buffer_memory_clear_time_temp, iterations);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;
    memory_clear_time += buffer_memory_clear_time_temp;

    // Sort scattered component IDs
    timer.start_timer();
    thrust::sort(thrust::device, cc_distributed, cc_distributed + cc_distributed_size, set_cmp());
    timer.stop_timer();
    kernel_time = timer.get_spent_time();
    deduplication_time += kernel_time;


    // Calculate Nodes in largest WCC
    start_time = MPI_Wtime();
    int *component_ids;
    checkCuda(hipMalloc((void **) &component_ids, cc_distributed_size * sizeof(int)));
    int *unique_component_ids, *component_sizes;
    checkCuda(hipMalloc((void **) &unique_component_ids, cc_distributed_size * sizeof(int)));
    checkCuda(hipMalloc((void **) &component_sizes, cc_distributed_size * sizeof(int)));
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;
    // Extract keys from the cc_distributed array
    timer.start_timer();
    thrust::transform(thrust::device,
                      cc_distributed, cc_distributed + cc_distributed_size, component_ids,
                      [] __device__(const Entity &e) { return e.key; });
    thrust::equal_to<int> binary_pred;
    // Count the occurrences of each component ID
    auto reduce_end = thrust::reduce_by_key(thrust::device,
                                            component_ids, component_ids + cc_distributed_size,
                                            thrust::constant_iterator<int>(1),
                                            unique_component_ids,
                                            component_sizes, binary_pred);
    // Calculate the number of total unique compoennt
    long long total_unique_component = thrust::distance(component_sizes, reduce_end.second);
    // Find the largest component size
    long long max_component_size_current_rank = thrust::reduce(thrust::device,
                                                               component_sizes,
                                                               component_sizes + total_unique_component, -1,
                                                               thrust::maximum<int>());
    timer.stop_timer();
    kernel_time = timer.get_spent_time();
    merge_time += kernel_time;

    start_time = MPI_Wtime();
    long long max_component_size = 0;
    MPI_Allreduce(&max_component_size_current_rank, &max_component_size, 1, MPI_LONG_LONG_INT, MPI_MAX, MPI_COMM_WORLD);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    communication_time += elapsed_time;

    // Deduplicate component IDs
    timer.start_timer();
    cc_distributed_size = (thrust::unique(thrust::device,
                                          cc_distributed, cc_distributed + cc_distributed_size,
                                          is_equal_key())) - cc_distributed;

    timer.stop_timer();
    kernel_time = timer.get_spent_time();
    deduplication_time += kernel_time;

    start_time = MPI_Wtime();
    long long global_component_size = 0;
    long long current_component_size = cc_distributed_size;
    MPI_Allreduce(&current_component_size, &global_component_size, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    communication_time += elapsed_time;

    start_time = MPI_Wtime();
    int *component_ar;
    checkCuda(hipMalloc((void **) &component_ar, cc_distributed_size * total_columns * sizeof(int)));
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    timer.start_timer();
    get_int_ar_from_entity_ar<<<grid_size, block_size>>>(cc_distributed, cc_distributed_size, component_ar);
    timer.stop_timer();
    kernel_time = timer.get_spent_time();
    finalization_time += kernel_time;

    start_time = MPI_Wtime();
    // Copy component ar to host for file write
    int *component_ar_host = (int *) malloc(cc_distributed_size * total_columns * sizeof(int));
    hipMemcpy(component_ar_host, component_ar, cc_distributed_size * total_columns * sizeof(int),
               hipMemcpyDeviceToHost);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    file_io_time += elapsed_time;

    // List the cc counts for each process and calculate the displacements in the final result
    start_time = MPI_Wtime();
    int *component_counts = (int *) calloc(total_rank, sizeof(int));
    MPI_Allgather(&cc_distributed_size, 1, MPI_INT, component_counts, 1, MPI_INT, MPI_COMM_WORLD);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    communication_time += elapsed_time;

    start_time = MPI_Wtime();
    int *component_displacements = (int *) calloc(total_rank, sizeof(int));
    for (i = 1; i < total_rank; i++) {
        component_displacements[i] = component_displacements[i - 1] + (component_counts[i - 1] * total_columns);
    }
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    if (job_run == 0) {
        // Write the cc to an offset of the output file
        double temp_file_write_time = 0.0;
        parallel_write(rank, total_rank, output_file_name, component_ar_host, component_displacements,
                       total_columns, cc_distributed_size, &temp_file_write_time);
        cout << "Generated: " << output_file_name << endl;
        file_io_time += temp_file_write_time;
    }

    start_time = MPI_Wtime();

    hipFree(distributed_edge);
    hipFree(edge_reverse_temp_device);
    hipFree(edge_temp_device);
    hipFree(edge);
    hipFree(cc);
    hipFree(t_delta);
    hipFree(component_ar);
    hipFree(hash_table);
    hipFree(cc_distributed);
    hipFree(component_ids);
    hipFree(unique_component_ids);
    hipFree(component_sizes);
    free(component_ar_host);
    free(component_counts);
    free(component_displacements);
    free(edge_host);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    memory_clear_time += elapsed_time;
    total_time = initialization_time + hashtable_build_time + join_time +
                 buffer_preparation_time + communication_time + deduplication_time + merge_time +
                 finalization_time + memory_clear_time;
    MPI_Allreduce(&total_time, &max_total_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    if (total_time == max_total_time) {
        output.block_size = block_size;
        output.grid_size = grid_size;
        output.input_rows = total_rows;
        output.total_rank = total_rank;
        output.iterations = iterations;
        output.output_file_name = output_file_name;
        output.output_size = global_component_size;
        output.output_size_secondary = max_component_size;
        output.total_time = max_total_time;
        output.initialization_time = initialization_time;
        output.fileio_time = file_io_time;
        output.hashtable_build_time = hashtable_build_time;
        output.join_time = join_time;
        output.buffer_preparation_time = buffer_preparation_time;
        output.communication_time = communication_time;
        output.deduplication_time = deduplication_time;
        output.merge_time = merge_time;
        output.finalization_time = finalization_time;
        output.memory_clear_time = memory_clear_time;
        printf("# Input,# Process,# Iterations,# CC,# Nodes in largest WCC,Total Time,Join,Buffer preparation,Communication,Deduplication,Merge,Initialization,Hashtable,Finalization,Clear,File I/O\n");
        printf("%d,%d,%d,%lld,%lld,%.4lf,%.4lf,%.4lf,%.4lf,%.4lf,%.4lf,%.4lf,%.4lf,%.4lf,%.4lf,%.4lf\n",
               output.input_rows, output.total_rank, output.iterations,
               output.output_size, output.output_size_secondary, output.total_time,
               output.join_time, output.buffer_preparation_time, output.communication_time, output.deduplication_time,
               output.merge_time,
               output.initialization_time, output.hashtable_build_time,
               output.finalization_time, output.memory_clear_time, output.fileio_time);
    }
    MPI_Finalize();
}

int main(int argc, char **argv) {
    benchmark(argc, argv);
    return 0;
}
// METHOD 0 = two pass method, 1 = sorting method
// make runwcc DATA_FILE=data/dummy.bin NPROCS=1 CUDA_AWARE_MPI=0 METHOD=0
// make runwcc DATA_FILE=data/dummy.bin NPROCS=2 CUDA_AWARE_MPI=0 METHOD=0
// make runwcc DATA_FILE=data/dummy.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runwcc DATA_FILE=data/flickr.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runwcc DATA_FILE=data/data_214078.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runwcc DATA_FILE=data/data_214078.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=1
// make runwcc DATA_FILE=data/web-Stanford.bin NPROCS=1 CUDA_AWARE_MPI=0 METHOD=0
// make runwcc DATA_FILE=data/roadNet-CA.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runwcc DATA_FILE=data/data/large_datasets/com-Orkut.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// /opt/nvidia/hpc_sdk/Linux_x86_64/24.1/comm_libs/hpcx/bin/mpirun -np 8 ./cc.out data/roadNet-CA.bin 1 0
// make runwcc DATA_FILE=data/data_cc.bin NPROCS=2 CUDA_AWARE_MPI=0 METHOD=0
// make runwccdebug DATA_FILE=data/paper.bin NPROCS=2 CUDA_AWARE_MPI=0 METHOD=0