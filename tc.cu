#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/kernels.cu"
#include "common/comm.cu"
#include "common/join.cu"

using namespace std;


void benchmark(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    Output output;
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    setlocale(LC_ALL, "");
    double start_time, end_time, elapsed_time;
    double max_fileio_time = 0.0, max_initialization_time = 0.0, max_hashtable_build_time = 0.0, max_finalization_time = 0.0;
    double max_join_time = 0.0, max_merge_time = 0.0;
    double max_buffer_preparation_time = 0.0, max_communication_time = 0.0;
    double buffer_preparation_time = 0.0, communication_time = 0.0;
    double buffer_preparation_time_temp = 0.0, communication_time_temp = 0.0;
    double join_time = 0.0, merge_time = 0.0;
    double deduplication_time = 0.0, max_deduplication_time = 0.0;;
    double file_io_time = 0.0;
    double total_time = 0.0, max_total_time = 0.0;
    int total_rank, rank;
    int i;
    MPI_Comm_size(MPI_COMM_WORLD, &total_rank);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    warm_up_kernel<<<1, 1>>>();
    // Should pass the input filename in command line argument
    const char *input_file;
    int comm_method = 0;
    int job_run = 0;
    int cuda_aware_mpi = 0;

    if (argc == 5) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
        job_run = atoi(argv[4]);
    } else if (argc == 4) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
    } else if (argc == 3) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
    } else if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "hipc_2019.bin";
    }
    string output_file = string(input_file) + "_tc.bin";
    const char *output_file_name = output_file.c_str();

    // READ THE FILE IN PARALLEL
    // Reading filesize in bytes
    start_time = MPI_Wtime();
    struct stat filestats;
    stat(input_file, &filestats);
    off_t filesize = filestats.st_size;

    // Calculating the current rank's starting row and number of rows
    // Scatter larger blocks among processes (non-uniform)
    int total_columns = 2;
    int total_rows = filesize / (sizeof(int) * total_columns);
    int row_start = BLOCK_START(rank, total_rank, total_rows);
    int row_size = BLOCK_SIZE(rank, total_rank, total_rows);
    int local_count = row_size * total_columns;

    // Reading specific portion from the file as char in parallel
    int offset = row_start * total_columns * sizeof(int);
    int *local_data_host = (int *) malloc(local_count * sizeof(int));
    MPI_File mpi_file_buffer;
    if (MPI_File_open(MPI_COMM_WORLD, input_file, MPI_MODE_RDONLY,
                      MPI_INFO_NULL, &mpi_file_buffer) != MPI_SUCCESS) {
        printf("Error opening file %s", input_file);
        MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    }
    MPI_File_read_at(mpi_file_buffer, offset, local_data_host, local_count, MPI_INT, MPI_STATUS_IGNORE);
    MPI_File_close(&mpi_file_buffer);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    file_io_time = elapsed_time;
    start_time = MPI_Wtime();
    int *local_data_device;
    checkCuda(hipMalloc((void **) &local_data_device, local_count * sizeof(int)));
    hipMemcpy(local_data_device, local_data_host, local_count * sizeof(int), hipMemcpyHostToDevice);
    Entity *local_data;
    checkCuda(hipMalloc((void **) &local_data, row_size * sizeof(Entity)));
    Entity *local_data_reverse;
    checkCuda(hipMalloc((void **) &local_data_reverse, row_size * sizeof(Entity)));
    create_entity_ar<<<grid_size, block_size>>>(local_data, row_size, local_data_device);
    create_entity_ar_reverse<<<grid_size, block_size>>>(local_data_reverse, row_size, local_data_device);
    int iterations = 0;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    MPI_Allreduce(&elapsed_time, &max_initialization_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    int input_relation_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    Entity *input_relation = get_split_relation(rank, local_data,
                                                row_size, total_columns, total_rank,
                                                grid_size, block_size, cuda_aware_mpi,
                                                &input_relation_size, comm_method,
                                                &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;

    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    int t_delta_size;
    Entity *t_delta = get_split_relation(rank, local_data_reverse,
                                         row_size, total_columns, total_rank,
                                         grid_size, block_size, cuda_aware_mpi, &t_delta_size, comm_method,
                                         &buffer_preparation_time_temp, &communication_time_temp);
    buffer_preparation_time += buffer_preparation_time_temp;
    communication_time += communication_time_temp;

    start_time = MPI_Wtime();
    thrust::stable_sort(thrust::device, t_delta, t_delta + t_delta_size, set_cmp());
    t_delta_size = (thrust::unique(thrust::device,
                                   t_delta, t_delta + t_delta_size,
                                   is_equal())) - t_delta;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;
    start_time = MPI_Wtime();
    // T_FULL is t delta with first column as key
    Entity *t_full;
    checkCuda(hipMalloc((void **) &t_full, t_delta_size * sizeof(Entity)));
    hipMemcpy(t_full, t_delta, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);

    long long global_t_full_size;
    long long t_full_size = t_delta_size;
    MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);
//    cout << "Rank: " << rank << ", iterations: " << iterations << ", t_full_size: " << t_full_size << ", global_t_full_size: " << global_t_full_size << endl;

    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    merge_time += elapsed_time;
    start_time = MPI_Wtime();
    Entity *hash_table;
    double load_factor = 0.4;
    int hash_table_rows = (int) input_relation_size / load_factor;
    hash_table_rows = pow(2, ceil(log(hash_table_rows) / log(2)));
    checkCuda(hipMalloc((void **) &hash_table, hash_table_rows * sizeof(Entity)));
    Entity negative_entity;
    negative_entity.key = -1;
    negative_entity.value = -1;
    thrust::fill(thrust::device, hash_table, hash_table + hash_table_rows, negative_entity);
    build_hash_table_entity<<<grid_size, block_size>>>(hash_table, hash_table_rows, input_relation,
                                                       input_relation_size);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    MPI_Allreduce(&elapsed_time, &max_hashtable_build_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    while (true) {
        Entity *new_t_full;
        double temp_join_time = 0.0;
        int join_result_size = 0;
        Entity *join_result = get_join(grid_size, block_size, hash_table, hash_table_rows,
                                       t_delta, t_delta_size,
                                       &join_result_size, &temp_join_time);

        join_time += temp_join_time;
        // Scatter the new facts among relevant processes
        buffer_preparation_time_temp = 0.0;
        communication_time_temp = 0.0;
        Entity *t_delta_temp = get_split_relation(rank, join_result,
                                                  join_result_size, total_columns, total_rank,
                                                  grid_size, block_size, cuda_aware_mpi, &t_delta_size,
                                                  comm_method,
                                                  &buffer_preparation_time_temp, &communication_time_temp);
        buffer_preparation_time += buffer_preparation_time_temp;
        communication_time += communication_time_temp;
        start_time = MPI_Wtime();
        // Deduplicate scattered facts
        thrust::stable_sort(thrust::device, t_delta_temp, t_delta_temp + t_delta_size, set_cmp());
        t_delta_size = (thrust::unique(thrust::device,
                                       t_delta_temp, t_delta_temp + t_delta_size,
                                       is_equal())) - t_delta_temp;
        hipFree(t_delta);
        checkCuda(hipMalloc((void **) &t_delta, t_delta_size * sizeof(Entity)));
        hipMemcpy(t_delta, t_delta_temp, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        deduplication_time += elapsed_time;

        start_time = MPI_Wtime();
        // Update t delta which is the only new facts which are not in t full and will be used in next iteration
        t_delta_size = thrust::set_difference(thrust::device,
                                              t_delta, t_delta + t_delta_size,
                                              t_full, t_full + t_full_size,
                                              t_delta, set_cmp()) - t_delta;

        // set union of two sets (sorted t full and t delta)
        long new_t_full_size = t_delta_size + t_full_size;
        checkCuda(hipMalloc((void **) &new_t_full, new_t_full_size * sizeof(Entity)));
        new_t_full_size = thrust::set_union(thrust::device,
                                            t_full, t_full + t_full_size,
                                            t_delta, t_delta + t_delta_size,
                                            new_t_full, set_cmp()) - new_t_full;
        hipFree(t_full);
        checkCuda(hipMalloc((void **) &t_full, new_t_full_size * sizeof(Entity)));
        hipMemcpy(t_full, new_t_full, new_t_full_size * sizeof(Entity), hipMemcpyDeviceToDevice);
        t_full_size = new_t_full_size;
        hipFree(join_result);
        hipFree(new_t_full);
        hipFree(t_delta_temp);
        // Check if the global t full size has changed in this iteration
        long long old_global_t_full_size = global_t_full_size;
        MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);
//        cout << "Rank: " << rank << ", iterations: " << iterations << ", t_full_size: " << t_full_size << ", global_t_full_size: " << global_t_full_size << endl;
        iterations++;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;
        if (old_global_t_full_size == global_t_full_size) {
            break;
        }
    }

    MPI_Allreduce(&deduplication_time, &max_deduplication_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&join_time, &max_join_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&merge_time, &max_merge_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&buffer_preparation_time, &max_buffer_preparation_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&communication_time, &max_communication_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    start_time = MPI_Wtime();
    // Reverse the t_full as we stored it in reverse order initially
    int *t_full_ar;
    checkCuda(hipMalloc((void **) &t_full_ar, t_full_size * total_columns * sizeof(int)));
    reverse_t_full<<<grid_size, block_size>>>(t_full_ar, t_full_size, t_full);

    // Copy t full to host for file write
    int *t_full_ar_host = (int *) malloc(t_full_size * total_columns * sizeof(int));
    hipMemcpy(t_full_ar_host, t_full_ar, t_full_size * total_columns * sizeof(int), hipMemcpyDeviceToHost);

    // List the t full counts for each process and calculate the displacements in the final result
    int *t_full_counts = (int *) calloc(total_rank, sizeof(int));
    MPI_Allgather(&t_full_size, 1, MPI_INT,
                  t_full_counts, 1, MPI_INT, MPI_COMM_WORLD);

    int *t_full_displacements = (int *) calloc(total_rank, sizeof(int));
    for (i = 1; i < total_rank; i++) {
        t_full_displacements[i] = t_full_displacements[i - 1] + (t_full_counts[i - 1] * total_columns);
    }
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    MPI_Allreduce(&elapsed_time, &max_finalization_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    if (job_run == 0) {
        // Write the t full to an offset of the output file
        start_time = MPI_Wtime();
        MPI_File fh;
        MPI_File_open(MPI_COMM_WORLD, output_file_name, MPI_MODE_WRONLY | MPI_MODE_CREATE, MPI_INFO_NULL, &fh);
        int file_offset = t_full_displacements[rank] * sizeof(int);
        MPI_File_write_at(fh, file_offset, t_full_ar_host, t_full_size * total_columns, MPI_INT, MPI_STATUS_IGNORE);
        // Close the file and clean up
        MPI_File_close(&fh);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        file_io_time += elapsed_time;
    }
    MPI_Allreduce(&file_io_time, &max_fileio_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    start_time = MPI_Wtime();
    hipFree(local_data_device);
    hipFree(input_relation);
    hipFree(local_data);
    hipFree(local_data_reverse);
    hipFree(t_full);
    hipFree(t_delta);
    hipFree(t_full_ar);
    hipFree(hash_table);

    free(t_full_ar_host);
    free(t_full_counts);
    free(t_full_displacements);
    free(local_data_host);
    end_time = MPI_Wtime();
    elapsed_time = max_finalization_time + (end_time - start_time);
    MPI_Allreduce(&elapsed_time, &max_finalization_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    total_time = max_initialization_time + max_hashtable_build_time + max_join_time +
                 max_buffer_preparation_time + max_communication_time + max_deduplication_time + max_merge_time +
                 max_finalization_time;
    MPI_Allreduce(&total_time, &max_total_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    if (rank == 0) {
        output.block_size = block_size;
        output.grid_size = grid_size;
        output.input_rows = total_rows;
        output.total_rank = total_rank;
        output.iterations = iterations;
        output.output_file_name = output_file_name;
        output.output_size = global_t_full_size;

        output.total_time = max_total_time;
        output.initialization_time = max_initialization_time;
        output.fileio_time = max_fileio_time;
        output.hashtable_build_time = max_hashtable_build_time;
        output.join_time = max_join_time;
        output.buffer_preparation_time = max_buffer_preparation_time;
        output.communication_time = max_communication_time;
        output.deduplication_time = max_deduplication_time;
        output.merge_time = max_merge_time;
        output.finalization_time = max_finalization_time;
        if (job_run == 0) {
            printf("| # Input | # Process | # Iterations | # TC | Total Time ");
            printf("| Initialization | File I/O | Hashtable | Join | Buffer preparation | Communication | Deduplication | Merge | Finalization | Output |\n");
            printf("| --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- |\n");
        }
        printf("| %'d | %'d | %'d | %'lld | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %s |\n",
               output.input_rows, output.total_rank, output.iterations,
               output.output_size, output.total_time,
               output.initialization_time, output.fileio_time, output.hashtable_build_time, output.join_time,
               output.buffer_preparation_time, output.communication_time, output.deduplication_time, output.merge_time,
               output.finalization_time,
               output.output_file_name);
    }
    MPI_Finalize();
}

int main(int argc, char **argv) {
    benchmark(argc, argv);
    return 0;
}
// METHOD 0 = two pass method, 1 = sorting method
// make runtc DATA_FILE=data/data_23874.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/data_23874.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=1
// make runtc DATA_FILE=data/data_10.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/data_23874.bin NPROCS=8 CUDA_AWARE_MPI=1 METHOD=1
// make runtc DATA_FILE=data/data_147892.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
