#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <sys/stat.h>
#include <fcntl.h>
#include <vector>
#include <unordered_map>
#include <set>
#include <cstring>
#include <string>
#include <clocale>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/kernels.cu"
#include "common/parallel_io.cu"
#include "common/comm.cu"
#include "common/hash_table.cu"
#include "common/join.cu"

using namespace std;


void benchmark(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    MPI_Barrier(MPI_COMM_WORLD);
    int total_rank, rank;
    int i;
    MPI_Comm_size(MPI_COMM_WORLD, &total_rank);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    Output output;
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    setlocale(LC_ALL, "");
    double start_time, end_time, elapsed_time;
    double initialization_time = 0.0;
    double finalization_time = 0.0;
    double file_io_time = 0.0;
    double buffer_preparation_time = 0.0, communication_time = 0.0;
    double buffer_preparation_time_temp = 0.0, communication_time_temp = 0.0;
    double join_time = 0.0, merge_time = 0.0;
    double deduplication_time = 0.0;
    double hashtable_build_time = 0.0;
    double set_diff_time = 0.0, cuda_merge_time = 0.0, t_full_copy_time = 0.0, inner_clear_time = 0.0, t_full_size_all_to_all_time = 0.0;

    double total_time = 0.0, max_total_time = 0.0;
    warm_up_kernel<<<1, 1>>>();
    int iterations = 0;
    // Should pass the input filename in command line argument
    const char *input_file;
    int comm_method = 0;
    int job_run = 0;
    int cuda_aware_mpi = 0;

    if (argc == 5) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
        job_run = atoi(argv[4]);
    } else if (argc == 4) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
        comm_method = atoi(argv[3]);
    } else if (argc == 3) {
        input_file = argv[1];
        cuda_aware_mpi = atoi(argv[2]);
    } else if (argc == 2) {
        input_file = argv[1];
    } else {
        input_file = "hipc_2019.bin";
    }
    string output_file = string(input_file) + "_tc.bin";
    const char *output_file_name = output_file.c_str();

    // Read file in parallel
    int total_columns = 2;
    double temp_file_io_time = 0.0;
    int row_size = 0;
    int total_rows = 0;
    int *local_data_host = parallel_read(rank, total_rank, input_file, total_columns,
                                         &row_size, &total_rows, &temp_file_io_time);
    int local_count = row_size * total_columns;
    file_io_time += temp_file_io_time;

    start_time = MPI_Wtime();
    int *local_data_device;
    checkCuda(hipMalloc((void **) &local_data_device, local_count * sizeof(int)));
    hipMemcpy(local_data_device, local_data_host, local_count * sizeof(int), hipMemcpyHostToDevice);
    Entity *local_data;
    checkCuda(hipMalloc((void **) &local_data, row_size * sizeof(Entity)));
    Entity *local_data_reverse;
    checkCuda(hipMalloc((void **) &local_data_reverse, row_size * sizeof(Entity)));
    create_entity_ar<<<grid_size, block_size>>>(local_data, row_size, local_data_device);
    create_entity_ar_reverse<<<grid_size, block_size>>>(local_data_reverse, row_size, local_data_device);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    initialization_time += elapsed_time;

    int input_relation_size = 0;
    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    Entity *input_relation = get_split_relation(rank, local_data,
                                                row_size, total_columns, total_rank,
                                                grid_size, block_size, cuda_aware_mpi,
                                                &input_relation_size, comm_method,
                                                &buffer_preparation_time_temp, &communication_time_temp, iterations);
    // Calculate LIR and CV
//    auto [lir, cv, max_min] = calculate_load_metrics(input_relation_size, total_rank);

    if (total_rank > 1) {
        buffer_preparation_time += buffer_preparation_time_temp;
        communication_time += communication_time_temp;
    }

    buffer_preparation_time_temp = 0.0;
    communication_time_temp = 0.0;
    int t_delta_size = 0;
    Entity *t_delta = get_split_relation(rank, local_data_reverse,
                                         row_size, total_columns, total_rank,
                                         grid_size, block_size, cuda_aware_mpi, &t_delta_size, comm_method,
                                         &buffer_preparation_time_temp, &communication_time_temp, iterations);
    if (total_rank > 1) {
        buffer_preparation_time += buffer_preparation_time_temp;
        communication_time += communication_time_temp;
    }
    start_time = MPI_Wtime();
    thrust::sort(thrust::device, t_delta, t_delta + t_delta_size, set_cmp());
    t_delta_size = (thrust::unique(thrust::device,
                                   t_delta, t_delta + t_delta_size,
                                   is_equal())) - t_delta;
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    deduplication_time += elapsed_time;
    start_time = MPI_Wtime();
    // T_FULL is t delta with first column as key
    Entity *t_full;
    checkCuda(hipMalloc((void **) &t_full, t_delta_size * sizeof(Entity)));
    hipMemcpy(t_full, t_delta, t_delta_size * sizeof(Entity), hipMemcpyDeviceToDevice);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
//    if (total_rank > 1) {
#ifdef DEBUG
    cout << "t_full initialization: " << elapsed_time << endl;
#endif
    merge_time += elapsed_time;
//    }
    start_time = MPI_Wtime();
    long long global_t_full_size;
    long long t_full_size = t_delta_size;
    MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    if (total_rank > 1) {
#ifdef DEBUG
        cout << "MPI_Allreduce initialization: " << elapsed_time << endl;
#endif
        merge_time += elapsed_time;
    }
    // Hash table is Edge
    double temp_hashtable_build_time = 0.0;
    int hash_table_rows = 0;
    Entity *hash_table = get_hash_table(grid_size, block_size, input_relation, input_relation_size,
                                        &hash_table_rows, &temp_hashtable_build_time);
    hashtable_build_time += temp_hashtable_build_time;

    Entity *new_t_full;
    while (true) {

        double temp_join_time = 0.0;
        int join_result_size = 0;

        Entity *join_result = get_join(grid_size, block_size, hash_table, hash_table_rows,
                                       t_delta, t_delta_size,
                                       &join_result_size, &temp_join_time);

        join_time += temp_join_time;

        // Scatter the new facts among relevant processes
        buffer_preparation_time_temp = 0.0;
        communication_time_temp = 0.0;
        hipFree(t_delta);
        t_delta = get_split_relation(rank, join_result,
                                     join_result_size, total_columns, total_rank,
                                     grid_size, block_size, cuda_aware_mpi, &t_delta_size,
                                     comm_method,
                                     &buffer_preparation_time_temp, &communication_time_temp, iterations);
        if (total_rank > 1) {
            buffer_preparation_time += buffer_preparation_time_temp;
            communication_time += communication_time_temp;
        }
        start_time = MPI_Wtime();
        // Deduplicate scattered facts
        thrust::sort(thrust::device, t_delta, t_delta + t_delta_size, set_cmp());
        t_delta_size = (thrust::unique(thrust::device,
                                       t_delta, t_delta + t_delta_size,
                                       is_equal())) - t_delta;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        deduplication_time += elapsed_time;

        start_time = MPI_Wtime();
        // Update t delta which is the only new facts which are not in t full and will be used in next iteration
        t_delta_size = thrust::set_difference(thrust::device,
                                              t_delta, t_delta + t_delta_size,
                                              t_full, t_full + t_full_size,
                                              t_delta, set_cmp()) - t_delta;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;
        set_diff_time += elapsed_time;
        // set union of two sets (sorted t full and t delta)
        start_time = MPI_Wtime();
        long long new_t_full_size = t_delta_size + t_full_size;
        checkCuda(hipMalloc((void **) &new_t_full, new_t_full_size * sizeof(Entity)));
        start_time = MPI_Wtime();
        thrust::merge(thrust::device,
                      t_full, t_full + t_full_size,
                      t_delta, t_delta + t_delta_size,
                      new_t_full, set_cmp());
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;
        cuda_merge_time += elapsed_time;
        start_time = MPI_Wtime();
        hipFree(t_full);
        t_full = new_t_full;
        t_full_size = new_t_full_size;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;
        t_full_copy_time += elapsed_time;
        start_time = MPI_Wtime();
        hipFree(join_result);
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;
        inner_clear_time += elapsed_time;
        // Check if the global t full size has changed in this iteration
        start_time = MPI_Wtime();
        long long old_global_t_full_size = global_t_full_size;
        MPI_Allreduce(&t_full_size, &global_t_full_size, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);
        iterations++;
        end_time = MPI_Wtime();
        elapsed_time = end_time - start_time;
        merge_time += elapsed_time;
        t_full_size_all_to_all_time += elapsed_time;
        if (old_global_t_full_size == global_t_full_size) {
            break;
        }
    }
//    cout << "Rank: " << rank << ", set diff: " << set_diff_time << ", concat: " << inner_concat_time << ", sort: "
//         << inner_sorting_time << ", merge: " << cuda_merge_time << ", t full cpy: "
//         << t_full_copy_time << ", t full all to all: " << t_full_size_all_to_all_time << ", inner clear: "
//         << inner_clear_time << endl;

    start_time = MPI_Wtime();
    // Reverse the t_full as we stored it in reverse order initially
    int *t_full_ar;
    checkCuda(hipMalloc((void **) &t_full_ar, t_full_size * total_columns * sizeof(int)));
    reverse_t_full<<<grid_size, block_size>>>(t_full_ar, t_full_size, t_full);

    // Copy t full to host for file write
    int *t_full_ar_host = (int *) malloc(t_full_size * total_columns * sizeof(int));
    hipMemcpy(t_full_ar_host, t_full_ar, t_full_size * total_columns * sizeof(int), hipMemcpyDeviceToHost);

    // List the t full counts for each process and calculate the displacements in the final result
    int *t_full_counts = (int *) calloc(total_rank, sizeof(int));
    MPI_Allgather(&t_full_size, 1, MPI_INT,
                  t_full_counts, 1, MPI_INT, MPI_COMM_WORLD);

    int *t_full_displacements = (int *) calloc(total_rank, sizeof(int));
    for (i = 1; i < total_rank; i++) {
        t_full_displacements[i] = t_full_displacements[i - 1] + (t_full_counts[i - 1] * total_columns);
    }
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    if (job_run == 0) {
        // Write the t full to an offset of the output file
        double temp_file_write_time = 0.0;
        parallel_write(rank, total_rank, output_file_name, t_full_ar_host, t_full_displacements,
                       total_columns, t_full_size, &temp_file_write_time);
        file_io_time += temp_file_write_time;
    }

    start_time = MPI_Wtime();
    hipFree(local_data_device);
    hipFree(input_relation);
    hipFree(local_data);
    hipFree(local_data_reverse);
    hipFree(t_full);
    hipFree(new_t_full);
    hipFree(t_delta);
    hipFree(t_full_ar);
    hipFree(hash_table);

    free(t_full_ar_host);
    free(t_full_counts);
    free(t_full_displacements);
    free(local_data_host);
    end_time = MPI_Wtime();
    elapsed_time = end_time - start_time;
    finalization_time += elapsed_time;

    total_time = initialization_time + hashtable_build_time + join_time +
                 buffer_preparation_time + communication_time + deduplication_time + merge_time +
                 finalization_time;
    MPI_Allreduce(&total_time, &max_total_time, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

    // Breakdown time is the breakdown times of the slowest process
    if (total_time == max_total_time) {
        output.block_size = block_size;
        output.grid_size = grid_size;
        output.input_rows = total_rows;
        output.total_rank = total_rank;
        output.iterations = iterations;
        output.output_file_name = output_file_name;
        output.output_size = global_t_full_size;

        output.total_time = max_total_time;
        output.initialization_time = initialization_time;
        output.fileio_time = file_io_time;
        output.hashtable_build_time = hashtable_build_time;
        output.join_time = join_time;
        output.buffer_preparation_time = buffer_preparation_time;
        output.communication_time = communication_time;
        output.deduplication_time = deduplication_time;
        output.merge_time = merge_time;
        output.finalization_time = finalization_time;
        if (job_run == 0) {
            printf("| # Input | # Process | # Iterations | # TC | Total Time ");
            printf("| Initialization | File I/O | Hashtable | Join | Buffer preparation | Communication | Deduplication | Merge | Finalization | Output |\n");
            printf("| --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- | --- |\n");
        }
        printf("| %'d | %'d | %'d | %'lld | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %'8.4lf | %s |\n",
               output.input_rows, output.total_rank, output.iterations,
               output.output_size, output.total_time,
               output.initialization_time, output.fileio_time, output.hashtable_build_time, output.join_time,
               output.buffer_preparation_time, output.communication_time, output.deduplication_time, output.merge_time,
               output.finalization_time,
               output.output_file_name);
    }
    MPI_Finalize();
}

int main(int argc, char **argv) {
    benchmark(argc, argv);
    return 0;
}
// METHOD 0 = two pass method, 1 = sorting method
// make runtc DATA_FILE=data/data_7035.bin NPROCS=3 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/data_5.bin NPROCS=3 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/data_10.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/data_23874.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/data_23874.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=1
// make runtc DATA_FILE=data/data_10.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/data_23874.bin NPROCS=8 CUDA_AWARE_MPI=1 METHOD=1
// make runtc DATA_FILE=data/data_147892.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/dummy.bin NPROCS=2 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/data_23874.bin NPROCS=3 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/data_163734.bin NPROCS=1 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/skewed_data.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/data_88234.bin NPROCS=8 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/data_22.bin NPROCS=3 CUDA_AWARE_MPI=0 METHOD=0
// make runtc DATA_FILE=data/com-dblpungraph.bin NPROCS=1 CUDA_AWARE_MPI=0 METHOD=1